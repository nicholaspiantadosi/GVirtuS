#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *       nvcc -ccbin gcc -I/usr/local/cuda/include  -c main.cpp -o main.o
 *       nvcc -cudart static main.o -lcusolverMg
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipsolver.h"
#include "util.hxx"

//#define SHOW_FORMAT

#ifndef IDX2F
#define IDX2F(i,j,lda) ((((j)-1)*((size_t)lda))+((i)-1))
#endif /* IDX2F */

#ifndef IDX1F
#define IDX1F(i) ((i)-1)
#endif /* IDX1F */

static void print_matrix(
        int m,
        int n,
        const double *A,
        int lda,
        const char* name)
{
    printf("%s = matlab base-1, %d-by-%d matrix\n", name, m, n);
    for(int row = 1 ; row <= m ; row++){
        for(int col = 1 ; col <= n ; col++){
            double Aij = A[IDX2F(row, col, lda)];
            printf("%s(%d,%d) = %20.16E\n", name, row, col, Aij );
        }
    }
}

/* compute |x|_inf */
static double vec_nrm_inf(
        int n,
        const double *x)
{
    double max_nrm = 0;
    for(int row = 1; row <= n ; row++){
        double xi = x[ IDX1F(row) ];
        max_nrm = ( max_nrm > fabs(xi) )? max_nrm : fabs(xi);
    }
    return max_nrm;
}

/* A is 1D laplacian, return A(N:-1:1, :) */
static void gen_1d_laplacian_perm(
        int N,
        double *A,
        int lda)
{
    memset(A, 0, sizeof(double)*lda*N);
    for(int J = 1 ; J <= N; J++ ){
        A[ IDX2F( N-J+1, J, lda ) ] = 2.0;
        if ( (J-1) >= 1 ){
            A[ IDX2F( N-J+1, J-1, lda ) ] = -1.0;
        }
        if ( (J+1) <= N ){
            A[ IDX2F( N-J+1, J+1, lda ) ] = -1.0;
        }
    }
}

int main( int argc, char* argv[])
{
    cusolverMgHandle_t handle = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat = hipSuccess;
/* maximum number of GPUs */
    const int MAX_NUM_DEVICES = 16;

    int nbGpus = 0;
    int deviceList[MAX_NUM_DEVICES];

    const int N   = 611;
    const int IA  = 1;
    const int JA  = 1;
    const int T_A = 256; /* tile size of A */
    const int lda = N;

    const int IB  = 1;
    const int JB  = 1;
    const int T_B = 100; /* tile size of B */
    const int ldb = N;

    double *A = NULL; /* A is N-by-N */
    double *B = NULL; /* B is N-by-1, right-hand-side vector */
    double *X = NULL; /* X is N-by-1, solution vector */
    int *IPIV = NULL; /* IPIV is 1-by-N, pivoting sequence */
    int  info = 0;

    cudaLibMgMatrixDesc_t descrA;
    cudaLibMgMatrixDesc_t descrB;
    cudaLibMgGrid_t gridA;
    cudaLibMgGrid_t gridB;
    cusolverMgGridMapping_t mapping = CUDALIBMG_GRID_MAPPING_COL_MAJOR;

    double **array_d_A = NULL;
    double **array_d_B = NULL;
    int **array_d_IPIV = NULL;

    int64_t lwork_getrf = 0 ;
    int64_t lwork_getrs = 0 ;
    int64_t lwork = 0 ; /* workspace: number of elements per device */
    double **array_d_work = NULL;

    printf("test permuted 1D Laplacian of order %d\n", N);
    printf("step 1: create Mg handle and select devices \n");
    status = cusolverMgCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat = hipGetDeviceCount( &nbGpus );
    assert( hipSuccess == cudaStat );

    nbGpus = (nbGpus < MAX_NUM_DEVICES)? nbGpus : MAX_NUM_DEVICES;
    printf("\tthere are %d GPUs \n", nbGpus);
    for(int j = 0 ; j < nbGpus ; j++){
        deviceList[j] = j;
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, j);
        printf("\tdevice %d, %s, cc %d.%d \n",j, prop.name, prop.major, prop.minor);
    }

    status = cusolverMgDeviceSelect(
            handle,
            nbGpus,
            deviceList);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    printf("step 2: Enable peer access.\n");
    assert( 0 == enablePeerAccess( nbGpus, deviceList ) );

    printf("step 3: allocate host memory A \n");
    A = (double *)malloc (sizeof(double)*lda*N);
    B = (double *)malloc (sizeof(double)*ldb*1);
    X = (double *)malloc (sizeof(double)*ldb*1);
    IPIV = (int *)malloc (sizeof(int)*N);
    assert( NULL != A );
    assert( NULL != B );
    assert( NULL != X );
    assert( NULL != IPIV );

/* permute 1D Laplacian to enable pivoting */
    printf("step 4: prepare permuted 1D Laplacian for A and B = ones(N,1) \n");
    gen_1d_laplacian_perm(
            N,
            &A[ IDX2F( IA, JA, lda ) ],
            lda
    );
#ifdef SHOW_FORMAT
    print_matrix( N, N, A, lda, "A");
#endif
/* B = ones(N,1) */
    for(int row = 1 ; row <= N ; row++){
        B[IDX1F(row)] = 1.0;
    }
    printf("step 5: create matrix descriptors for A and B \n");
    status = cusolverMgCreateDeviceGrid(&gridA, 1, nbGpus, deviceList, mapping );
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    status = cusolverMgCreateDeviceGrid(&gridB, 1, nbGpus, deviceList, mapping );
    assert(HIPSOLVER_STATUS_SUCCESS == status);
/* (global) A is N-by-N */
    status = cusolverMgCreateMatrixDesc(
            &descrA,
            N,   /* nubmer of rows of (global) A */
            N,   /* number of columns of (global) A */
            N,   /* number or rows in a tile */
            T_A, /* number of columns in a tile */
            HIP_R_64F,
            gridA );
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    /* (global) B is N-by-1 */
    status = cusolverMgCreateMatrixDesc(
            &descrB,
            N,   /* nubmer of rows of (global) B */
            1,   /* number of columns of (global) B */
            N,   /* number or rows in a tile */
            T_B, /* number of columns in a tile */
            HIP_R_64F,
            gridB );
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    printf("step 6: allocate distributed matrices A, B and IPIV \n");
    array_d_A = (double**)malloc(sizeof(double*)*nbGpus);
    assert(NULL != array_d_A);
    array_d_B = (double**)malloc(sizeof(double*)*nbGpus);
    assert(NULL != array_d_B);
    array_d_IPIV = (int**)malloc(sizeof(int*)*nbGpus);
    assert(NULL != array_d_IPIV);

/* A := 0 */
    createMat<double>(
            nbGpus,
            deviceList,
            N,   /* number of columns of global A */
            T_A, /* number of columns per column tile */
            lda, /* leading dimension of local A */
            array_d_A
    );
/* B := 0 */
    createMat<double>(
            nbGpus,
            deviceList,
            1,   /* number of columns of global B */
            T_B, /* number of columns per column tile */
            ldb, /* leading dimension of local B */
            array_d_B
    );
/* IPIV := 0, IPIV is consistent with A */
    createMat<int>(
            nbGpus,
            deviceList,
            N,   /* number of columns of global IPIV */
            T_A, /* number of columns per column tile */
            1,   /* leading dimension of local IPIV */
            array_d_IPIV
    );
    printf("step 7: prepare data on devices \n");
/* distribute A to array_d_A */
    memcpyH2D<double>(
            nbGpus,
            deviceList,
            N,
            N,
/* input */
            A,
            lda,
/* output */
            N,   /* number of columns of global A */
            T_A, /* number of columns per column tile */
            lda, /* leading dimension of local A */
            array_d_A,   /* host pointer array of dimension nbGpus */
            IA,
            JA
    );
    /* distribute B to array_d_B */
    memcpyH2D<double>(
            nbGpus,
            deviceList,
            N,
            1,
/* input */
            B,
            ldb,
/* output */
            1,   /* number of columns of global B */
            T_B, /* number of columns per column tile */
            ldb, /* leading dimension of local B */
            array_d_B,   /* host pointer array of dimension nbGpus */
            IB,
            JB
    );

    printf("step 8: allocate workspace space \n");
    status = cusolverMgGetrf_bufferSize(
            handle,
            N,
            N,
            (void**)array_d_A,
            IA, /* base-1 */
            JA, /* base-1 */
            descrA,
            array_d_IPIV,
            HIP_R_64F,
            &lwork_getrf);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = cusolverMgGetrs_bufferSize(
            handle,
            HIPBLAS_OP_N,
            N,
            1, /* NRHS */
            (void**)array_d_A,
            IA,
            JA,
            descrA,
            array_d_IPIV,
            (void**)array_d_B,
            IB,
            JB,
            descrB,
            HIP_R_64F,
            &lwork_getrs);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    lwork = (lwork_getrf > lwork_getrs)? lwork_getrf : lwork_getrs;
    printf("\tallocate device workspace, lwork = %lld \n", (long long)lwork);
    array_d_work = (double**)malloc(sizeof(double*)*nbGpus);
    assert( NULL != array_d_work);

/* array_d_work[j] points to device workspace of device j */
    workspaceAlloc(
            nbGpus,
            deviceList,
            sizeof(double)*lwork, /* number of bytes per device */
            (void**)array_d_work
    );
    cudaStat = hipDeviceSynchronize(); /* sync all devices */
    assert(hipSuccess == cudaStat);
    printf("step 9: solve A*X = B by GETRF and GETRS \n");
    status = cusolverMgGetrf(
            handle,
            N,
            N,
            (void**)array_d_A,
            IA,
            JA,
            descrA,
            array_d_IPIV,
            HIP_R_64F,
            (void**)array_d_work,
            lwork,
            &info  /* host */
    );
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat = hipDeviceSynchronize(); /* sync all devices */
    assert(hipSuccess == cudaStat);
    assert(0 == info); /* check if A is singular  */

    status = cusolverMgGetrs(
            handle,
            HIPBLAS_OP_N,
            N,
            1, /* NRHS */
            (void**)array_d_A,
            IA,
            JA,
            descrA,
            array_d_IPIV,
            (void**)array_d_B,
            IB,
            JB,
            descrB,
            HIP_R_64F,
            (void**)array_d_work,
            lwork,
            &info  /* host */
    );

    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat = hipDeviceSynchronize(); /* sync all devices */
    assert(hipSuccess == cudaStat);
    assert(0 == info); /* check if parameters are valid */

    printf("step 10: retrieve IPIV and solution vector X \n");
    memcpyD2H<double>(
            nbGpus,
            deviceList,
            N,
            1,
/* input */
            1,   /* number of columns of global B */
            T_B, /* number of columns per column tile */
            ldb, /* leading dimension of local B */
            array_d_B,
            IB,
            JB,
/* output */
            X,   /* N-by-1 */
            ldb
    );
    /* IPIV is consistent with A, use JA and T_A */
    memcpyD2H<int>(
            nbGpus,
            deviceList,
            1,
            N,
/* input */
            N,   /* number of columns of global IPIV */
            T_A, /* number of columns per column tile */
            1, /* leading dimension of local IPIV */
            array_d_IPIV,
            1,
            JA,
/* output */
            IPIV,   /* 1-by-N */
            1
    );

#ifdef SHOW_FORMAT
    /* X is N-by-1 */
    print_matrix(N, 1, X, ldb, "X");
#endif

#ifdef SHOW_FORMAT
    /* IPIV is 1-by-N */
    printf("IPIV = matlab base-1, 1-by-%d matrix\n", N);
    for(int row = 1; row <= N ; row++){
        printf("IPIV(%d) = %d \n", row, IPIV[ IDX1F(row) ]);
    }
#endif

    printf("step 11: measure residual error |b - A*x| \n");
    double max_err = 0;
    for(int row = 1; row <= N ; row++){
        double sum = 0.0;
        for(int col = 1; col <= N ; col++){
            double Aij = A[ IDX2F( row, col, lda ) ];
            double  xj = X[ IDX1F(col) ];
            sum += Aij*xj;
        }
        double bi = B[ IDX1F(row) ];
        double err = fabs( bi - sum );

        max_err = ( max_err > err )? max_err : err;
    }
    double x_nrm_inf = vec_nrm_inf(N, X);
    double b_nrm_inf = vec_nrm_inf(N, B);;
    double A_nrm_inf = 4.0;
    double rel_err = max_err/(A_nrm_inf * x_nrm_inf + b_nrm_inf);
    printf("\n|b - A*x|_inf = %E\n", max_err);
    printf("|x|_inf = %E\n", x_nrm_inf);
    printf("|b|_inf = %E\n", b_nrm_inf);
    printf("|A|_inf = %E\n", A_nrm_inf);
/* relative error is around machine zero  */
/* the user can use |b - A*x|/(N*|A|*|x|+|b|) as well */
    printf("|b - A*x|/(|A|*|x|+|b|) = %E\n\n", rel_err);
    printf("step 12: free resources \n");
    destroyMat(
            nbGpus,
            deviceList,
            N,   /* number of columns of global A */
            T_A, /* number of columns per column tile */
            (void**)array_d_A );
    destroyMat(
            nbGpus,
            deviceList,
            1,   /* number of columns of global B */
            T_B, /* number of columns per column tile */
            (void**)array_d_B );
    destroyMat(
            nbGpus,
            deviceList,
            N,   /* number of columns of global IPIV */
            T_A, /* number of columns per column tile */
            (void**)array_d_IPIV );

    workspaceFree( nbGpus, deviceList, (void**)array_d_work );

    if (NULL != A) free(A);
    if (NULL != B) free(B);
    if (NULL != X) free(X);
    if (NULL != IPIV) free(IPIV);

    if (NULL != array_d_A   ) free(array_d_A);
    if (NULL != array_d_B   ) free(array_d_B);
    if (NULL != array_d_IPIV) free(array_d_IPIV);
    if (NULL != array_d_work) free(array_d_work);

    return 0;
}
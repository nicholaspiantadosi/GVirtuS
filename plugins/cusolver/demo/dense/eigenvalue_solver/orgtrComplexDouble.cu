#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    hipDoubleComplex htau_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipDoubleComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (n - 1) * sizeof(hipDoubleComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZungtr_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZungtr(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex valuesTau[n - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (n - 1) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (n - 1) ; i++) {
        printf("%f\n", valuesTau[i].x);
        if (fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgtr test PASSED\n");
    } else {
        printf("orgtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
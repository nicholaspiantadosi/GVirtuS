#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float htau_result[] = {0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    float *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(float)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSorgbr_bufferSize(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSorgbr(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgbr test PASSED\n");
    } else {
        printf("orgbr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
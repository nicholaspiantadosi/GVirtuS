/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include batchsyevj_example.cpp
 *   g++ -o batchsyevj_example batchsyevj_example.o -L/usr/local/cuda/lib64 -lcusolver -lcudart
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 3;
    const int lda = m;
    const int batchSize = 2;
/*
 *        |  1  -1   0 |
 *   A0 = | -1   2   0 |
 *        |  0   0   0 |
 *
 *   A0 = V0 * W0 * V0**T
 *
 *   W0 = diag(0, 0.3820, 2.6180)
 *
 *        |  3   4  0 |
 *   A1 = |  4   7  0 |
 *        |  0   0  0 |
 *
 *   A1 = V1 * W1 * V1**T
 *
 *   W1 = diag(0, 0.5279, 9.4721)
 *
 */
    double A[lda*m*batchSize]; /* A = [A0 ; A1] */
    double V[lda*m*batchSize]; /* V = [V0 ; V1] */
    double W[m*batchSize];     /* W = [W0 ; W1] */
    int info[batchSize];       /* info = [info0 ; info1] */

    double *d_A  = NULL; /* lda-by-m-by-batchSize */
    double *d_W  = NULL; /* m-by-batchSizee */
    int* d_info  = NULL; /* batchSize */
    int lwork = 0;  /* size of workspace */
    double *d_work = NULL; /* device workspace for syevjBatched */

    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_eig  = 0;   /* don't sort eigenvalues */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
    const hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_LOWER;

/* residual and executed_sweeps are not supported on syevjBatched */
    double residual = 0;
    int executed_sweeps = 0;

    double *A0 = A;
    double *A1 = A + lda*m;
/*
 *        |  1  -1   0 |
 *   A0 = | -1   2   0 |
 *        |  0   0   0 |
 *   A0 is column-major
 */
    A0[0 + 0*lda] =  1.0;
    A0[1 + 0*lda] = -1.0;
    A0[2 + 0*lda] =  0.0;

    A0[0 + 1*lda] = -1.0;
    A0[1 + 1*lda] =  2.0;
    A0[2 + 1*lda] =  0.0;

    A0[0 + 2*lda] =  0.0;
    A0[1 + 2*lda] =  0.0;
    A0[2 + 2*lda] =  0.0;
/*
 *        |  3   4  0 |
 *   A1 = |  4   7  0 |
 *        |  0   0  0 |
 *   A1 is column-major
 */
    A1[0 + 0*lda] = 3.0;
    A1[1 + 0*lda] = 4.0;
    A1[2 + 0*lda] = 0.0;

    A1[0 + 1*lda] = 4.0;
    A1[1 + 1*lda] = 7.0;
    A1[2 + 1*lda] = 0.0;

    A1[0 + 2*lda] = 0.0;
    A1[1 + 2*lda] = 0.0;
    A1[2 + 2*lda] = 0.0;
    /* step 1: create cusolver handle, bind a stream  */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: configuration of syevj */
    status = hipsolverDnCreateSyevjInfo(&syevj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of tolerance is machine zero */
    status = hipsolverDnXsyevjSetTolerance(
            syevj_params,
            tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of max. sweeps is 100 */
    status = hipsolverDnXsyevjSetMaxSweeps(
            syevj_params,
            max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* disable sorting */
    status = hipsolverDnXsyevjSetSortEig(
            syevj_params,
            sort_eig);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 3: copy A to device */
    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(double) * lda * m * batchSize);
    cudaStat2 = hipMalloc ((void**)&d_W   , sizeof(double) * m * batchSize);
    cudaStat3 = hipMalloc ((void**)&d_info, sizeof(int   ) * batchSize);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m * batchSize, hipMemcpyHostToDevice);
    cudaStat2 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

/* step 4: query working space of syevjBatched */
    status = hipsolverDnDsyevjBatched_bufferSize(
            cusolverH,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_W,
            &lwork,
            syevj_params,
            batchSize
    );
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);
    /* step 5: compute spectrum of A0 and A1 */
    status = hipsolverDnDsyevjBatched(
            cusolverH,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_W,
            d_work,
            lwork,
            d_info,
            syevj_params,
            batchSize
    );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(V    , d_A   , sizeof(double) * lda * m * batchSize, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(W    , d_W   , sizeof(double) * m * batchSize      , hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int) * batchSize             , hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    for(int i = 0 ; i < batchSize ; i++){
        if ( 0 == info[i] ){
            printf("matrix %d: syevj converges \n", i);
        }else if ( 0 > info[i] ){
/* only info[0] shows if some input parameter is wrong.
 * If so, the error is CUSOLVER_STATUS_INVALID_VALUE.
 */
            printf("Error: %d-th parameter is wrong \n", -info[i] );
            exit(1);
        }else { /* info = m+1 */
/* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            printf("WARNING: matrix %d, info = %d : sygvj does not converge \n", i, info[i] );
        }
    }

/* Step 6: show eigenvalues and eigenvectors */
    double *W0 = W;
    double *W1 = W + m;
    printf("==== \n");
    for(int i = 0 ; i < m ; i++){
        printf("W0[%d] = %f\n", i, W0[i]);
    }
    printf("==== \n");
    for(int i = 0 ; i < m ; i++){
        printf("W1[%d] = %f\n", i, W1[i]);
    }
    printf("==== \n");

    double *V0 = V;
    double *V1 = V + lda*m;
    printf("V0 = (matlab base-1)\n");
    printMatrix(m, m, V0, lda, "V0");
    printf("V1 = (matlab base-1)\n");
    printMatrix(m, m, V1, lda, "V1");
    /*
 * The folowing two functions do not support batched version.
 * The error CUSOLVER_STATUS_NOT_SUPPORTED is returned.
 */
    status = hipsolverDnXsyevjGetSweeps(
            cusolverH,
            syevj_params,
            &executed_sweeps);
    assert(HIPSOLVER_STATUS_NOT_SUPPORTED == status);

    status = hipsolverDnXsyevjGetResidual(
            cusolverH,
            syevj_params,
            &residual);
    assert(HIPSOLVER_STATUS_NOT_SUPPORTED == status);

/* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_W    ) hipFree(d_W);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);

    hipDeviceReset();

    return 0;
}
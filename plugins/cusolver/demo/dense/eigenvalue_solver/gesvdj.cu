#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include gesvdj_example.cpp
 *   g++ -o gesvdj_example gesvdj_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %20.16E\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    const int m = 3;
    const int n = 2;
    const int lda = m; /* A is m-by-n */
    const int ldu = m; /* U is m-by-m */
    const int ldv = n; /* V is n-by-n */
    const int minmn = min(m,n);
/*       | 1 2  |
 *   A = | 4 5  |
 *       | 2 1  |
 */
    double A[lda*n] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    double U[ldu*m]; /* m-by-m unitary matrix, left singular vectors  */
    double V[ldv*n]; /* n-by-n unitary matrix, right singular vectors */
    double S[minmn];     /* numerical singular value */
/* exact singular values */
    double S_exact[] = {7.065283497082729, 1.040081297712078};
    double *d_A = NULL;  /* device copy of A */
    double *d_S = NULL;  /* singular values */
    double *d_U = NULL;  /* left singular vectors */
    double *d_V = NULL;  /* right singular vectors */
    int *d_info = NULL;  /* error info */
    int lwork = 0;       /* size of workspace */
    double *d_work = NULL; /* devie workspace for gesvdj */
    int info = 0;        /* host copy of error info */
    /* configuration of gesvdj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const int econ = 0 ; /* econ = 1 for economy size */

/* numerical results of gesvdj  */
    double residual = 0;
    int executed_sweeps = 0;

    printf("example of gesvdj \n");
    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);
    printf("econ = %d \n", econ);

    printf("A = (matlab base-1)\n");
    printMatrix(m, n, A, lda, "A");
    printf("=====\n");

/* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: configuration of gesvdj */
    status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of tolerance is machine zero */
    status = hipsolverDnXgesvdjSetTolerance(
            gesvdj_params,
            tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of max. sweeps is 100 */
    status = hipsolverDnXgesvdjSetMaxSweeps(
            gesvdj_params,
            max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 3: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(double)*lda*n);
    cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(double)*minmn);
    cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(double)*ldu*m);
    cudaStat4 = hipMalloc ((void**)&d_V   , sizeof(double)*ldv*n);
    cudaStat5 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    /* step 4: query workspace of SVD */
    status = hipsolverDnDgesvdj_bufferSize(
            cusolverH,
            jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
            /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
            econ, /* econ = 1 for economy size */
            m,    /* nubmer of rows of A, 0 <= m */
            n,    /* number of columns of A, 0 <= n  */
            d_A,  /* m-by-n */
            lda,  /* leading dimension of A */
            d_S,  /* min(m,n) */
            /* the singular values in descending order */
            d_U,  /* m-by-m if econ = 0 */
            /* m-by-min(m,n) if econ = 1 */
            ldu,  /* leading dimension of U, ldu >= max(1,m) */
            d_V,  /* n-by-n if econ = 0  */
            /* n-by-min(m,n) if econ = 1  */
            ldv,  /* leading dimension of V, ldv >= max(1,n) */
            &lwork,
            gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work , sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

/* step 5: compute SVD */
    status = hipsolverDnDgesvdj(
            cusolverH,
            jobz,  /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
            /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
            econ,  /* econ = 1 for economy size */
            m,     /* nubmer of rows of A, 0 <= m */
            n,     /* number of columns of A, 0 <= n  */
            d_A,   /* m-by-n */
            lda,   /* leading dimension of A */
            d_S,   /* min(m,n)  */
            /* the singular values in descending order */
            d_U,   /* m-by-m if econ = 0 */
            /* m-by-min(m,n) if econ = 1 */
            ldu,   /* leading dimension of U, ldu >= max(1,m) */
            d_V,   /* n-by-n if econ = 0  */
            /* n-by-min(m,n) if econ = 1  */
            ldv,   /* leading dimension of V, ldv >= max(1,n) */
            d_work,
            lwork,
            d_info,
            gesvdj_params);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(U, d_U, sizeof(double)*ldu*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_V, sizeof(double)*ldv*n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S, d_S, sizeof(double)*minmn, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat5 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    if ( 0 == info ){
        printf("gesvdj converges \n");
    }else if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }else{
        printf("WARNING: info = %d : gesvdj does not converge \n", info );
    }

    printf("S = singular values (matlab base-1)\n");
    printMatrix(minmn, 1, S, minmn, "S");
    printf("=====\n");

    printf("U = left singular vectors (matlab base-1)\n");
    printMatrix(m, m, U, ldu, "U");
    printf("=====\n");

    printf("V = right singular vectors (matlab base-1)\n");
    printMatrix(n, n, V, ldv, "V");
    printf("=====\n");

/* step 6: measure error of singular value */
    double ds_sup = 0;
    for(int j = 0; j < minmn; j++){
        double err = fabs( S[j] - S_exact[j] );
        ds_sup = (ds_sup > err)? ds_sup : err;
    }
    printf("|S - S_exact|_sup = %E \n", ds_sup);

    status = hipsolverDnXgesvdjGetSweeps(
            cusolverH,
            gesvdj_params,
            &executed_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = hipsolverDnXgesvdjGetResidual(
            cusolverH,
            gesvdj_params,
            &residual);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    printf("residual |A - U*S*V**H|_F = %E \n", residual );
    printf("number of executed sweeps = %d \n", executed_sweeps );

/*  free resources  */
    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    if (d_V    ) hipFree(d_V);
    if (d_info) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);

    hipDeviceReset();
    return 0;
}
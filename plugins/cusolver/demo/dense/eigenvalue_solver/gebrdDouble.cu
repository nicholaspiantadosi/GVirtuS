#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double hD_result[] = {-3.741657, -1.573688, -0.339662};
    double hE_result[] = {14.952305, 2.415928, 0.000000};
    double hTAUQ_result[] = {1.267261, 1.654486, 0.000000};
    double hTAUP_result[] = {1.482605, 0.000000, 0.000000};

    double *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDgebrd_bufferSize(handle, m, n, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    double *dD, *dE, *dTAUQ, *dTAUP;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dD, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dE, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dTAUQ, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dTAUP, n * sizeof(double)));
    CUSOLVER_CHECK(hipsolverDnDgebrd(handle, m, n, dA, lda, dD, dE, dTAUQ, dTAUP, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesD[n];
    double valuesE[n];
    double valuesTAUQ[n];
    double valuesTAUP[n];
    CUDA_CHECK( hipMemcpy(valuesD, dD, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, dE, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUQ, dTAUQ, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUP, dTAUP, n * sizeof(double), hipMemcpyDeviceToHost) );

    //int correct = (hdevInfo == 0);
    int correct = 1;
    for (int i = 0; i < n ; i++) {
        printf("%f \t %f \t %f \t %f\n", valuesD[i], valuesE[i], valuesTAUQ[i], valuesTAUP[i]);
        if (fabsf(valuesD[i] - hD_result[i]) > 0.001
        || fabsf(valuesE[i] - hE_result[i]) > 0.001
        || fabsf(valuesTAUQ[i] - hTAUQ_result[i]) > 0.001
        || fabsf(valuesTAUP[i] - hTAUP_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("gebrd test PASSED\n");
    } else {
        printf("gebrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
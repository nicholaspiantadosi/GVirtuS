#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double htau_result[] = {0, 0};

    double *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    double *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (n - 1) * sizeof(double)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDorgtr_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDorgtr(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesTau[n - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (n - 1) * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (n - 1) ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgtr test PASSED\n");
    } else {
        printf("orgtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
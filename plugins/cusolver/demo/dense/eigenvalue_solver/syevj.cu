/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include sygvd_example.cpp
 *   g++ -o a.out sygvd_example.o -L/usr/local/cuda/lib64 -lcusolver
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 3;
    const int lda = m;
/*
 *       | 3.5 0.5 0 |
 *   A = | 0.5 3.5 0 |
 *       | 0   0   2 |
 *
 *       | 10  2   3 |
 *   B = | 2  10   5 |
 *       | 3   5  10 |
 */
    double A[lda*m] = { 3.5, 0.5, 0, 0.5, 3.5, 0, 0, 0, 2.0};
    double B[lda*m] = { 10.0, 2.0, 3.0, 2.0, 10.0, 5.0, 3.0, 5.0, 10.0};
    double lambda[m] = { 0.158660256604, 0.370751508101882, 0.6};

    double V[lda*m]; // eigenvectors
    double W[m]; // eigenvalues

    double *d_A = NULL;
    double *d_B = NULL;
    double *d_W = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;

    printf("A = (matlab base-1)\n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");

    printf("B = (matlab base-1)\n");
    printMatrix(m, m, B, lda, "B");
    printf("=====\n");
    // step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * lda * m);
    cudaStat3 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

// step 3: query working space of sygvd
    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnDsygvd_bufferSize(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute spectrum of (A,B)
    cusolver_status = hipsolverDnDsygvd(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda,
            d_W,
            d_work,
            lwork,
            devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    printf("after sygvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    printf("eigenvalue = (matlab base-1), ascending order\n");
    for(int i = 0 ; i < m ; i++){
        printf("W[%d] = %E\n", i+1, W[i]);
    }

    printf("V = (matlab base-1)\n");
    printMatrix(m, m, V, lda, "V");
    printf("=====\n");

// step 4: check eigenvalues
    double lambda_sup = 0;
    for(int i = 0 ; i < m ; i++){
        double error = fabs( lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error)? lambda_sup : error;
    }
    printf("|lambda - W| = %E\n", lambda_sup);

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return 0;
}
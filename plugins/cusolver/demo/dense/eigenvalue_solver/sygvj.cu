/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include sygvj_example.cpp
 *   g++ -o sygvj_example sygvj_example.o -L/usr/local/cuda/lib64 -lcusolver -lcudart
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 3;
    const int lda = m;
/*
 *       | 3.5 0.5 0 |
 *   A = | 0.5 3.5 0 |
 *       | 0   0   2 |
 *
 *       | 10  2   3 |
 *   B = | 2  10   5 |
 *       | 3   5  10 |
 */
    double A[lda*m] = { 3.5, 0.5, 0, 0.5, 3.5, 0, 0, 0, 2.0};
    double B[lda*m] = { 10.0, 2.0, 3.0, 2.0, 10.0, 5.0, 3.0, 5.0, 10.0};
    double lambda[m] = { 0.158660256604, 0.370751508101882, 0.6};

    double V[lda*m]; /* eigenvectors */
    double W[m];     /* eigenvalues  */

    double *d_A = NULL; /* device copy of A */
    double *d_B = NULL; /* device copy of B */
    double *d_W = NULL; /* numerical eigenvalue */
    int *d_info = NULL; /* error info */
    int  lwork = 0;  /* size of workspace */
    double *d_work = NULL; /* device workspace for sygvj */
    int info = 0; /* host copy of error info */
    /* configuration of sygvj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1; // A*x = (lambda)*B*x
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_LOWER;

/* numerical results of syevj  */
    double residual = 0;
    int executed_sweeps = 0;

    printf("example of sygvj \n");
    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    printf("A = (matlab base-1)\n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");

    printf("B = (matlab base-1)\n");
    printMatrix(m, m, B, lda, "B");
    printf("=====\n");

/* step 1: create cusolver handle, bind a stream  */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: configuration of syevj */
    status = hipsolverDnCreateSyevjInfo(&syevj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of tolerance is machine zero */
    status = hipsolverDnXsyevjSetTolerance(
            syevj_params,
            tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of max. sweeps is 100 */
    status = hipsolverDnXsyevjSetMaxSweeps(
            syevj_params,
            max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    /* step 3: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * lda * m);
    cudaStat3 = hipMalloc ((void**)&d_W, sizeof(double) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

/* step 4: query working space of sygvj */
    status = hipsolverDnDsygvj_bufferSize(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda, /* ldb */
            d_W,
            &lwork,
            syevj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

/* step 5: compute spectrum of (A,B) */
    status = hipsolverDnDsygvj(
            cusolverH,
            itype,
            jobz,
            uplo,
            m,
            d_A,
            lda,
            d_B,
            lda, /* ldb */
            d_W,
            d_work,
            lwork,
            d_info,
            syevj_params);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    if ( 0 == info ){
        printf("sygvj converges \n");
    }else if ( 0 > info ){
        printf("Error: %d-th parameter is wrong \n", -info);
        exit(1);
    }else if ( m >= info ){
        printf("Error: leading minor of order %d of B is not positive definite\n", -info);
        exit(1);
    }else { /* info = m+1 */
        printf("WARNING: info = %d : sygvj does not converge \n", info );
    }

    printf("Eigenvalue = (matlab base-1), ascending order\n");
    for(int i = 0 ; i < m ; i++){
        printf("W[%d] = %E\n", i+1, W[i]);
    }

    printf("V = (matlab base-1)\n");
    printMatrix(m, m, V, lda, "V");
    printf("=====\n");

/* step 6: check eigenvalues */
    double lambda_sup = 0;
    for(int i = 0 ; i < m ; i++){
        double error = fabs( lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error)? lambda_sup : error;
    }
    printf("|lambda - W| = %E\n", lambda_sup);

    status = hipsolverDnXsyevjGetSweeps(
            cusolverH,
            syevj_params,
            &executed_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status = hipsolverDnXsyevjGetResidual(
            cusolverH,
            syevj_params,
            &residual);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    printf("residual |M - V*W*V**H|_F = %E \n", residual );
    printf("number of executed sweeps = %d \n", executed_sweeps );

/* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_W    ) hipFree(d_W);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (syevj_params) hipsolverDnDestroySyevjInfo(syevj_params);

    hipDeviceReset();
    return 0;
}
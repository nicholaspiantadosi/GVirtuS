#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double htau_result[] = {0, 0, 0};

    double *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    double *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(double)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDorgbr_bufferSize(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDorgbr(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgbr test PASSED\n");
    } else {
        printf("orgbr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
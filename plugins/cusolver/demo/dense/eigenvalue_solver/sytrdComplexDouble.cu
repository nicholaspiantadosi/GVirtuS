#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    double hd_result[] = {0.294117, 5.705881, 12};
    double he_result[] = {0.823529, -5.830952};
    hipDoubleComplex htau_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1.857493, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    double *dd, *de;
    hipDoubleComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dd, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &de, (n - 1) * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(hipDoubleComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZhetrd_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dd, de, dtau, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZhetrd(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dd, de, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesD[n];
    double valuesE[n-1];
    hipDoubleComplex valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesD, dd, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, de, (n - 1) * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f \t %f\n", valuesD[i], valuesTau[i]);
        if (fabsf(valuesD[i] - hd_result[i]) > 0.001
        || fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (n - 1) ; i++) {
        printf("%f\n", valuesE[i]);
        if (fabsf(valuesE[i] - he_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("sytrd test PASSED\n");
    } else {
        printf("sytrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;
}
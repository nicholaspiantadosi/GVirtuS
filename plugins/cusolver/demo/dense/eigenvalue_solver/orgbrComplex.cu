#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = 3;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};;

    hipComplex htau_result[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0.000000, 0)};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(hipComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCungbr_bufferSize(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCungbr(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f\n", valuesTau[i].x);
        if (fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgbr test PASSED\n");
    } else {
        printf("orgbr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
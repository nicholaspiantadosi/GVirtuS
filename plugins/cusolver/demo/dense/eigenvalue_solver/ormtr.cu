#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    int ldc = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hC[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float htau_result[] = {0, 0};

    float *dA, *dC;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dC, n * ldc * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, n * ldc * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    float *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (m - 1) * sizeof(float)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSormtr_bufferSize(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSormtr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float valuesTau[m - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (m - 1) * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (m - 1) ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("ormtr test PASSED\n");
    } else {
        printf("ormtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};;

    float hD_result[] = {-3.741657, -1.573688, -0.339662};
    float hE_result[] = {14.952305, 2.415928, 0.000000};
    hipComplex hTAUQ_result[] = {make_hipComplex(1.267261, 0), make_hipComplex(1.654486, 0), make_hipComplex(0.000000, 0)};
    hipComplex hTAUP_result[] = {make_hipComplex(1.482605, 0), make_hipComplex(0.000000, 0), make_hipComplex(0.000000, 0)};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCgebrd_bufferSize(handle, m, n, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    float *dD, *dE;
    hipComplex *dTAUQ, *dTAUP;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((float**) &dD, n * sizeof(float)));
    CUDA_CHECK( hipMalloc((float**) &dE, n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTAUQ, n * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTAUP, n * sizeof(hipComplex)));
    CUSOLVER_CHECK(hipsolverDnCgebrd(handle, m, n, dA, lda, dD, dE, dTAUQ, dTAUP, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float valuesD[n];
    float valuesE[n];
    hipComplex valuesTAUQ[n];
    hipComplex valuesTAUP[n];
    CUDA_CHECK( hipMemcpy(valuesD, dD, n * sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, dE, n * sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUQ, dTAUQ, n * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUP, dTAUP, n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    //int correct = (hdevInfo == 0);
    int correct = 1;
    for (int i = 0; i < n ; i++) {
        printf("%f \t %f \t %f \t %f\n", valuesD[i], valuesE[i], valuesTAUQ[i].x, valuesTAUP[i].x);
        if (fabsf(valuesD[i] - hD_result[i]) > 0.001
        || fabsf(valuesE[i] - hE_result[i]) > 0.001
        || fabsf(valuesTAUQ[i].x - hTAUQ_result[i].x) > 0.001
        || fabsf(valuesTAUP[i].x - hTAUP_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("gebrd test PASSED\n");
    } else {
        printf("gebrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
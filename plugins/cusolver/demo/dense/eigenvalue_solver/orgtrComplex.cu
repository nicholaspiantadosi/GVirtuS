#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};

    hipComplex htau_result[] = {make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (n - 1) * sizeof(hipComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCungtr_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCungtr(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex valuesTau[n - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (n - 1) * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (n - 1) ; i++) {
        printf("%f\n", valuesTau[i].x);
        if (fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgtr test PASSED\n");
    } else {
        printf("orgtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
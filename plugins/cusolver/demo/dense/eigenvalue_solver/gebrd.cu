#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float hD_result[] = {-3.741657, -1.573688, -0.339662};
    float hE_result[] = {14.952305, 2.415928, 0.000000};
    float hTAUQ_result[] = {1.267261, 1.654486, 0.000000};
    float hTAUP_result[] = {1.482605, 0.000000, 0.000000};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSgebrd_bufferSize(handle, m, n, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    float *dD, *dE, *dTAUQ, *dTAUP;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dD, n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dE, n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTAUQ, n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTAUP, n * sizeof(float)));
    CUSOLVER_CHECK(hipsolverDnSgebrd(handle, m, n, dA, lda, dD, dE, dTAUQ, dTAUP, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float valuesD[n];
    float valuesE[n];
    float valuesTAUQ[n];
    float valuesTAUP[n];
    CUDA_CHECK( hipMemcpy(valuesD, dD, n * sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, dE, n * sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUQ, dTAUQ, n * sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUP, dTAUP, n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        //printf("%f \t %f \t %f \t %f\n", valuesD[i], valuesE[i], valuesTAUQ[i], valuesTAUP[i]);
        if (fabsf(valuesD[i] - hD_result[i]) > 0.001
        || fabsf(valuesE[i] - hE_result[i]) > 0.001
        || fabsf(valuesTAUQ[i] - hTAUQ_result[i]) > 0.001
        || fabsf(valuesTAUP[i] - hTAUP_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("gebrd test PASSED\n");
    } else {
        printf("gebrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
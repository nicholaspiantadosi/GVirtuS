#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    int ldc = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hC[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double htau_result[] = {0, 0};

    double *dA, *dC;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dC, n * ldc * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, n * ldc * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    double *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (m - 1) * sizeof(double)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDormtr_bufferSize(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDormtr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesTau[m - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (m - 1) * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (m - 1) ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("ormtr test PASSED\n");
    } else {
        printf("ormtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
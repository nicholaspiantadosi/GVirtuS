#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double hd_result[] = {0.294117, 5.705881, 12};
    double he_result[] = {0.823529, -5.830952};
    double htau_result[] = {0, 1.857493, 0};

    double *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    double *dd, *de, *dtau;
    CUDA_CHECK( hipMalloc((void**) &dd, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &de, (n - 1) * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(double)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDsytrd_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dd, de, dtau, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDsytrd(handle, HIPBLAS_FILL_MODE_UPPER, n, dA, lda, dd, de, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesD[n];
    double valuesE[n-1];
    double valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesD, dd, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, de, (n - 1) * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f \t %f\n", valuesD[i], valuesTau[i]);
        if (fabsf(valuesD[i] - hd_result[i]) > 0.001
        || fabsf(valuesTau[i] - htau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (n - 1) ; i++) {
        printf("%f\n", valuesE[i]);
        if (fabsf(valuesE[i] - he_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("sytrd test PASSED\n");
    } else {
        printf("sytrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;
}
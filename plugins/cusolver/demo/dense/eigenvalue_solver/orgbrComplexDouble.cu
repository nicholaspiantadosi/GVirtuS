#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};;

    hipDoubleComplex htau_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0.000000, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipDoubleComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, n * sizeof(hipDoubleComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZungbr_bufferSize(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZungbr(handle, HIPBLAS_SIDE_LEFT, m, n, k, dA, lda, dtau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex valuesTau[n];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n ; i++) {
        printf("%f\n", valuesTau[i].x);
        if (fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("orgbr test PASSED\n");
    } else {
        printf("orgbr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    int ldc = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hC[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    hipDoubleComplex htau_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA, *dC;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dC, n * ldc * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, n * ldc * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipDoubleComplex *dtau;
    CUDA_CHECK( hipMalloc((void**) &dtau, (m - 1) * sizeof(hipDoubleComplex)));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZunmtr_bufferSize(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZunmtr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, m, n, dA, lda, dtau, dC, ldc, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex valuesTau[m - 1];
    CUDA_CHECK( hipMemcpy(valuesTau, dtau, (m - 1) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < (m - 1) ; i++) {
        printf("%f\n", valuesTau[i]);
        if (fabsf(valuesTau[i].x - htau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("ormtr test PASSED\n");
    } else {
        printf("ormtr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
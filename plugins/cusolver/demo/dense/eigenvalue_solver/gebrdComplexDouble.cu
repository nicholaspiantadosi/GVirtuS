#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};;

    double hD_result[] = {-0.339662, 0.000000, -0.339662};
    double hE_result[] = {14.952305, 2.415928, 0.000000};
    hipDoubleComplex hTAUQ_result[] = {make_hipDoubleComplex(1.267261, 0), make_hipDoubleComplex(1.654486, 0), make_hipDoubleComplex(0.000000, 0)};
    hipDoubleComplex hTAUP_result[] = {make_hipDoubleComplex(1.482605, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.000000, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZgebrd_bufferSize(handle, m, n, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    double *dD, *dE;
    hipDoubleComplex *dTAUQ, *dTAUP;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((double**) &dD, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((double**) &dE, n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dTAUQ, n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTAUP, n * sizeof(hipDoubleComplex)));
    CUSOLVER_CHECK(hipsolverDnZgebrd(handle, m, n, dA, lda, dD, dE, dTAUQ, dTAUP, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double valuesD[n];
    double valuesE[n];
    hipDoubleComplex valuesTAUQ[n];
    hipDoubleComplex valuesTAUP[n];
    CUDA_CHECK( hipMemcpy(valuesD, dD, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesE, dE, n * sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUQ, dTAUQ, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(valuesTAUP, dTAUP, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    //int correct = (hdevInfo == 0);
    int correct = 1;
    for (int i = 0; i < n ; i++) {
        printf("%f \t %f \t %f \t %f\n", valuesD[i], valuesE[i], valuesTAUQ[i].x, valuesTAUP[i].x);
        if (fabsf(valuesD[i] - hD_result[i]) > 0.001
        || fabsf(valuesE[i] - hE_result[i]) > 0.001
        || fabsf(valuesTAUQ[i].x - hTAUQ_result[i].x) > 0.001
        || fabsf(valuesTAUP[i].x - hTAUP_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("gebrd test PASSED\n");
    } else {
        printf("gebrd test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
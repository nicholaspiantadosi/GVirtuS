#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hA_result[] = {make_hipComplex(1.166944, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCpotri_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCpotri(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        printf("%f == %f\n", values[i].x, hA_result[i].x);
        if (fabsf(values[i].x - hA_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotri test PASSED\n");
    } else {
        printf("cusolver_dnpotri test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
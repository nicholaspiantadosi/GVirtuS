#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    int ldx = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hX[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex hX_result[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0)};

    hipDoubleComplex *dA, *dB, *dX;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, m * nrhs * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dX, m * nrhs * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, m * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dX, hX, m * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t lwork_bytes;
    CUSOLVER_CHECK(cusolverDnZEgels_bufferSize(handle, m, n, nrhs, NULL, lda, NULL, ldb, NULL, ldx, NULL, &lwork_bytes));
    //printf("%d\n", lwork_bytes);

    void *dWorkspace;
    hipMalloc((void**)&dWorkspace, lwork_bytes);

    int *devInfo;
    int niter;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnZEgels(handle, m, n, nrhs, dA, lda, dB, ldb, dX, ldx, dWorkspace, lwork_bytes, &niter, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dX, n * nrhs * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            //break;
        }
    }

    if (correct == 1) {
        printf("DnZEgels test PASSED\n");
    } else {
        printf("DnZEgels test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
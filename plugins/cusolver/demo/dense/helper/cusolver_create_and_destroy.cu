
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    hipsolverHandle_t handle = NULL;
    hipStream_t streamIn = NULL;
    hipStream_t streamOut = NULL;

    hipsolverStatus_t cs = hipsolverDnCreate(&handle);

    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverSetStream(handle, streamIn);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverGetStream(handle, &streamOut);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverDnDestroy(handle);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("cusolver_create_and_destroy test PASSED\n");
    } else {
        printf("cusolver_create_and_destroy test FAILED\n");
    }

    hipStreamDestroy(streamIn);
    hipStreamDestroy(streamOut);

    return EXIT_SUCCESS;
}
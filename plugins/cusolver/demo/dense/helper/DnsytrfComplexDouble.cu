#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = n;

    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    int hIpiv_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * n  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZsytrf_bufferSize(handle, n, dA, lda, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    int *devIpiv;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZsytrf(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, devIpiv, Workspace, Lwork, devInfo));
    int hdevInfo;
    int hIpiv[n];
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(&hIpiv, devIpiv, n * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n; i++) {
        printf("%d == %d\n", hIpiv[i], hIpiv_result[i]);
        if (fabsf(hIpiv[i] - hIpiv_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnsytrf test PASSED\n");
    } else {
        printf("Dnsytrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = n;
    int ldc = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hC[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float hTau_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dC, *dTau;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * k * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dC, ldc * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTau, m * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * k  * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, ldc * n  * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSormqr_bufferSize(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, k, dA, lda, dTau, dC, ldc, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSormqr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, k, dA, lda, dTau, dC, ldc, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTau, m * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hTau_result[i]);
        if (fabsf(values[i] - hTau_result[i]) > 0.001) {
            correct = 0;
            //break;
        }
    }

    if (correct == 1) {
        printf("Dnormqr test PASSED\n");
    } else {
        printf("Dnormqr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
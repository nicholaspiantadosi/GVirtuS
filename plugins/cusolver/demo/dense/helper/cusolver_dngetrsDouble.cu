#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hB_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    double *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, n, nrhs, dA, lda, devIpiv, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * n; i++) {
        printf("%f == %f\n", values[i], hB_result[i]);
        if (fabsf(values[i] - hB_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("hipsolverDnSgetrs test PASSED\n");
    } else {
        printf("hipsolverDnSgetrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
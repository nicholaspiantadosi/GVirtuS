#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    int ldb = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hX_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * ldb * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * ldb * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZpotrs(handle, HIPBLAS_FILL_MODE_LOWER, n, ldb, dA, lda, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[n*ldb];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * ldb; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrs test PASSED\n");
    } else {
        printf("cusolver_dnpotrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = 3;
    int ldb = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB_result[] = {-0.4, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    int *info;
    int64_t *devIpiv;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int64_t)));
    CUSOLVER_CHECK(cusolverDnGetrs(handle, params, HIPBLAS_OP_N, n, nrhs, HIP_R_32F, dA, lda, devIpiv, HIP_R_32F, dB, ldb, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i], hB_result[i]);
        if (fabsf(values[i] - hB_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dngetrs test PASSED\n");
    } else {
        printf("dngetrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
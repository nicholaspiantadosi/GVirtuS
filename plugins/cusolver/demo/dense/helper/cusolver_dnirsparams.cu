
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    hipsolverHandle_t handle = NULL;
    cusolverDnIRSParams_t params = NULL;

    hipsolverStatus_t cs = hipsolverDnCreate(&handle);

    int correct = 1;

    cs = cusolverDnIRSParamsCreate(&params);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsSetSolverPrecisions(params, CUSOLVER_R_64F, CUSOLVER_R_16F);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsSetSolverMainPrecision(params, CUSOLVER_R_64F);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsSetSolverLowestPrecision(params, CUSOLVER_R_16F);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsSetRefinementSolver(params, CUSOLVER_IRS_REFINE_CLASSICAL);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double tolerance = 0.1;
    cs = cusolverDnIRSParamsSetTol(params, tolerance);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double toleranceInner = 0.1;
    cs = cusolverDnIRSParamsSetTolInner(params, toleranceInner);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int max_iters = 40;
    cs = cusolverDnIRSParamsSetMaxIters(params, max_iters);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int maxiters_inner = 30;
    cs = cusolverDnIRSParamsSetMaxItersInner(params, maxiters_inner);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsEnableFallback(params);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsDisableFallback(params);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int max_iters_get = 0;
    cs = cusolverDnIRSParamsGetMaxIters(params, &max_iters_get);
    if (cs != HIPSOLVER_STATUS_SUCCESS || max_iters_get != max_iters) {
        correct = 0;
    }

    cusolverDnIRSInfos_t infos = NULL;
    cs = cusolverDnIRSInfosCreate(&infos);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int maxiters_get_2 = 0;
    cs = cusolverDnIRSInfosGetMaxIters(infos, &maxiters_get_2);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int niters = 0;
    cs = cusolverDnIRSInfosGetMaxIters(infos, &niters);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int outer_niters = 0;
    cs = cusolverDnIRSInfosGetMaxIters(infos, &outer_niters);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSInfosRequestResidual(infos);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    /*
    void* residual_history;
    cs = cusolverDnIRSInfosGetResidualHistory(infos, &residual_history);
    if (cs != CUSOLVER_STATUS_SUCCESS) {
        correct = 0;
        printf("%d\n", cs);
    }
    */

    hipsolverDnParams_t dnParams = NULL;
    cs = hipsolverDnCreateParams(&dnParams);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverDnSetAdvOptions(dnParams, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverDnDestroyParams(dnParams);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSInfosDestroy(infos);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsDestroy(params);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("cusolver_dnirsparams test PASSED\n");
    } else {
        printf("cusolver_dnirsparams test FAILED\n");
    }

    hipsolverDnDestroy(handle);

    return EXIT_SUCCESS;

}
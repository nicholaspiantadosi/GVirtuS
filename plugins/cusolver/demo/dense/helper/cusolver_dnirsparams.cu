
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    hipsolverHandle_t handle = NULL;
    cusolverDnIRSParams_t params = NULL;

    hipsolverStatus_t cs = hipsolverDnCreate(&handle);

    cs = cusolverDnIRSParamsCreate(&params);

    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverDnIRSParamsDestroy(params);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("cusolver_dnirsparams test PASSED\n");
    } else {
        printf("cusolver_dnirsparams test FAILED\n");
    }

    hipsolverDnDestroy(handle);

    return EXIT_SUCCESS;
}
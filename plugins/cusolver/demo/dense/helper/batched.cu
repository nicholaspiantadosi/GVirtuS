/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include batchchol_example.cpp 
 *   g++ -o a.out batchchol_example.o -L/usr/local/cuda/lib64 -lcusolver -lcudart
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int batchSize = 2;
    const int nrhs = 1;
    const int m = 3;
    const int lda = m;
    const int ldb = m;
/*       
 *      | 1     2     3 |
 * A0 = | 2     5     5 | = L0 * L0**T
 *      | 3     5    12 |
 *
 *            | 1.0000         0         0 |
 * where L0 = | 2.0000    1.0000         0 |
 *            | 3.0000   -1.0000    1.4142 |
 *
 *      | 1     2     3 |
 * A1 = | 2     4     5 | is not s.p.d., failed at row 2
 *      | 3     5    12 |
 *
 */



    double A0[lda*m] = { 1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0 };
    double A1[lda*m] = { 1.0, 2.0, 3.0, 2.0, 4.0, 5.0, 3.0, 5.0, 12.0 };
    double B0[m] = { 1.0, 1.0, 1.0 };
    double X0[m]; /* X0 = A0\B0 */
    int infoArray[batchSize]; /* host copy of error info */

    double L0[lda*m]; /* cholesky factor of A0 */

    double *Aarray[batchSize];
    double *Barray[batchSize];

    double **d_Aarray = NULL;
    double **d_Barray = NULL;
    int *d_infoArray = NULL;

    printf("example of batched Cholesky \n");

    printf("A0 = (matlab base-1)\n");
    printMatrix(m, m, A0, lda, "A0");
    printf("=====\n");

    printf("A1 = (matlab base-1)\n");
    printMatrix(m, m, A1, lda, "A1");
    printf("=====\n");

    printf("B0 = (matlab base-1)\n");
    printMatrix(m, 1, B0, ldb, "B0");
    printf("=====\n");

/* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: copy A to device */
    for(int j = 0 ; j < batchSize ; j++){
        cudaStat1 = hipMalloc ((void**)&Aarray[j], sizeof(double) * lda * m);
        assert(hipSuccess == cudaStat1);
        cudaStat2 = hipMalloc ((void**)&Barray[j], sizeof(double) * ldb * nrhs);
        assert(hipSuccess == cudaStat2);
    }
    cudaStat1 = hipMalloc ((void**)&d_infoArray, sizeof(int)*batchSize);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMalloc ((void**)&d_Aarray, sizeof(double*) * batchSize);
    cudaStat2 = hipMalloc ((void**)&d_Barray, sizeof(double*) * batchSize);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cudaStat1 = hipMemcpy(Aarray[0], A0, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(Aarray[1], A1, sizeof(double) * lda * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);



    cudaStat1 = hipMemcpy(Barray[0], B0, sizeof(double) * m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(Barray[1], B0, sizeof(double) * m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cudaStat1 = hipMemcpy(d_Aarray, Aarray, sizeof(double*)*batchSize, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_Barray, Barray, sizeof(double*)*batchSize, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    hipDeviceSynchronize();

/* step 3: Cholesky factorization */
    status = hipsolverDnDpotrfBatched(
            handle,
            uplo,
            m,
            d_Aarray,
            lda,
            d_infoArray,
            batchSize);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(infoArray, d_infoArray, sizeof(int)*batchSize, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(L0, Aarray[0], sizeof(double) * lda * m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    for(int j = 0 ; j < batchSize ; j++){
        printf("info[%d] = %d\n", j, infoArray[j]);
    }

    assert( 0 == infoArray[0] );
/* A1 is singular */
    assert( 2 == infoArray[1] );

    printf("L = (matlab base-1), upper triangle is don't care \n");
    printMatrix(m, m, L0, lda, "L0");
    printf("=====\n");

/*
 * step 4: solve A0*X0 = B0 
 *        | 1 |        | 10.5 |
 *   B0 = | 1 |,  X0 = | -2.5 |
 *        | 1 |        | -1.5 |
 */
    status = hipsolverDnDpotrsBatched(
            handle,
            uplo,
            m,
            nrhs, /* only support rhs = 1*/
            d_Aarray,
            lda,
            d_Barray,
            ldb,
            d_infoArray,
            batchSize);



    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(infoArray, d_infoArray, sizeof(int), hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(X0 , Barray[0], sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    hipDeviceSynchronize();

    printf("info = %d\n", infoArray[0]);
    assert( 0 == infoArray[0] );

    printf("X0 = (matlab base-1)\n");
    printMatrix(m, 1, X0, ldb, "X0");
    printf("=====\n");

/* free resources */
    if (d_Aarray    ) hipFree(d_Aarray);
    if (d_Barray    ) hipFree(d_Barray);
    if (d_infoArray ) hipFree(d_infoArray);

    if (handle      ) hipsolverDnDestroy(handle);
    if (stream      ) hipStreamDestroy(stream);

    hipDeviceReset();


    return 0;
}
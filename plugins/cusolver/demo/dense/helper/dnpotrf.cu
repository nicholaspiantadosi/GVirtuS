#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hA_result[] = {1, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    size_t workspaceInBytes;
    CUSOLVER_CHECK(cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, n, HIP_R_32F, dA, lda, HIP_R_32F, &workspaceInBytes));

    void *pBuffer;
    hipMalloc((void**)&pBuffer, workspaceInBytes);

    int *info;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, n, HIP_R_32F, dA, lda, HIP_R_32F, pBuffer, workspaceInBytes, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.05) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dnpotrf test PASSED\n");
    } else {
        printf("dnpotrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
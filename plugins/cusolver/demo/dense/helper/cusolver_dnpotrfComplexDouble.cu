#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hA_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZpotrf(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        if (fabsf(values[i].x - hA_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrf test PASSED\n");
    } else {
        printf("cusolver_dnpotrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
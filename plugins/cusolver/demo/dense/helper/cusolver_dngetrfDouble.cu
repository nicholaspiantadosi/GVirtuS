#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hA_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    double *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, m * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDgetrf(handle, m, n, dA, lda, Workspace, devIpiv, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double values[m*n];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dngetrf test PASSED\n");
    } else {
        printf("cusolver_dngetrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
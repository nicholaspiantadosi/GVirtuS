#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hA_result[] = {make_hipComplex(3, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCgetrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, m * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCgetrf(handle, m, n, dA, lda, Workspace, devIpiv, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hA_result[i].x);
        if (fabsf(values[i].x - hA_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dngetrf test PASSED\n");
    } else {
        printf("cusolver_dngetrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};

    hipComplex hTAU_result[] = {make_hipComplex(1.267261, 0), make_hipComplex(1.801257, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA, *dTAU;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTAU, m * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCgeqrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCgeqrf(handle, m, n, dA, lda, dTAU, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTAU, m * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hTAU_result[i].x);
        if (fabsf(values[i].x - hTAU_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dngeqrf test PASSED\n");
    } else {
        printf("Dngeqrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
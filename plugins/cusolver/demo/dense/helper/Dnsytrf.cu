#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    int hIpiv_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * n  * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSsytrf_bufferSize(handle, n, dA, lda, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    int *devIpiv;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSsytrf(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, devIpiv, Workspace, Lwork, devInfo));
    int hdevInfo;
    int hIpiv[n];
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(&hIpiv, devIpiv, n * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n; i++) {
        printf("%d == %d\n", hIpiv[i], hIpiv_result[i]);
        if (fabsf(hIpiv[i] - hIpiv_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnsytrf test PASSED\n");
    } else {
        printf("Dnsytrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
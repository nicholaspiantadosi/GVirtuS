#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = n;
    int ldc = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hC[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    hipDoubleComplex hTau_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA, *dC, *dTau;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * k * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dC, ldc * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTau, m * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * k  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, ldc * n  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZunmqr_bufferSize(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, k, dA, lda, dTau, dC, ldc, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZunmqr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, k, dA, lda, dTau, dC, ldc, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTau, m * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hTau_result[i].x);
        if (fabsf(values[i].x - hTau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnormqr test PASSED\n");
    } else {
        printf("Dnormqr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    int ldb = 3;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    double hX_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    double *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * ldb * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * ldb * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_LOWER, n, ldb, dA, lda, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double values[n*ldb];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * ldb; i++) {
        printf("%f == %f\n", values[i], hX_result[i]);
        if (fabsf(values[i] - hX_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrs test PASSED\n");
    } else {
        printf("cusolver_dnpotrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
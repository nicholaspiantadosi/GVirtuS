
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    hipsolverHandle_t handle = NULL;
    hipsolverSyevjInfo_t info = NULL;

    hipsolverStatus_t cs = hipsolverDnCreate(&handle);

    cs = hipsolverDnCreateSyevjInfo(&info);

    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double tolerance = 0.1;
    cs = hipsolverDnXsyevjSetTolerance(info, tolerance);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int max_sweeps = 99;
    cs = hipsolverDnXsyevjSetMaxSweeps(info, max_sweeps);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int sort_eig = 1;
    cs = hipsolverDnXsyevjSetSortEig(info, sort_eig);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double residual;
    cs = hipsolverDnXsyevjGetResidual(handle, info, &residual);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int sweeps;
    cs = hipsolverDnXsyevjGetSweeps(handle, info, &sweeps);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverDnDestroySyevjInfo(info);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("cusolver_syevjinfo test PASSED\n");
    } else {
        printf("cusolver_syevjinfo test FAILED\n");
    }

    hipsolverDnDestroy(handle);

    return EXIT_SUCCESS;
}
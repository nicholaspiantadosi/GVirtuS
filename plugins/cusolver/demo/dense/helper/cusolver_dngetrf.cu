#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hA_result[] = {3, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSgetrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, m * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSgetrf(handle, m, n, dA, lda, Workspace, devIpiv, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[m*n];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dngetrf test PASSED\n");
    } else {
        printf("cusolver_dngetrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
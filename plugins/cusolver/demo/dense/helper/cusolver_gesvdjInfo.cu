
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    hipsolverHandle_t handle = NULL;
    hipsolverGesvdjInfo_t info = NULL;

    hipsolverStatus_t cs = hipsolverDnCreate(&handle);

    cs = hipsolverDnCreateGesvdjInfo(&info);

    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double tolerance = 0.1;
    cs = hipsolverDnXgesvdjSetTolerance(info, tolerance);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int max_sweeps = 99;
    cs = hipsolverDnXgesvdjSetMaxSweeps(info, max_sweeps);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int sort_eig = 1;
    cs = hipsolverDnXgesvdjSetSortEig(info, sort_eig);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double residual;
    cs = hipsolverDnXgesvdjGetResidual(handle, info, &residual);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int sweeps;
    cs = hipsolverDnXgesvdjGetSweeps(handle, info, &sweeps);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverDnDestroyGesvdjInfo(info);

    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("cusolver_gesvdjInfo test PASSED\n");
    } else {
        printf("cusolver_gesvdjInfo test FAILED\n");
    }

    hipsolverDnDestroy(handle);

    return EXIT_SUCCESS;
}
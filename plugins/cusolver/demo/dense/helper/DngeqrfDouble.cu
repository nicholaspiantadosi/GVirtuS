#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double hTAU_result[] = {1.267261, 1.801257, 0, 0, 0, 0, 0, 0, 0};

    double *dA, *dTAU;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dTAU, m * n * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDgeqrf(handle, m, n, dA, lda, dTAU, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTAU, m * n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hTAU_result[i]);
        if (fabsf(values[i] - hTAU_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dngeqrf test PASSED\n");
    } else {
        printf("Dngeqrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
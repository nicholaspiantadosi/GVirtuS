#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    int ldb = 3;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hX_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * ldb * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * ldb * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCpotrs(handle, HIPBLAS_FILL_MODE_LOWER, n, ldb, dA, lda, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[n*ldb];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * ldb; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrs test PASSED\n");
    } else {
        printf("cusolver_dnpotrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
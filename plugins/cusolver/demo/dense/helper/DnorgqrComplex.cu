#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};

    hipComplex hTau_result[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA, *dTau;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * k * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTau, m * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * k  * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCungqr_bufferSize(handle, m, n, k, dA, lda, dTau, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCungqr(handle, m, n, k, dA, lda, dTau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTau, m * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hTau_result[i].x);
        if (fabsf(values[i].x - hTau_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnorgqr test PASSED\n");
    } else {
        printf("Dnorgqr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
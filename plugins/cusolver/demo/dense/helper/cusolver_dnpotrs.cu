#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    int ldb = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hX_result[] = {1, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * ldb * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * ldb * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_LOWER, n, ldb, dA, lda, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*ldb];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * ldb; i++) {
        printf("%f == %f\n", values[i], hX_result[i]);
        if (fabsf(values[i] - hX_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrs test PASSED\n");
    } else {
        printf("cusolver_dnpotrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
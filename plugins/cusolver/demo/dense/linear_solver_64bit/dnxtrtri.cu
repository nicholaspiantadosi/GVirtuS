#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float hA_result[] = {1, -0.4, -0.083333, 2, 0.2, -0.083333, 3, 5, 0.083333};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * n * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
    CUSOLVER_CHECK(cusolverDnXtrtri_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_DIAG_NON_UNIT, n, HIP_R_32F, dA, lda, &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

    void *bufferOnDevice;
    hipMalloc((void**)&bufferOnDevice, workspaceInBytesOnDevice);
    size_t bufferOnHost[workspaceInBytesOnHost];

    int *info;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_DIAG_NON_UNIT, n, HIP_R_32F, dA, lda, bufferOnDevice, workspaceInBytesOnDevice, &bufferOnHost, workspaceInBytesOnHost, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[lda * n];
    CUDA_CHECK( hipMemcpy(values, dA, lda * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < lda * n; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.05) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dnxtrtri test PASSED\n");
    } else {
        printf("dnxtrtri test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hTAU_result[] = {1.267261, 1.801257, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dTAU;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTAU, m * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
    CUSOLVER_CHECK(hipsolverDnXgeqrf_bufferSize(handle, params, m, n, HIP_R_32F, dA, lda, HIP_R_32F, dTAU, HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

    void *bufferOnDevice;
    hipMalloc((void**)&bufferOnDevice, workspaceInBytesOnDevice);
    size_t bufferOnHost[workspaceInBytesOnHost];

    int *info;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnXgeqrf(handle, params, m, n, HIP_R_32F, dA, lda, HIP_R_32F, dTAU, HIP_R_32F, bufferOnDevice, workspaceInBytesOnDevice, &bufferOnHost, workspaceInBytesOnHost, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[m * n];
    CUDA_CHECK( hipMemcpy(values, dTAU, m * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hTAU_result[i]);
        if (fabsf(values[i] - hTAU_result[i]) > 0.05) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dnxgeqrf test PASSED\n");
    } else {
        printf("dnxgeqrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hA_result[] = {3, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
    CUSOLVER_CHECK(hipsolverDnXgetrf_bufferSize(handle, params, m, n, HIP_R_32F, dA, lda, HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

    void *bufferOnDevice;
    hipMalloc((void**)&bufferOnDevice, workspaceInBytesOnDevice);
    size_t bufferOnHost[workspaceInBytesOnHost];

    int *info;
    int64_t *devIpiv;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devIpiv, m * sizeof(int64_t)));
    CUSOLVER_CHECK(hipsolverDnXgetrf(handle, params, m, n, HIP_R_32F, dA, lda, devIpiv, HIP_R_32F, bufferOnDevice, workspaceInBytesOnDevice, bufferOnHost, workspaceInBytesOnHost, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.05) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dnxgetrf test PASSED\n");
    } else {
        printf("dnxgetrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
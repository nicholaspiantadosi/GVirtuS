#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    int ldb = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hX_result[] = {1, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * ldb * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * ldb * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    int *info;
    CUDA_CHECK( hipMalloc((void**) &info, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, n, ldb, HIP_R_32F, dA, lda, HIP_R_32F, dB, ldb, info));
    int hInfo;
    CUDA_CHECK( hipMemcpy(&hInfo, info, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        printf("%f == %f\n", values[i], hX_result[i]);
        if (fabsf(values[i] - hX_result[i]) > 0.05) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("dnxpotrs test PASSED\n");
    } else {
        printf("dnxpotrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
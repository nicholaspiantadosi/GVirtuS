#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    hipsolverDnParams_t params = NULL;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    CUSOLVER_CHECK(hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));

    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
    CUSOLVER_CHECK(hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, n, HIP_R_32F, dA, lda, HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

    float *bufferOnDevice;
    hipMalloc((void**)&bufferOnDevice, workspaceInBytesOnDevice);
    float bufferOnHost[workspaceInBytesOnHost];

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, n, HIP_R_32F, dA, lda, HIP_R_32F, bufferOnDevice, workspaceInBytesOnDevice, &bufferOnHost, workspaceInBytesOnHost, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);

    if (correct == 1) {
        printf("dnxpotrf test PASSED\n");
    } else {
        printf("dnxpotrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
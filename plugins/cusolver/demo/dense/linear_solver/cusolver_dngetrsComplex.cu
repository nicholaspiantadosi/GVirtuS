#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hB_result[] = {make_hipComplex(-0.4, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCgetrs(handle, HIPBLAS_OP_N, n, nrhs, dA, lda, devIpiv, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * n; i++) {
        printf("%f == %f\n", values[i].x, hB_result[i].x);
        if (fabsf(values[i].x - hB_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("hipsolverDnSgetrs test PASSED\n");
    } else {
        printf("hipsolverDnSgetrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
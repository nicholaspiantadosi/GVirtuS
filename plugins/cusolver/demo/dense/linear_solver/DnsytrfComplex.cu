#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = n;

    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};

    int hIpiv_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    hipComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * n  * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnCsytrf_bufferSize(handle, n, dA, lda, &Lwork));

    hipComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    int *devIpiv;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnCsytrf(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, devIpiv, Workspace, Lwork, devInfo));
    int hdevInfo;
    int hIpiv[n];
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(&hIpiv, devIpiv, n * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n; i++) {
        printf("%d == %d\n", hIpiv[i], hIpiv_result[i]);
        if (fabsf(hIpiv[i] - hIpiv_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnsytrf test PASSED\n");
    } else {
        printf("Dnsytrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
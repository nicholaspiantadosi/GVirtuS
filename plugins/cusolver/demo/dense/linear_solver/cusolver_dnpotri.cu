#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hA_result[] = {1.166944, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSpotri_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSpotri(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        printf("%f == %f\n", values[i], hA_result[i]);
        if (fabsf(values[i] - hA_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotri test PASSED\n");
    } else {
        printf("cusolver_dnpotri test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
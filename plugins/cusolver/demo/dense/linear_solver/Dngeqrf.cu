#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    float hTAU_result[] = {1.267261, 1.801257, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dTAU;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dTAU, m * n * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSgeqrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSgeqrf(handle, m, n, dA, lda, dTAU, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTAU, m * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hTAU_result[i]);
        if (fabsf(values[i] - hTAU_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dngeqrf test PASSED\n");
    } else {
        printf("Dngeqrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};

    hipDoubleComplex hTAU_result[] = {make_hipDoubleComplex(1.267261, 0), make_hipDoubleComplex(1.801257, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA, *dTAU;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dTAU, m * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZgeqrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZgeqrf(handle, m, n, dA, lda, dTAU, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTAU, m * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hTAU_result[i].x);
        if (fabsf(values[i].x - hTAU_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dngeqrf test PASSED\n");
    } else {
        printf("Dngeqrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
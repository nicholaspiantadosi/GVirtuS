#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB_result[] = {-0.4, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, nrhs, dA, lda, devIpiv, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * n; i++) {
        printf("%f == %f\n", values[i], hB_result[i]);
        if (fabsf(values[i] - hB_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("hipsolverDnSgetrs test PASSED\n");
    } else {
        printf("hipsolverDnSgetrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
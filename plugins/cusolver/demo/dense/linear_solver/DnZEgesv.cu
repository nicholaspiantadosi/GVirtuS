#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    int ldx = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hX[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex hX_result[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0)};

    hipDoubleComplex *dA, *dB, *dX;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dX, n * nrhs * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n  * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dX, hX, n * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t lwork_bytes;
    CUSOLVER_CHECK(cusolverDnZEgesv_bufferSize(handle, n, nrhs, NULL, lda, NULL, NULL, ldb, NULL, ldx, NULL, &lwork_bytes));
    //lwork_bytes = 314880;
    printf("%d\n", lwork_bytes);

    void *dWorkspace;
    hipMalloc((void**)&dWorkspace, lwork_bytes);

    int *devIpiv;
    int *devInfo;
    int niter;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnZEgesv(handle, n, nrhs, dA, lda, devIpiv, dB, ldb, dX, ldx, dWorkspace, lwork_bytes, &niter, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dX, n * nrhs * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("DnZEgesv test PASSED\n");
    } else {
        printf("DnZEgesv test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int lda = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hA_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnZgetrf_bufferSize(handle, m, n, dA, lda, &Lwork));

    hipDoubleComplex *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, m * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZgetrf(handle, m, n, dA, lda, Workspace, devIpiv, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[m*n];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i].x, hA_result[i].x);
        if (fabsf(values[i].x - hA_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dngetrf test PASSED\n");
    } else {
        printf("cusolver_dngetrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
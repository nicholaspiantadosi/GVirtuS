#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    int ldx = n;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hB[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hX[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    float hX_result[] = {1, 0, 0, 0, 1, 0, 0, 0, 1};

    float *dA, *dB, *dX;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dB, m * nrhs * sizeof(float)));
    CUDA_CHECK( hipMalloc((void**) &dX, m * nrhs * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, m * nrhs * sizeof(float), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dX, hX, m * nrhs * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t lwork_bytes;
    CUSOLVER_CHECK(hipsolverDnSSgels_bufferSize(handle, m, n, nrhs, NULL, lda, NULL, ldb, NULL, ldx, NULL, &lwork_bytes));
    //printf("%d\n", lwork_bytes);

    void *dWorkspace;
    hipMalloc((void**)&dWorkspace, lwork_bytes);

    int *devInfo;
    int niter;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSSgels(handle, m, n, nrhs, dA, lda, dB, ldb, dX, ldx, dWorkspace, lwork_bytes, &niter, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dX, n * nrhs * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i], hX_result[i]);
        if (fabsf(values[i] - hX_result[i]) > 0.001) {
            correct = 0;
            //break;
        }
    }

    if (correct == 1) {
        printf("DnSSgels test PASSED\n");
    } else {
        printf("DnSSgels test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
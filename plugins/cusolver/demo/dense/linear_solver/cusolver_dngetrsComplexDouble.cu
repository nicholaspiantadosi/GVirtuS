#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    hipDoubleComplex hA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(12, 0)};
    hipDoubleComplex hB_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex *dA, *dB;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(hipDoubleComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int *devIpiv;
    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnZgetrs(handle, HIPBLAS_OP_N, n, nrhs, dA, lda, devIpiv, dB, ldb, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipDoubleComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dB, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * n; i++) {
        printf("%f == %f\n", values[i].x, hB_result[i].x);
        if (fabsf(values[i].x - hB_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("hipsolverDnSgetrs test PASSED\n");
    } else {
        printf("hipsolverDnSgetrs test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
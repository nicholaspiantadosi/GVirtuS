#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    int ldx = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hX[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex hX_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0)};

    hipComplex *dA, *dB, *dX;
    CUDA_CHECK( hipMalloc((void**) &dA, n * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, n * nrhs * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dX, n * nrhs * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * n  * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, n * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dX, hX, n * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t lwork_bytes;
    CUSOLVER_CHECK(cusolverDnCYgesv_bufferSize(handle, n, nrhs, NULL, lda, NULL, NULL, ldb, NULL, ldx, NULL, &lwork_bytes));
    //lwork_bytes = 88832;
    printf("%d\n", lwork_bytes);

    void *dWorkspace;
    hipMalloc((void**)&dWorkspace, lwork_bytes);

    int *devIpiv;
    int *devInfo;
    int niter;
    CUDA_CHECK( hipMalloc((void**) &devIpiv, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnCYgesv(handle, n, nrhs, dA, lda, devIpiv, dB, ldb, dX, ldx, dWorkspace, lwork_bytes, &niter, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dX, n * nrhs * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("DnCYgesv test PASSED\n");
    } else {
        printf("DnCYgesv test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
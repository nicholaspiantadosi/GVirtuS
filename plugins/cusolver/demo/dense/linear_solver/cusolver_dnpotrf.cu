#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int n = 3;
    int lda = 3;
    float hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};
    float hA_result[] = {1, 0, 0, 0, 0, 0, 0, 0, 0};

    float *dA;
    CUDA_CHECK( hipMalloc((void**) &dA, n * lda * sizeof(float)));
    CUDA_CHECK( hipMemcpy(dA, hA, n * lda * sizeof(float), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, &Lwork));

    float *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_LOWER, n, dA, lda, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    float values[n*lda];
    CUDA_CHECK( hipMemcpy(values, dA, sizeof(int), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * lda; i++) {
        if (fabsf(values[i] - hA_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("cusolver_dnpotrf test PASSED\n");
    } else {
        printf("cusolver_dnpotrf test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
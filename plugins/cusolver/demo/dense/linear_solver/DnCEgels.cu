#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int nrhs = 3;
    int lda = n;
    int ldb = n;
    int ldx = n;
    hipComplex hA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(2, 0), make_hipComplex(5, 0), make_hipComplex(5, 0), make_hipComplex(3, 0), make_hipComplex(5, 0), make_hipComplex(12, 0)};
    hipComplex hX[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex hX_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0)};

    hipComplex *dA, *dB, *dX;
    CUDA_CHECK( hipMalloc((void**) &dA, m * n * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dB, m * nrhs * sizeof(hipComplex)));
    CUDA_CHECK( hipMalloc((void**) &dX, m * nrhs * sizeof(hipComplex)));
    CUDA_CHECK( hipMemcpy(dA, hA, m * n  * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, m * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dX, hX, m * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    size_t lwork_bytes;
    CUSOLVER_CHECK(cusolverDnCEgels_bufferSize(handle, m, n, nrhs, NULL, lda, NULL, ldb, NULL, ldx, NULL, &lwork_bytes));
    //printf("%d\n", lwork_bytes);

    void *dWorkspace;
    hipMalloc((void**)&dWorkspace, lwork_bytes);

    int *devInfo;
    int niter;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(cusolverDnCEgels(handle, m, n, nrhs, dA, lda, dB, ldb, dX, ldx, dWorkspace, lwork_bytes, &niter, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    hipComplex values[n*nrhs];
    CUDA_CHECK( hipMemcpy(values, dX, n * nrhs * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < n * nrhs; i++) {
        printf("%f == %f\n", values[i].x, hX_result[i].x);
        if (fabsf(values[i].x - hX_result[i].x) > 0.001) {
            correct = 0;
            //break;
        }
    }

    if (correct == 1) {
        printf("DnCEgels test PASSED\n");
    } else {
        printf("DnCEgels test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
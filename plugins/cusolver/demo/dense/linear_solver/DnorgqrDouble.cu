#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include "../../cusolver_utils.h"
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

int main(void) {

    int m = 3;
    int n = 3;
    int k = 3;
    int lda = n;
    double hA[] = {1, 2, 3, 2, 5, 5, 3, 5, 12};

    double hTau_result[] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

    double *dA, *dTau;
    CUDA_CHECK( hipMalloc((void**) &dA, lda * k * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dTau, m * n * sizeof(double)));
    CUDA_CHECK( hipMemcpy(dA, hA, lda * k  * sizeof(double), hipMemcpyHostToDevice) );

    hipsolverHandle_t handle = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));

    int Lwork;
    CUSOLVER_CHECK(hipsolverDnDorgqr_bufferSize(handle, m, n, k, dA, lda, dTau, &Lwork));

    double *Workspace;
    hipMalloc((void**)&Workspace, Lwork);

    int *devInfo;
    CUDA_CHECK( hipMalloc((void**) &devInfo, sizeof(int)));
    CUSOLVER_CHECK(hipsolverDnDorgqr(handle, m, n, k, dA, lda, dTau, Workspace, Lwork, devInfo));
    int hdevInfo;
    CUDA_CHECK( hipMemcpy(&hdevInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost) );
    double values[m*n];
    CUDA_CHECK( hipMemcpy(values, dTau, m * n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = (hdevInfo == 0);
    for (int i = 0; i < m * n; i++) {
        printf("%f == %f\n", values[i], hTau_result[i]);
        if (fabsf(values[i] - hTau_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    if (correct == 1) {
        printf("Dnorgqr test PASSED\n");
    } else {
        printf("Dnorgqr test FAILED\n");
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));

    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../cusolver_utils.h"

int main(void) {

    hipsolverRfHandle_t handle = NULL;

    hipsolverStatus_t cs = hipsolverRfCreate(&handle);
    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int n = 3;
    int nnzA = 9;
    double hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    int hCsrRowPtrA[] = {0, 3, 6, 9};
    int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzL = 9;
    double hCsrValL[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    int hCsrRowPtrL[] = {0, 3, 6, 9};
    int hCsrColIndL[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzU = 9;
    double hCsrValU[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    int hCsrRowPtrU[] = {0, 3, 6, 9};
    int hCsrColIndU[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int hP[] = {1, 1, 1};
    int hQ[] = {1, 1, 1};
    cs = hipsolverRfSetupHost(n, nnzA, hCsrRowPtrA, hCsrColIndA, hCsrValA, nnzL, hCsrRowPtrL, hCsrColIndL, hCsrValL, nnzU, hCsrRowPtrU, hCsrColIndU, hCsrValU, hP, hQ, handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfAnalyze(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfRefactor(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int nnzM;
    int *hMp;
    int *hMi;
    double *hMx;
    cs = hipsolverRfExtractBundledFactorsHost(handle, &nnzM, &hMp, &hMi, &hMx);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    printf("nnzM: %d\n", nnzM);
    correct = nnzM == 15;
    int hMp_result[] = {0, 5, 10, 15};
    int hMi_result[] = {0, 1, 0, 1, 2, 0, 1, 0, 1, 2, 0, 1, 0, 1, 2};
    double hMx_result[] = {0, 0, 10, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    printArray(hMp, (n+1), "Mp");
    printArray(hMi, nnzM, "Mi");
    printArrayD(hMx, nnzM, "Mx");
    for (int i = 0; i < (n + 1); i++) {
        if (fabsf(hMp[i] - hMp_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzM; i++) {
        if (fabsf(hMi[i] - hMi_result[i]) > 0.001 || fabsf(hMx[i] - hMx_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    int nnzL2;
    int *hLp2;
    int *hLi2;
    double *hLx2;
    int nnzU2;
    int *hUp2;
    int *hUi2;
    double *hUx2;
    cs = hipsolverRfExtractSplitFactorsHost(handle, &nnzL2, &hLp2, &hLi2, &hLx2, &nnzU2, &hUp2, &hUi2, &hUx2);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    printf("nnzL: %d\n", nnzL2);
    correct = nnzL2 == 11;
    int hLp2_result[] = {0, 2, 6, 11};
    int hLi2_result[] = {0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 2};
    double hLx2_result[] = {1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 1};
    printArray(hLp2, (n+1), "Lp");
    printArray(hLi2, nnzL2, "Li");
    printArrayD(hLx2, nnzL2, "Lx");
    for (int i = 0; i < (n + 1); i++) {
        if (fabsf(hLp2[i] - hLp2_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzL2; i++) {
        if (fabsf(hLi2[i] - hLi2_result[i]) > 0.001 || fabsf(hLx2[i] - hLx2_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    printf("nnzU: %d\n", nnzU2);
    correct = nnzU2 == 9;
    int hUp2_result[] = {0, 5, 8, 9};
    int hUi2_result[] = {0, 1, 0, 1, 2, 1, 1, 2, 2};
    double hUx2_result[] = {0., 0., 10., 0., 0., 0., 0., 0., 0.};
    printArray(hUp2, (n+1), "Up");
    printArray(hUi2, nnzU2, "Ui");
    printArrayD(hUx2, nnzU2, "Ux");
    for (int i = 0; i < (n + 1); i++) {
        if (fabsf(hUp2[i] - hUp2_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzU2; i++) {
        if (fabsf(hUi2[i] - hUi2_result[i]) > 0.001 || fabsf(hUx2[i] - hUx2_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    cs = hipsolverRfDestroy(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    if (correct == 1) {
        printf("rf_create_destroy_host test PASSED\n");
    } else {
        printf("rf_create_destroy_host test FAILED\n");
    }

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../cusolver_utils.h"

int main(void) {

    hipsolverRfHandle_t handle = NULL;

    hipsolverStatus_t cs = hipsolverRfCreate(&handle);
    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int n = 3;
    int nnzA = 9;
    double hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzL = 9;
    double hCsrValL[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrL[] = {0, 3, 6, 9};
    const int hCsrColIndL[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzU = 9;
    double hCsrValU[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrU[] = {0, 3, 6, 9};
    const int hCsrColIndU[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int hP[] = {1, 1, 1};
    int hQ[] = {1, 1, 1};
    double *dCsrValA, *dCsrValL, *dCsrValU;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrL, *dCsrColIndL, *dCsrRowPtrU, *dCsrColIndU, *dP, *dQ;
    CUDA_CHECK( hipMalloc((void**) &dCsrValA, nnzA * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrA, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndA, nnzA * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrValL, nnzL * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrL, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndL, nnzL * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrValU, nnzU * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrU, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndU, nnzU * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dP, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dQ, n * sizeof(int)));
    CUDA_CHECK( hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrValL, hCsrValL, nnzL * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrL, hCsrRowPtrL, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndL, hCsrColIndL, nnzL * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrValU, hCsrValU, nnzU * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrU, hCsrRowPtrU, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndU, hCsrColIndU, nnzU * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dP, hP, n * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dQ, hQ, n * sizeof(int), hipMemcpyHostToDevice) );
    cs = hipsolverRfSetupDevice(n, nnzA, dCsrRowPtrA, dCsrColIndA, dCsrValA, nnzL, dCsrRowPtrL, dCsrColIndL, dCsrValL, nnzU, dCsrRowPtrU, dCsrColIndU, dCsrValU, dP, dQ, handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfAnalyze(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfRefactor(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int nnzM;
    int *dMp, *dMi;
    double *dMx;
    CUDA_CHECK( hipMalloc((void**) &dMp, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dMi, nnzM * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dMx, nnzM * sizeof(double)));
    cs = hipsolverRfAccessBundledFactorsDevice(handle, &nnzM, &dMp, &dMi, &dMx);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("nnzM: %d\n", nnzM);
    correct = nnzM == 15;
    int hMp[n + 1];
    int hMi[nnzM];
    double hMx[nnzM];
    int hMp_result[] = {0, 5, 10, 15};
    int hMi_result[] = {0, 1, 0, 1, 2, 0, 1, 0, 1, 2, 0, 1, 0, 1, 2};
    double hMx_result[] = {0.000000, 0.000000, 10.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000};
    CUDA_CHECK( hipMemcpy(hMp, dMp, (n + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(hMi, dMi, nnzM * sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(hMx, dMx, nnzM * sizeof(double), hipMemcpyDeviceToHost) );
    //printArray(hMp, (n+1), "Mp");
    //printArray(hMi, nnzM, "Mi");
    //printArrayD(hMx, nnzM, "Mx");
    for (int i = 0; i < (n + 1); i++) {
        if (fabsf(hMp[i] - hMp_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzM; i++) {
        if (fabsf(hMi[i] - hMi_result[i]) > 0.001 || fabsf(hMx[i] - hMx_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    cs = hipsolverRfDestroy(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    CUDA_CHECK(hipFree(dCsrValA));
    CUDA_CHECK(hipFree(dCsrRowPtrA));
    CUDA_CHECK(hipFree(dCsrColIndA));
    CUDA_CHECK(hipFree(dCsrValL));
    CUDA_CHECK(hipFree(dCsrRowPtrL));
    CUDA_CHECK(hipFree(dCsrColIndL));
    CUDA_CHECK(hipFree(dCsrValU));
    CUDA_CHECK(hipFree(dCsrRowPtrU));
    CUDA_CHECK(hipFree(dCsrColIndU));
    CUDA_CHECK(hipFree(dP));
    CUDA_CHECK(hipFree(dQ));

    if (correct == 1) {
        printf("rf_create_destroy test PASSED\n");
    } else {
        printf("rf_create_destroy test FAILED\n");
    }

    return EXIT_SUCCESS;
}
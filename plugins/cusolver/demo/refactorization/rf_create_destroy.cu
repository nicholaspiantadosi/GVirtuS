#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsolver.h>         // cusolverDn
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "../cusolver_utils.h"

int main(void) {

    hipsolverRfHandle_t handle = NULL;

    hipsolverStatus_t cs = hipsolverRfCreate(&handle);
    int correct = 1;
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int n = 3;
    int nnzA = 9;
    double hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzL = 9;
    double hCsrValL[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrL[] = {0, 3, 6, 9};
    const int hCsrColIndL[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int nnzU = 9;
    double hCsrValU[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrU[] = {0, 3, 6, 9};
    const int hCsrColIndU[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int hP[] = {1, 1, 1};
    int hQ[] = {1, 1, 1};
    double *dCsrValA, *dCsrValL, *dCsrValU;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrL, *dCsrColIndL, *dCsrRowPtrU, *dCsrColIndU, *dP, *dQ;
    CUDA_CHECK( hipMalloc((void**) &dCsrValA, nnzA * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrA, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndA, nnzA * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrValL, nnzL * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrL, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndL, nnzL * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrValU, nnzU * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrU, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndU, nnzU * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dP, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dQ, n * sizeof(int)));
    CUDA_CHECK( hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrValL, hCsrValL, nnzL * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrL, hCsrRowPtrL, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndL, hCsrColIndL, nnzL * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrValU, hCsrValU, nnzU * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrU, hCsrRowPtrU, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndU, hCsrColIndU, nnzU * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dP, hP, n * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dQ, hQ, n * sizeof(int), hipMemcpyHostToDevice) );
    cs = hipsolverRfSetupDevice(n, nnzA, dCsrRowPtrA, dCsrColIndA, dCsrValA, nnzL, dCsrRowPtrL, dCsrColIndL, dCsrValL, nnzU, dCsrRowPtrU, dCsrColIndU, dCsrValU, dP, dQ, handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfAnalyze(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfRefactor(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int nnzM;
    int *dMp, *dMi;
    double *dMx;
    CUDA_CHECK( hipMalloc((void**) &dMp, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dMi, nnzM * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dMx, nnzM * sizeof(double)));
    cs = hipsolverRfAccessBundledFactorsDevice(handle, &nnzM, &dMp, &dMi, &dMx);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("nnzM: %d\n", nnzM);
    correct = nnzM == 15;
    int hMp[n + 1];
    int hMi[nnzM];
    double hMx[nnzM];
    int hMp_result[] = {0, 5, 10, 15};
    int hMi_result[] = {0, 1, 0, 1, 2, 0, 1, 0, 1, 2, 0, 1, 0, 1, 2};
    double hMx_result[] = {0.000000, 0.000000, 10.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000, 0.000000};
    CUDA_CHECK( hipMemcpy(hMp, dMp, (n + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(hMi, dMi, nnzM * sizeof(int), hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipMemcpy(hMx, dMx, nnzM * sizeof(double), hipMemcpyDeviceToHost) );
    //printArray(hMp, (n+1), "Mp");
    //printArray(hMi, nnzM, "Mi");
    //printArrayD(hMx, nnzM, "Mx");
    for (int i = 0; i < (n + 1); i++) {
        if (fabsf(hMp[i] - hMp_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzM; i++) {
        if (fabsf(hMi[i] - hMi_result[i]) > 0.001 || fabsf(hMx[i] - hMx_result[i]) > 0.001) {
            correct = 0;
            break;
        }
    }

    hipsolverRfMatrixFormat_t format;
    hipsolverRfUnitDiagonal_t diag;
    cs = hipsolverRfGetMatrixFormat(handle, &format, &diag);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("format: %d\n", format);
    correct = format == HIPSOLVERRF_MATRIX_FORMAT_CSR;
    //printf("diag: %d\n", diag);
    correct = diag == HIPSOLVERRF_UNIT_DIAGONAL_STORED_L;

    cs = hipsolverRfSetMatrixFormat(handle, HIPSOLVERRF_MATRIX_FORMAT_CSC, HIPSOLVERRF_UNIT_DIAGONAL_STORED_U);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfGetMatrixFormat(handle, &format, &diag);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    correct = format == HIPSOLVERRF_MATRIX_FORMAT_CSC;
    correct = diag == HIPSOLVERRF_UNIT_DIAGONAL_STORED_U;

    cs = hipsolverRfSetMatrixFormat(handle, HIPSOLVERRF_MATRIX_FORMAT_CSR, HIPSOLVERRF_UNIT_DIAGONAL_STORED_L);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    double zero;
    double boost;
    cs = hipsolverRfGetNumericProperties(handle, &zero, &boost);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("zero: %f\n", zero);
    correct = zero == 0;
    //printf("boost: %f\n", boost);
    correct = boost == 0;

    cs = hipsolverRfSetNumericProperties(handle, 1, 1);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfGetNumericProperties(handle, &zero, &boost);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    correct = zero == 1;
    correct = boost == 1;

    cs = hipsolverRfSetNumericProperties(handle, 0, 0);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    hipsolverRfNumericBoostReport_t report;
    cs = hipsolverRfGetNumericBoostReport(handle, &report);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("report: %d\n", report);
    correct = report == HIPSOLVERRF_NUMERIC_BOOST_NOT_USED;

    hipsolverRfResetValuesFastMode_t fastMode;
    cs = hipsolverRfGetResetValuesFastMode(handle, &fastMode);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("fastMode: %d\n", fastMode);
    correct = fastMode == HIPSOLVERRF_RESET_VALUES_FAST_MODE_OFF;

    cs = hipsolverRfSetResetValuesFastMode(handle, HIPSOLVERRF_RESET_VALUES_FAST_MODE_ON);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfGetResetValuesFastMode(handle, &fastMode);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    correct = fastMode == HIPSOLVERRF_RESET_VALUES_FAST_MODE_ON;

    cs = hipsolverRfSetResetValuesFastMode(handle, HIPSOLVERRF_RESET_VALUES_FAST_MODE_OFF);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    hipsolverRfFactorization_t fact_alg;
    hipsolverRfTriangularSolve_t solve_alg;
    cs = cusolverRfGetAlgs(handle, &fact_alg, &solve_alg);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    //printf("fact_alg: %d\n", fact_alg);
    correct = fact_alg == HIPSOLVERRF_FACTORIZATION_ALG0;
    //printf("solve_alg: %d\n", solve_alg);
    correct = solve_alg == HIPSOLVERRF_TRIANGULAR_SOLVE_ALG1;

    cs = hipsolverRfSetAlgs(handle, HIPSOLVERRF_FACTORIZATION_ALG1, HIPSOLVERRF_TRIANGULAR_SOLVE_ALG2);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = cusolverRfGetAlgs(handle, &fact_alg, &solve_alg);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }
    correct = fact_alg == HIPSOLVERRF_FACTORIZATION_ALG1;
    correct = solve_alg == HIPSOLVERRF_TRIANGULAR_SOLVE_ALG2;

    cs = hipsolverRfSetAlgs(handle, HIPSOLVERRF_FACTORIZATION_ALG0, HIPSOLVERRF_TRIANGULAR_SOLVE_ALG1);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    int nrhs = 3;
    int ldt = 3;
    int ldxf = 3;
    double *Temp, *XF;
    CUDA_CHECK( hipMalloc((void**) &Temp, ldt * nrhs * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &XF, ldxf * nrhs * sizeof(double)));
    cs = hipsolverRfSolve(handle, dP, dQ, 1, Temp, ldt, XF, ldxf);
    double hXF[ldxf * nrhs];
    CUDA_CHECK( hipMemcpy(hXF, XF, ldxf * nrhs * sizeof(double), hipMemcpyDeviceToHost) );

    double hCsrValA2[] = {20, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA2[] = {0, 3, 6, 9};
    const int hCsrColIndA2[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    int hP2[] = {2, 2, 2};
    int hQ2[] = {2, 2, 2};
    double *dCsrValA2;
    int *dCsrRowPtrA2, *dCsrColIndA2, *dP2, *dQ2;
    CUDA_CHECK( hipMalloc((void**) &dCsrValA2, nnzA * sizeof(double)));
    CUDA_CHECK( hipMalloc((void**) &dCsrRowPtrA2, (n + 1) * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dCsrColIndA2, nnzA * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dP2, n * sizeof(int)));
    CUDA_CHECK( hipMalloc((void**) &dQ2, n * sizeof(int)));
    CUDA_CHECK( hipMemcpy(dCsrValA2, hCsrValA2, nnzA * sizeof(double), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrRowPtrA2, hCsrRowPtrA2, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dCsrColIndA2, hCsrColIndA2, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dP2, hP2, n * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dQ2, hQ2, n * sizeof(int), hipMemcpyHostToDevice) );
    cs = hipsolverRfResetValues(n, nnzA, dCsrRowPtrA2, dCsrColIndA2, dCsrValA2, dP2, dQ2, handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    cs = hipsolverRfDestroy(handle);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    CUDA_CHECK(hipFree(dCsrValA));
    CUDA_CHECK(hipFree(dCsrRowPtrA));
    CUDA_CHECK(hipFree(dCsrColIndA));
    CUDA_CHECK(hipFree(dCsrValL));
    CUDA_CHECK(hipFree(dCsrRowPtrL));
    CUDA_CHECK(hipFree(dCsrColIndL));
    CUDA_CHECK(hipFree(dCsrValU));
    CUDA_CHECK(hipFree(dCsrRowPtrU));
    CUDA_CHECK(hipFree(dCsrColIndU));
    CUDA_CHECK(hipFree(dP));
    CUDA_CHECK(hipFree(dQ));
    CUDA_CHECK(hipFree(dCsrValA2));
    CUDA_CHECK(hipFree(dCsrRowPtrA2));
    CUDA_CHECK(hipFree(dCsrColIndA2));
    CUDA_CHECK(hipFree(dP2));
    CUDA_CHECK(hipFree(dQ2));

    if (correct == 1) {
        printf("rf_create_destroy test PASSED\n");
    } else {
        printf("rf_create_destroy test FAILED\n");
    }

    return EXIT_SUCCESS;
}
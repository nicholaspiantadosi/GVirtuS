
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int n = 3;
    int nnzA = 9;
    double hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    int P_result[] = {0, 1, 2};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int P[n];
    int numnz;

    int correct = 1;
    cs = cusolverSpDcsrzfdHost(handle, n, nnzA, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, P, &numnz);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", numnz);
    correct = numnz == 3;

    for (int i = 0; i < n; i++) {
        printf("%d\n", P[i]);
        if (fabsf(P[i] - P_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrzfd test PASSED\n");
    } else {
        printf("spcsrzfd test FAILED\n");
    }

    return EXIT_SUCCESS;
}
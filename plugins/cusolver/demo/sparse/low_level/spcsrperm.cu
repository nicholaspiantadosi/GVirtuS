
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int n = 3;
    int nnzA = 9;
    int hCsrRowPtrA[] = {0, 3, 6, 9};
    int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    int p[] = {0, 1, 2};
    int q[] = {0, 1, 2};

    int hCsrRowPtrA_result[] = {0, 3, 6, 9};
    int hCsrColIndA_result[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t bufferSizeInBytes;

    int correct = 1;
    cs = cusolverSpXcsrperm_bufferSizeHost(handle, m, n, nnzA, descrA, hCsrRowPtrA, hCsrColIndA, p, q, &bufferSizeInBytes);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    size_t pBuffer[bufferSizeInBytes];
    int map[nnzA];

    cs = cusolverSpXcsrpermHost(handle, m, n, nnzA, descrA, hCsrRowPtrA, hCsrColIndA, p, q, map, pBuffer);

    for (int i = 0; i < m+1; i++) {
        printf("%d\n", hCsrRowPtrA[i]);
        if (fabsf(hCsrRowPtrA[i] - hCsrRowPtrA_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < nnzA; i++) {
        printf("%d\n", hCsrColIndA[i]);
        if (fabsf(hCsrColIndA[i] - hCsrColIndA_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrperm test PASSED\n");
    } else {
        printf("spcsrperm test FAILED\n");
    }

    return EXIT_SUCCESS;
}
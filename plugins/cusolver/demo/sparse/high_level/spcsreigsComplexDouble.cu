
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int nnz = 9;
    hipDoubleComplex hCsrValA[] = {make_hipDoubleComplex(10, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(-6, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    hipDoubleComplex left_bottom_corner = make_hipDoubleComplex(0, 0);
    hipDoubleComplex right_upper_corner = make_hipDoubleComplex(10, 10);

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int num_eigs;

    int correct = 1;
    cs = cusolverSpZcsreigsHost(handle, m, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, left_bottom_corner, right_upper_corner, &num_eigs);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", num_eigs);
    correct = num_eigs == 1;

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsreigs test PASSED\n");
    } else {
        printf("spcsreigs test FAILED\n");
    }

    return EXIT_SUCCESS;
}
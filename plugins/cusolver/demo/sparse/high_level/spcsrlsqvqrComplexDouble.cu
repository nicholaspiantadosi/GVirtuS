
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int n = 3;
    int nnz = 9;
    hipDoubleComplex hCsrValA[] = {make_hipDoubleComplex(10, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(-6, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipDoubleComplex b[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0)};

    hipDoubleComplex x_result[] = {make_hipDoubleComplex(0.097473, 0), make_hipDoubleComplex(0.155235, 0), make_hipDoubleComplex(-0.014440, 0)};
    int p_result[] = {0, 1, 2};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipDoubleComplex x[n];
    int p[n];
    int rankA;
    double min_norm;

    int correct = 1;
    cs = cusolverSpZcsrlsqvqrHost(handle, m, n, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, &rankA, x, p, &min_norm);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", rankA);
    correct = rankA == 3;
    correct = min_norm == 0;

    for (int i = 0; i < n; i++) {
        printf("%f \t %d\n", x[i].x, p[i]);
        if (fabsf(x[i].x - x_result[i].x) > 0.01 || fabsf(p[i] - p_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsqvqr test PASSED\n");
    } else {
        printf("spcsrlsqvqr test FAILED\n");
    }

    return EXIT_SUCCESS;
}
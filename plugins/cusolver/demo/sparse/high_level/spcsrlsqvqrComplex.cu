
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int n = 3;
    int nnz = 9;
    hipComplex hCsrValA[] = {make_hipComplex(10, 0), make_hipComplex(1, 0), make_hipComplex(9, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(-6, 0), make_hipComplex(1, 0), make_hipComplex(6, 0), make_hipComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipComplex b[] = {make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0)};

    hipComplex x_result[] = {make_hipComplex(0.097473, 0), make_hipComplex(0.155235, 0), make_hipComplex(-0.014440, 0)};
    int p_result[] = {0, 1, 2};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipComplex x[n];
    int p[n];
    int rankA;
    float min_norm;

    int correct = 1;
    cs = cusolverSpCcsrlsqvqrHost(handle, m, n, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, &rankA, x, p, &min_norm);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", rankA);
    correct = rankA == 3;
    correct = min_norm == 0;

    for (int i = 0; i < n; i++) {
        printf("%f \t %d\n", x[i].x, p[i]);
        if (fabsf(x[i].x - x_result[i].x) > 0.01 || fabsf(p[i] - p_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsqvqr test PASSED\n");
    } else {
        printf("spcsrlsqvqr test FAILED\n");
    }

    return EXIT_SUCCESS;
}
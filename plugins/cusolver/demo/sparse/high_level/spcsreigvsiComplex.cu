
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int nnz = 9;
    hipComplex hCsrValA[] = {make_hipComplex(10, 0), make_hipComplex(1, 0), make_hipComplex(9, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(-6, 0), make_hipComplex(1, 0), make_hipComplex(6, 0), make_hipComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    const hipComplex x0[] = {make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0)};

    hipComplex x_result[] = {make_hipComplex(0.512307, 0), make_hipComplex(-0.048724, 0), make_hipComplex(-0.857419, 0)};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipComplex x[m];
    hipComplex mu;

    int correct = 1;
    cs = cusolverSpCcsreigvsiHost(handle, m, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, make_hipComplex(1, 0), x0, 50, 1, &mu, x);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%f\n", mu.x);
    correct = fabsf(mu.x - 3.934861) < 0.01;

    for (int i = 0; i < m; i++) {
        printf("%f\n", x[i].x);
        if (fabsf(x[i].x - x_result[i].x) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsreigvsi test PASSED\n");
    } else {
        printf("spcsreigvsi test FAILED\n");
    }

    return EXIT_SUCCESS;
}
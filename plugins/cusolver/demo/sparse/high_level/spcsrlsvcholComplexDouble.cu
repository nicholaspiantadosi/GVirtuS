
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int nnz = 9;
    hipDoubleComplex hCsrValA[] = {make_hipDoubleComplex(10, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(-6, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipDoubleComplex b[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0)};

    hipDoubleComplex x_result[] = {make_hipDoubleComplex(0.011885, 0), make_hipDoubleComplex(0.308756, 0), make_hipDoubleComplex(-0.045113, 0)};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipDoubleComplex x[m];
    int singularity;

    int correct = 1;
    cs = cusolverSpZcsrlsvcholHost(handle, m, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, 0, x, &singularity);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", singularity);
    correct = singularity == 2;

    for (int i = 0; i < m; i++) {
        printf("%f\n", x[i].x);
        if (fabsf(x[i].x - x_result[i].x) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsvchol test PASSED\n");
    } else {
        printf("spcsrlsvchol test FAILED\n");
    }

    return EXIT_SUCCESS;
}
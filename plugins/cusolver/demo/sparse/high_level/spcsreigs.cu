
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int nnz = 9;
    float hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};
    hipComplex left_bottom_corner = make_hipComplex(0, 0);
    hipComplex right_upper_corner = make_hipComplex(10, 10);

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int num_eigs;

    int correct = 1;
    cs = cusolverSpScsreigsHost(handle, m, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, left_bottom_corner, right_upper_corner, &num_eigs);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", num_eigs);
    correct = num_eigs == 1;

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsreigs test PASSED\n");
    } else {
        printf("spcsreigs test FAILED\n");
    }

    return EXIT_SUCCESS;
}
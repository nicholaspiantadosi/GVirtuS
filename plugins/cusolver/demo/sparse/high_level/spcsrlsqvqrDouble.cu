
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int n = 3;
    int nnz = 9;
    double hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    double b[] = {1, 1, 1};

    double x_result[] = {0.097473, 0.155235, -0.014440};
    int p_result[] = {0, 1, 2};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    double x[n];
    int p[n];
    int rankA;
    double min_norm;

    int correct = 1;
    cs = cusolverSpDcsrlsqvqrHost(handle, m, n, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, &rankA, x, p, &min_norm);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", rankA);
    correct = rankA == 3;
    correct = min_norm == 0;

    for (int i = 0; i < n; i++) {
        printf("%f \t %d\n", x[i], p[i]);
        if (fabsf(x[i] - x_result[i]) > 0.01 || fabsf(p[i] - p_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsqvqr test PASSED\n");
    } else {
        printf("spcsrlsqvqr test FAILED\n");
    }

    return EXIT_SUCCESS;
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int n = 3;
    int nnzA = 9;
    hipDoubleComplex hCsrValA[] = {make_hipDoubleComplex(10, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(-6, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipDoubleComplex b[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0)};

    hipDoubleComplex x_result[] = {make_hipDoubleComplex(0.097473, 0), make_hipDoubleComplex(0.155235, 0), make_hipDoubleComplex(-0.014440, 0)};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipDoubleComplex x[n];
    int singularity;

    int correct = 1;
    cs = cusolverSpZcsrlsvluHost(handle, n, nnzA, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, 0, x, &singularity);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", singularity);
    correct = singularity == -1;

    for (int i = 0; i < n; i++) {
        printf("%f\n", x[i].x);
        if (fabsf(x[i].x - x_result[i].x) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsvlu test PASSED\n");
    } else {
        printf("spcsrlsvlu test FAILED\n");
    }

    return EXIT_SUCCESS;
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int m = 3;
    int nnz = 9;
    float hCsrValA[] = {10, 1, 9, 3, 4, -6, 1, 6, 2};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    float b[] = {1, 1, 1};

    float x_result[] = {0.011885, 0.308756, -0.045113};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    float x[m];
    int singularity;

    int correct = 1;
    cs = hipsolverSpScsrlsvcholHost(handle, m, nnz, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, 0, x, &singularity);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", singularity);
    correct = singularity == 2;

    for (int i = 0; i < m; i++) {
        printf("%f\n", x[i]);
        if (fabsf(x[i] - x_result[i]) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsvchol test PASSED\n");
    } else {
        printf("spcsrlsvchol test FAILED\n");
    }

    return EXIT_SUCCESS;
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsolver.h>         // cusolverSp
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipsparse.h>

int main(void) {

    int n = 3;
    int nnzA = 9;
    hipComplex hCsrValA[] = {make_hipComplex(10, 0), make_hipComplex(1, 0), make_hipComplex(9, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(-6, 0), make_hipComplex(1, 0), make_hipComplex(6, 0), make_hipComplex(2, 0)};
    const int hCsrRowPtrA[] = {0, 3, 6, 9};
    const int hCsrColIndA[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    hipComplex b[] = {1, 1, 1};

    hipComplex x_result[] = {make_hipComplex(0.173285, 0), make_hipComplex(-0.001805, 0), make_hipComplex(-0.081227, 0)};

    hipsolverSpHandle_t handle = NULL;
    hipsolverStatus_t cs = hipsolverSpCreate(&handle);

    hipsparseMatDescr_t descrA = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    hipComplex x[n];
    int singularity;

    int correct = 1;
    cs = cusolverSpCcsrlsvluHost(handle, n, nnzA, descrA, hCsrValA, hCsrRowPtrA, hCsrColIndA, b, 1, 0, x, &singularity);
    if (cs != HIPSOLVER_STATUS_SUCCESS) {
        correct = 0;
    }

    printf("%d\n", singularity);
    correct = singularity == -1;

    for (int i = 0; i < n; i++) {
        printf("%f\n", x[i].x);
        if (fabsf(x[i].x - x_result[i].x) > 0.01) {
            correct = 0;
            break;
        }
    }

    hipsolverSpDestroy(handle);

    if (correct == 1) {
        printf("spcsrlsvlu test PASSED\n");
    } else {
        printf("spcsrlsvlu test FAILED\n");
    }

    return EXIT_SUCCESS;
}
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 3;
    const int nnz = 7;
    double fractionToColor = 0.8;

    double hCsrValA[] = {1, 1, -1, 1, 2, -1, 5};
    int hCsrRowPtrA[] = {0, 3, 5, 7};
    int hCsrColIndA[] = {0, 1, 2, 0, 1, 0, 2};

    int hColoring[m];
    int hReordering[] = {0, 0, 0};

    int hColoring_result[] = {3, 2, 4};
    int hReordering_result[] = {1, 0, 2};

    // Device memory management

    double *dCsrValA;
    int *dCsrRowPtrA, *dCsrColIndA;
    int ncolors;
    int *dColoring, *dReordering;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dColoring, m * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dReordering, m * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dReordering, hReordering, m * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseColorInfo_t info = 0;
    hipsparseCreateColorInfo(&info);

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    //hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    //hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);

    hipsparseDcsrcolor(handle, m, nnz, descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA, &fractionToColor, &ncolors, dColoring, dReordering, info);

    // device result check
    CHECK_CUDA( hipMemcpy(hColoring, dColoring, m * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hReordering, dReordering, m * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if(hColoring[i] != hColoring_result[i]) {
            correct = 0;
            break;
        }
        if(hReordering[i] != hReordering_result[i]) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csrcolor test PASSED\n");
    else
        printf("csrcolor test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyColorInfo(info));
    CHECK_CUDA(hipFree(dCsrValA));
    CHECK_CUDA(hipFree(dCsrRowPtrA));
    CHECK_CUDA(hipFree(dCsrColIndA));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
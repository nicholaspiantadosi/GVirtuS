#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[]) {

    // Host problem definition

    int m = 4;
    int n = 4;
    int k = 4;
    double alpha = 1;
    double beta = 1;

    int nnzA = 9;
    double hCsrValA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    int nnzB = 8;
    double hCsrValB[] = {1, 2, 3, 4, 5, 6, 7, 8};
    int hCsrRowPtrB[] = {0, 3, 4, 6, 8};
    int hCsrColIndB[] = {0, 1, 3, 3, 2, 3, 2, 3};

    int nnzD = 7;
    double hCsrValD[] = {1, 2, 3, 4, 5, 6, 7};
    int hCsrRowPtrD[] = {0, 1, 4, 5, 7};
    int hCsrColIndD[] = {0, 0, 2, 3, 2, 2, 3};

    double hCsrValC_result[] = {2, 2, 31, 39, 2, 3, 20, 5, 10, 84, 107, 69, 111};
    int hCsrRowPtrC_result[] = {0, 4, 7, 11, 13};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 0, 2, 3, 0, 1, 2, 3, 2, 3};

    // Device memory management

    double *dCsrValA, *dCsrValB, *dCsrValC, *dCsrValD;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrB, *dCsrColIndB, *dCsrRowPtrC, *dCsrColIndC, *dCsrRowPtrD, *dCsrColIndD;

    dCsrValC = NULL;
    dCsrRowPtrC = NULL;
    dCsrColIndC = NULL;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA, nnzA * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnzA * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrValB, nnzB * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrB, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndB, nnzB * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrValD, nnzD * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrD, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndD, nnzD * sizeof(int)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValB, hCsrValB, nnzB * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrB, hCsrRowPtrB, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndB, hCsrColIndB, nnzB * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValD, hCsrValD, nnzD * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrD, hCsrRowPtrD, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndD, hCsrColIndD, nnzD * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrB = 0;
    hipsparseCreateMatDescr(&descrB);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrD = 0;
    hipsparseCreateMatDescr(&descrD);
    hipsparseSetMatType(descrD, HIPSPARSE_MATRIX_TYPE_GENERAL );

    int baseC, nnzC;
    csrgemm2Info_t info = NULL;
    /* alpha, nnzTotalDevHostPtr points to host memory */
    size_t bufferSize;
    char *buffer = NULL;
    int *nnzTotalDevHostPtr = &nnzC;

    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    hipsparseCreateCsrgemm2Info(&info);

    hipsparseDcsrgemm2_bufferSizeExt(handle, m, n, k, &alpha,
                                    descrA, nnzA, dCsrRowPtrA, dCsrColIndA,
                                    descrB, nnzB, dCsrRowPtrB, dCsrColIndB,
                                    &beta,
                                    descrD, nnzD, dCsrRowPtrD, dCsrColIndD,
                                    info,
                                    &bufferSize);

    hipMalloc(&buffer, bufferSize);
    hipMalloc((void**)&dCsrRowPtrC, sizeof(int)*(m+1));

    hipsparseXcsrgemm2Nnz(handle, m, n, k,
                         descrA, nnzA, dCsrRowPtrA, dCsrColIndA,
                         descrB, nnzB, dCsrRowPtrB, dCsrColIndB,
                         descrD, nnzD, dCsrRowPtrD, dCsrColIndD,
                         descrC, dCsrRowPtrC, nnzTotalDevHostPtr,
                         info, buffer );

    if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzC, dCsrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, dCsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    hipMalloc((void**)&dCsrColIndC, sizeof(int)*nnzC);
    hipMalloc((void**)&dCsrValC, sizeof(double)*nnzC);

    hipsparseDcsrgemm2(handle, m, n, k, &alpha,
                      descrA, nnzA, dCsrValA, dCsrRowPtrA, dCsrColIndA,
                      descrB, nnzB, dCsrValB, dCsrRowPtrB, dCsrColIndB,
                      &beta,
                      descrD, nnzD, dCsrValD, dCsrRowPtrD, dCsrColIndD,
                      descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC,
                      info, buffer);

    hipsparseDestroyCsrgemm2Info(info);

    // device result check

    double hCsrValC[nnzC];
    int hCsrRowPtrC[m+1];
    int hCsrColIndC[nnzC];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzC * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzC * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;

    for (int i = 0; i < nnzC; i++) {
        if(hCsrColIndC[i] != hCsrColIndC_result[i]) {
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < (m + 1); i++) {
        if(hCsrRowPtrC[i] != hCsrRowPtrC_result[i]) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csrgeam2 test PASSED\n");
    else
        printf("csrgeam2 test FAILED: wrong result\n");

    // destroy

    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation

    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dCsrValB) );
    CHECK_CUDA( hipFree(dCsrRowPtrB) );
    CHECK_CUDA( hipFree(dCsrColIndB) );
    CHECK_CUDA( hipFree(dCsrValC) );
    CHECK_CUDA( hipFree(dCsrRowPtrC) );
    CHECK_CUDA( hipFree(dCsrColIndC) );
    CHECK_CUDA( hipFree(dCsrValD) );
    CHECK_CUDA( hipFree(dCsrRowPtrD) );
    CHECK_CUDA( hipFree(dCsrColIndD) );

    return EXIT_SUCCESS;
}
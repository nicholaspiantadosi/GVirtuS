#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[]) {

    // Host problem definition

    int m = 4;
    int n = 5;
    float alpha = 1;
    float beta = 1;

    int nnzA = 9;
    float hCsrValA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    int nnzB = 8;
    float hCsrValB[] = {1, 2, 3, 4, 5, 6, 7, 8};
    int hCsrRowPtrB[] = {0, 3, 4, 6, 8};
    int hCsrColIndB[] = {0, 1, 3, 3, 2, 3, 2, 3};

    float hCsrValC_result[] = {2, 2, 2, 6, 4, 4, 5, 11, 13, 8, 7, 17};
    int hCsrRowPtrC_result[] = {0, 4, 6, 9, 12};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 1, 3, 0, 2, 3, 1, 2, 3};

    // Device memory management

    float *dCsrValA, *dCsrValB, *dCsrValC;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrB, *dCsrColIndB, *dCsrRowPtrC, *dCsrColIndC;

    dCsrValC = NULL;
    dCsrRowPtrC = NULL;
    dCsrColIndC = NULL;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA, nnzA * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnzA * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrValB, nnzB * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrB, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndB, nnzB * sizeof(int)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValB, hCsrValB, nnzB * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrB, hCsrRowPtrB, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndB, hCsrColIndB, nnzB * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrB = 0;
    hipsparseCreateMatDescr(&descrB);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    int baseC, nnzC;
    /* alpha, nnzTotalDevHostPtr points to host memory */
    size_t bufferSizeInBytes;
    char *buffer = NULL;
    int *nnzTotalDevHostPtr = &nnzC;
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
    hipMalloc((void**)&dCsrRowPtrC, sizeof(int)*(m+1));
    hipsparseScsrgeam2_bufferSizeExt(handle, m, n, &alpha,
                                    descrA, nnzA, dCsrValA, dCsrRowPtrA, dCsrColIndA, &beta,
                                    descrB, nnzB, dCsrValB, dCsrRowPtrB, dCsrColIndB,
                                    descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC,
                                    &bufferSizeInBytes);
    hipMalloc((void**)&buffer, sizeof(char)*bufferSizeInBytes);
    hipsparseXcsrgeam2Nnz(handle, m, n,
                         descrA, nnzA, dCsrRowPtrA, dCsrColIndA,
                         descrB, nnzB, dCsrRowPtrB, dCsrColIndB,
                         descrC, dCsrRowPtrC, nnzTotalDevHostPtr,
                         buffer);
    if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzC, dCsrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, dCsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }
    hipMalloc((void**)&dCsrColIndC, sizeof(int)*nnzC);
    hipMalloc((void**)&dCsrValC, sizeof(float)*nnzC);
    hipsparseScsrgeam2(handle, m, n, &alpha, descrA, nnzA, dCsrValA, dCsrRowPtrA, dCsrColIndA, &beta, descrB, nnzB, dCsrValB, dCsrRowPtrB, dCsrColIndB,
                      descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC, buffer);

    // device result check

    float hCsrValC[nnzC];
    int hCsrRowPtrC[m+1];
    int hCsrColIndC[nnzC];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzC * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzC * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;

    for (int i = 0; i < nnzC; i++) {
        if((fabs(hCsrValC[i] - hCsrValC_result[i]) > 0.000001) ||
                hCsrColIndC[i] != hCsrColIndC_result[i]) {
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < (m + 1); i++) {
        if(hCsrRowPtrC[i] != hCsrRowPtrC_result[i]) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csrgeam2 test PASSED\n");
    else
        printf("csrgeam2 test FAILED: wrong result\n");

    // destroy

    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation

    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dCsrValB) );
    CHECK_CUDA( hipFree(dCsrRowPtrB) );
    CHECK_CUDA( hipFree(dCsrColIndB) );
    CHECK_CUDA( hipFree(dCsrValC) );
    CHECK_CUDA( hipFree(dCsrRowPtrC) );
    CHECK_CUDA( hipFree(dCsrColIndC) );

    return EXIT_SUCCESS;
}
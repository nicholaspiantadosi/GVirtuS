#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[]) {

    // Host problem definition

    int m = 4;
    int n = 4;
    int k = 4;
    hipComplex alpha = make_hipComplex(1, 0);
    hipComplex beta = make_hipComplex(1, 0);

    int nnzA = 9;
    hipComplex hCsrValA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0), make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(9, 0)};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    int nnzB = 8;
    hipComplex hCsrValB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0), make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(8, 0)};
    int hCsrRowPtrB[] = {0, 3, 4, 6, 8};
    int hCsrColIndB[] = {0, 1, 3, 3, 2, 3, 2, 3};

    int nnzD = 7;
    hipComplex hCsrValD[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0), make_hipComplex(6, 0), make_hipComplex(7, 0)};
    int hCsrRowPtrD[] = {0, 1, 4, 5, 7};
    int hCsrColIndD[] = {0, 0, 2, 3, 2, 2, 3};

    hipComplex hCsrValC_result[] = {make_hipComplex(2, 0), make_hipComplex(2, 0), make_hipComplex(31, 0), make_hipComplex(39, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(20, 0), make_hipComplex(5, 0), make_hipComplex(10, 0), make_hipComplex(84, 0), make_hipComplex(107, 0), make_hipComplex(69, 0), make_hipComplex(111, 0)};
    int hCsrRowPtrC_result[] = {0, 4, 7, 11, 13};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 0, 2, 3, 0, 1, 2, 3, 2, 3};

    // Device memory management

    hipComplex *dCsrValA, *dCsrValB, *dCsrValC, *dCsrValD;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrB, *dCsrColIndB, *dCsrRowPtrC, *dCsrColIndC, *dCsrRowPtrD, *dCsrColIndD;

    dCsrValC = NULL;
    dCsrRowPtrC = NULL;
    dCsrColIndC = NULL;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA, nnzA * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnzA * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrValB, nnzB * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrB, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndB, nnzB * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrValD, nnzD * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrD, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndD, nnzD * sizeof(int)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValB, hCsrValB, nnzB * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrB, hCsrRowPtrB, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndB, hCsrColIndB, nnzB * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValD, hCsrValD, nnzD * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrD, hCsrRowPtrD, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndD, hCsrColIndD, nnzD * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrB = 0;
    hipsparseCreateMatDescr(&descrB);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrD = 0;
    hipsparseCreateMatDescr(&descrD);
    hipsparseSetMatType(descrD, HIPSPARSE_MATRIX_TYPE_GENERAL );

    int baseC, nnzC;
    csrgemm2Info_t info = NULL;
    /* alpha, nnzTotalDevHostPtr points to host memory */
    size_t bufferSize;
    char *buffer = NULL;
    int *nnzTotalDevHostPtr = &nnzC;

    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    hipsparseCreateCsrgemm2Info(&info);

    hipsparseCcsrgemm2_bufferSizeExt(handle, m, n, k, &alpha,
                                    descrA, nnzA, dCsrRowPtrA, dCsrColIndA,
                                    descrB, nnzB, dCsrRowPtrB, dCsrColIndB,
                                    &beta,
                                    descrD, nnzD, dCsrRowPtrD, dCsrColIndD,
                                    info,
                                    &bufferSize);

    hipMalloc(&buffer, bufferSize);
    hipMalloc((void**)&dCsrRowPtrC, sizeof(int)*(m+1));

    hipsparseXcsrgemm2Nnz(handle, m, n, k,
                         descrA, nnzA, dCsrRowPtrA, dCsrColIndA,
                         descrB, nnzB, dCsrRowPtrB, dCsrColIndB,
                         descrD, nnzD, dCsrRowPtrD, dCsrColIndD,
                         descrC, dCsrRowPtrC, nnzTotalDevHostPtr,
                         info, buffer );

    if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzC, dCsrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, dCsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    hipMalloc((void**)&dCsrColIndC, sizeof(int)*nnzC);
    hipMalloc((void**)&dCsrValC, sizeof(hipComplex)*nnzC);

    hipsparseCcsrgemm2(handle, m, n, k, &alpha,
                      descrA, nnzA, dCsrValA, dCsrRowPtrA, dCsrColIndA,
                      descrB, nnzB, dCsrValB, dCsrRowPtrB, dCsrColIndB,
                      &beta,
                      descrD, nnzD, dCsrValD, dCsrRowPtrD, dCsrColIndD,
                      descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC,
                      info, buffer);

    hipsparseDestroyCsrgemm2Info(info);

    // device result check

    hipComplex hCsrValC[nnzC];
    int hCsrRowPtrC[m+1];
    int hCsrColIndC[nnzC];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzC * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzC * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;

    for (int i = 0; i < nnzC; i++) {
        if(hCsrColIndC[i] != hCsrColIndC_result[i]) {
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < (m + 1); i++) {
        if(hCsrRowPtrC[i] != hCsrRowPtrC_result[i]) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csrgeam2 test PASSED\n");
    else
        printf("csrgeam2 test FAILED: wrong result\n");

    // destroy

    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation

    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dCsrValB) );
    CHECK_CUDA( hipFree(dCsrRowPtrB) );
    CHECK_CUDA( hipFree(dCsrColIndB) );
    CHECK_CUDA( hipFree(dCsrValC) );
    CHECK_CUDA( hipFree(dCsrRowPtrC) );
    CHECK_CUDA( hipFree(dCsrColIndC) );
    CHECK_CUDA( hipFree(dCsrValD) );
    CHECK_CUDA( hipFree(dCsrRowPtrD) );
    CHECK_CUDA( hipFree(dCsrColIndD) );

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int batchStride = m;

    hipComplex hdl[] = {make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(1, 0)};
    hipComplex hd[] = {make_hipComplex(1, 0), make_hipComplex(4, 0), make_hipComplex(6, 0), make_hipComplex(9, 0), make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(1, 0)};
    hipComplex hdu[] = {make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(7, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(3, 0), make_hipComplex(1, 0), make_hipComplex(1, 0)};

    hipComplex hx[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(3, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(4, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(7, 0), make_hipComplex(8, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0),
                      make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(3, 0), make_hipComplex(4, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(1, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(6, 0), make_hipComplex(7, 0)};

    hipComplex hx_result[] = {make_hipComplex(0.553030, 0), make_hipComplex(0.446970, 0), make_hipComplex(-0.340909, 0), make_hipComplex(0.371212, 0),
                             make_hipComplex(1.600000, 0), make_hipComplex(-0.600000, 0), make_hipComplex(0.200000, 0), make_hipComplex(3.800000, 0),
                             make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(5.000000, 0), make_hipComplex(6.000000, 0),
                             make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(7.000000, 0), make_hipComplex(8.000000, 0),
                             make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0),
                             make_hipComplex(2.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(3.000000, 0), make_hipComplex(4.000000, 0),
                             make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(5.000000, 0), make_hipComplex(1.000000, 0),
                             make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(6.000000, 0), make_hipComplex(7.000000, 0)};

    // Device memory management
    hipComplex *ddl, *dd, *ddu, *dx;

    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(hipComplex)) );

    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseCgtsv2StridedBatch_bufferSizeExt(handle, m, ddl, dd, ddu, dx, batchCount, batchStride, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseCgtsv2StridedBatch(handle, m, ddl, dd, ddu, dx, batchCount, batchStride, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i].x - hx_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gtsv2Batched test PASSED\n");
    else
        printf("gtsv2Batched test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
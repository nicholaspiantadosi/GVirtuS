#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int algo = 0;

    hipComplex hds[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(8, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(7, 0)};
    hipComplex hdl[] = {make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(1, 0)};
    hipComplex hd[] = {make_hipComplex(1, 0), make_hipComplex(4, 0), make_hipComplex(6, 0), make_hipComplex(9, 0), make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(1, 0)};
    hipComplex hdu[] = {make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(7, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(3, 0), make_hipComplex(1, 0), make_hipComplex(0, 0)};
    hipComplex hdw[] = {make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex hx[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(3, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(4, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(7, 0), make_hipComplex(8, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(1, 0),
                      make_hipComplex(2, 0), make_hipComplex(1, 0), make_hipComplex(3, 0), make_hipComplex(4, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(1, 0),
                      make_hipComplex(1, 0), make_hipComplex(1, 0), make_hipComplex(6, 0), make_hipComplex(7, 0)};

    hipComplex hx_result[] = {make_hipComplex(2.000000, 0), make_hipComplex(-0.436364, 0), make_hipComplex(1.000000, 0), make_hipComplex(0.381818, 0),
                            make_hipComplex(-1.000000, 0), make_hipComplex(3.363636, 0), make_hipComplex(2.000000, 0), make_hipComplex(-2.036364, 0),
                            make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(5.000000, 0), make_hipComplex(6.000000, 0),
                            make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(7.000000, 0), make_hipComplex(8.000000, 0),
                            make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0),
                            make_hipComplex(2.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(3.000000, 0), make_hipComplex(4.000000, 0),
                            make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(5.000000, 0), make_hipComplex(1.000000, 0),
                            make_hipComplex(1.000000, 0), make_hipComplex(1.000000, 0), make_hipComplex(6.000000, 0), make_hipComplex(7.000000, 0)};

    // Device memory management
    hipComplex *ddl, *dd, *ddu, *dx, *dds, *ddw;

    CHECK_CUDA( hipMalloc((void**) &dds,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddw,  m * batchCount * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(hipComplex)) );

    CHECK_CUDA( hipMemcpy(dds, hds, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddw, hdw, m * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseCgpsvInterleavedBatch_bufferSizeExt(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseCgpsvInterleavedBatch(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i].x - hx_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gpsvBatch test PASSED\n");
    else
        printf("gpsvBatch test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dds));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));
    CHECK_CUDA(hipFree(ddw));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
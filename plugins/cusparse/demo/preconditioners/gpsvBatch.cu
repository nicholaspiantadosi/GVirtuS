#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int algo = 0;

    float hds[] = {0, 0, 5, 8, 0, 0, 4, 7};
    float hdl[] = {0, 1, 1, 1, 0, 1, 5, 1};
    float hd[] = {1, 4, 6, 9, 1, 2, 1, 1};
    float hdu[] = {1, 1, 7, 0, 1, 3, 1, 0};
    float hdw[] = {2, 1, 0, 0, 1, 1, 0, 0};

    float hx[] = {1, 2, 1, 3,
                  1, 1, 1, 4,
                  1, 1, 5, 6,
                  1, 1, 7, 8,
                  1, 1, 1, 1,
                  2, 1, 3, 4,
                  1, 1, 5, 1,
                  1, 1, 6, 7};

    float hx_result[] = {2.000000, -0.436364, 1.000000, 0.381818,
                         -1.000000, 3.363636, 2.000000, -2.036364,
                         1.000000, 1.000000, 5.000000, 6.000000,
                         1.000000, 1.000000, 7.000000, 8.000000,
                         1.000000, 1.000000, 1.000000, 1.000000,
                         2.000000, 1.000000, 3.000000, 4.000000,
                         1.000000, 1.000000, 5.000000, 1.000000,
                         1.000000, 1.000000, 6.000000, 7.000000};

    // Device memory management
    float *ddl, *dd, *ddu, *dx, *dds, *ddw;

    CHECK_CUDA( hipMalloc((void**) &dds,  m * batchCount * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &ddw,  m * batchCount * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(float)) );

    CHECK_CUDA( hipMemcpy(dds, hds, m * batchCount * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddw, hdw, m * batchCount * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseSgpsvInterleavedBatch_bufferSizeExt(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseSgpsvInterleavedBatch(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i] - hx_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gpsvBatch test PASSED\n");
    else
        printf("gpsvBatch test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dds));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));
    CHECK_CUDA(hipFree(ddw));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
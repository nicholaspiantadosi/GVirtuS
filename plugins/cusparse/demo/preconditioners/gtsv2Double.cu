#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;

    double hdl[] = {0, 0, 0, 0};
    double hd[] = {1, 4, 6, 9};
    double hdu[] = {0, 0, 7, 0};

    int ldb = 4;
    double hB[] = {1, 2, 0, 3,
                  0, 0, 0, 4,
                  0, 0, 5, 6,
                  0, 0, 7, 8};

    double hB_result[] = {1.000000, 0.500000, -0.388889, 0.333333,
                         0.000000, 0.000000, -0.518519, 0.444444,
                         0.000000, 0.000000, 0.055556, 0.666667,
                         0.000000, 0.000000, 0.129630, 0.888889};

    // Device memory management
    double *ddl, *dd, *ddu, *dB;

    CHECK_CUDA( hipMalloc((void**) &ddl,  m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dB, ldb * n * sizeof(double)) );

    CHECK_CUDA( hipMemcpy(ddl, hdl, m * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * n * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseDgtsv2_bufferSizeExt(handle, m, n, ddl, dd, ddu, dB, ldb, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseDgtsv2(handle, m, n, ddl, dd, ddu, dB, ldb, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hB, dB, ldb * n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (ldb * n); i++) {
        if((fabs(hB[i] - hB_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gtsv2 test PASSED\n");
    else
        printf("gtsv2 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
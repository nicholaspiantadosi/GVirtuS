#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int algo = 0;

    double hds[] = {0, 0, 5, 8, 0, 0, 4, 7};
    double hdl[] = {0, 1, 1, 1, 0, 1, 5, 1};
    double hd[] = {1, 4, 6, 9, 1, 2, 1, 1};
    double hdu[] = {1, 1, 7, 0, 1, 3, 1, 0};
    double hdw[] = {2, 1, 0, 0, 1, 1, 0, 0};

    double hx[] = {1, 2, 1, 3,
                  1, 1, 1, 4,
                  1, 1, 5, 6,
                  1, 1, 7, 8,
                  1, 1, 1, 1,
                  2, 1, 3, 4,
                  1, 1, 5, 1,
                  1, 1, 6, 7};

    double hx_result[] = {1.000000, -0.436364, -0.000000, 0.381818,
                          -0.000000, 3.363636, 1.000000, -2.036364,
                          1.000000, 1.000000, 5.000000, 6.000000,
                          1.000000, 1.000000, 7.000000, 8.000000,
                          1.000000, 1.000000, 1.000000, 1.000000,
                          2.000000, 1.000000, 3.000000, 4.000000,
                          1.000000, 1.000000, 5.000000, 1.000000,
                          1.000000, 1.000000, 6.000000, 7.000000};

    // Device memory management
    double *ddl, *dd, *ddu, *dx, *dds, *ddw;

    CHECK_CUDA( hipMalloc((void**) &dds,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &ddw,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(double)) );

    CHECK_CUDA( hipMemcpy(dds, hds, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddw, hdw, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseDgpsvInterleavedBatch_bufferSizeExt(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseDgpsvInterleavedBatch(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i] - hx_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gpsvBatch test PASSED\n");
    else
        printf("gpsvBatch test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dds));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));
    CHECK_CUDA(hipFree(ddw));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
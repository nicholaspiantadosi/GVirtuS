#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;

    hipDoubleComplex hdl[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};
    hipDoubleComplex hd[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(9, 0)};
    hipDoubleComplex hdu[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0)};

    int ldb = 4;
    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0)};

    hipDoubleComplex hB_result[] = {make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(0.500000, 0), make_hipDoubleComplex(-0.388889, 0), make_hipDoubleComplex(0.333333, 0),
                             make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(-0.518519, 0), make_hipDoubleComplex(0.444444, 0),
                             make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.055556, 0), make_hipDoubleComplex(0.666667, 0),
                             make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.129630, 0), make_hipDoubleComplex(0.888889, 0)};

    // Device memory management
    hipDoubleComplex *ddl, *dd, *ddu, *dB;

    CHECK_CUDA( hipMalloc((void**) &ddl,  (m - 1) * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  (m - 1) * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dB, ldb * n * sizeof(hipDoubleComplex)) );

    CHECK_CUDA( hipMemcpy(ddl, hdl, (m - 1) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, (m - 1) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseZgtsv2_nopivot_bufferSizeExt(handle, m, n, ddl, dd, ddu, dB, ldb, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseZgtsv2_nopivot(handle, m, n, ddl, dd, ddu, dB, ldb, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hB, dB, ldb * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (ldb * n); i++) {
        if((fabs(hB[i].x - hB_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gtsv2 test PASSED\n");
    else
        printf("gtsv2 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
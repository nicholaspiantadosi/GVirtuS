#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    /*
    const int m = 4;
    const int nnz = 9;

    float hCsrValA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    float hx[] = {1, 2, 3, 4};
    float hy[] = {5, 6, 7, 8};
    float hz[] = {9, 10, 11, 12};

    float hx_result[] = {1, 2, 3, 4};
    float hy_result[] = {5, 6, 7, 8};
    float hz_result[] = {9, 10, 11, 12};
    */

    const int m = 2;
    const int nnz = 3;

    float hCsrValA[] = {1, 2, 3};
    int hCsrRowPtrA[] = {0, 2, 3};
    int hCsrColIndA[] = {0, 1, 1};

    float hx[] = {1, 2};
    float hy[] = {3, 4};
    float hz[] = {5, 6};

    // Device memory management
    float *dCsrValA, *dx, *dy, *dz;
    int *dCsrRowPtrA, *dCsrColIndA;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dx,  m * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dy,  m * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dz,  m * sizeof(float)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dy, hy, m * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dz, hz, m * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    // Suppose that A is m x m sparse matrix represented by CSR format,
    // Assumption:
    // - handle is already created by hipsparseCreate(),
    // - (d_csrRowPtr, d_csrColInd, d_csrVal) is CSR of A on device memory,
    // - d_x is right hand side vector on device memory,
    // - d_y is solution vector on device memory.
    // - d_z is intermediate result on device memory.

    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    csric02Info_t info_M  = 0;
    csrsv2Info_t  info_L  = 0;
    csrsv2Info_t  info_Lt = 0;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_Lt;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const float alpha = 1.;
    const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for csric02 and two info's for csrsv2
    hipsparseCreateCsric02Info(&info_M);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_Lt);

    // step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
    hipsparseScsric02_bufferSize(handle, m, nnz,
                                descr_M, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M, &pBufferSize_M);
    hipsparseScsrsv2_bufferSize(handle, trans_L, m, nnz,
                               descr_L, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L, &pBufferSize_L);
    hipsparseScsrsv2_bufferSize(handle, trans_Lt, m, nnz,
                               descr_L, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_Lt, &pBufferSize_Lt);

    pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_Lt));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete Cholesky on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of csric02 and csrsv2 simultaneously.

    hipsparseScsric02_analysis(handle, m, nnz, descr_M,
                              dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M,
                              policy_M, pBuffer);
    hipsparseStatus_t status = hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseScsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA,
                             info_L, policy_L, pBuffer);

    hipsparseScsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA,
                             info_Lt, policy_Lt, pBuffer);

    // step 5: M = L * L'
    hipsparseScsric02(handle, m, nnz, descr_M,
                     dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M, policy_M, pBuffer);

    // WHY IS IT BLOCKING HERE???
    status = hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseScsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L,
                          dx, dz, policy_L, pBuffer);

    // step 7: solve L'*y = z
    hipsparseScsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, info_Lt,
                          dz, dy, policy_Lt, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hy, dy, m * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hz, dz, m * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        /*if((fabs(hC[i] - hC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }*/
        printf("%f \t %f \t %f\n", hx[i], hx[i], hz[i]);
    }
    if (correct)
        printf("csric02 test PASSED\n");
    else
        printf("csric02 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_M));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_L));
    CHECK_CUSPARSE(hipsparseDestroyCsric02Info(info_M));
    CHECK_CUSPARSE(hipsparseDestroyCsrsv2Info(info_L));
    CHECK_CUSPARSE(hipsparseDestroyCsrsv2Info(info_Lt));
    CHECK_CUDA(hipFree(dCsrValA) );
    CHECK_CUDA(hipFree(dCsrRowPtrA) );
    CHECK_CUDA(hipFree(dCsrColIndA) );
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dy));
    CHECK_CUDA(hipFree(dz));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
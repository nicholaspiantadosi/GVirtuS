#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int nnz = 9;

    hipDoubleComplex hCsrValA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(9, 0)};
    int hCsrRowPtrA[] = {1, 4, 5, 8, 10};
    int hCsrColIndA[] = {1, 3, 4, 2, 1, 3, 4, 2, 4};

    hipDoubleComplex hx[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0)};
    hipDoubleComplex hy[4];

    hipDoubleComplex hy_result[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0.5, 0), make_hipDoubleComplex(0.5, 0), make_hipDoubleComplex(0, 0)};

    // Device memory management
    hipDoubleComplex *dCsrValA, *dx, *dy, *dz;
    int *dCsrRowPtrA, *dCsrColIndA;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dx,  m * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dy,  m * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dz,  m * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    //hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    // Suppose that A is m x m sparse matrix represented by CSR format,
    // Assumption:
    // - handle is already created by hipsparseCreate(),
    // - (dCsrRowPtrA, dCsrColIndA, dCsrValA) is CSR of A on device memory,
    // - dx is right hand side vector on device memory,
    // - dy is solution vector on device memory.
    // - dz is intermediate result on device memory.

    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    hipsparseMatDescr_t descr_U = 0;
    csrilu02Info_t info_M  = 0;
    csrsv2Info_t  info_L  = 0;
    csrsv2Info_t  info_U  = 0;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_U;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipDoubleComplex alpha = make_hipDoubleComplex(1, 0);
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has unit diagonal
    // - matrix U is base-1
    // - matrix U is upper triangular
    // - matrix U has non-unit diagonal
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for csrilu02 and two info's for csrsv2
    hipsparseCreateCsrilu02Info(&info_M);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    // step 3: query how much memory used in csrilu02 and csrsv2, and allocate the buffer
    hipsparseZcsrilu02_bufferSize(handle, m, nnz,
                                 descr_M, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M, &pBufferSize_M);
    hipsparseZcsrsv2_bufferSize(handle, trans_L, m, nnz,
                               descr_L, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L, &pBufferSize_L);
    hipsparseZcsrsv2_bufferSize(handle, trans_U, m, nnz,
                               descr_U, dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U, &pBufferSize_U);

    pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete Cholesky on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on U
    // The lower(upper) triangular part of M has the same sparsity pattern as L(U),
    // we can do analysis of csrilu0 and csrsv2 simultaneously.

    hipsparseZcsrilu02_analysis(handle, m, nnz, descr_M,
                               dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M,
                               policy_M, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseZcsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA,
                             info_L, policy_L, pBuffer);

    hipsparseZcsrsv2_analysis(handle, trans_U, m, nnz, descr_U,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA,
                             info_U, policy_U, pBuffer);

    // step 5: M = L * U
    hipsparseZcsrilu02(handle, m, nnz, descr_M,
                      dCsrValA, dCsrRowPtrA, dCsrColIndA, info_M, policy_M, pBuffer);
    status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseZcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, info_L,
                          dx, dz, policy_L, pBuffer);

    // step 7: solve U*y = z
    hipsparseZcsrsv2_solve(handle, trans_U, m, nnz, &alpha, descr_U,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, info_U,
                          dz, dy, policy_U, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hy, dy, m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hy[i].x - hy_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csrilu02 test PASSED\n");
    else
        printf("csrilu02 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_M));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_L));
    CHECK_CUSPARSE(hipsparseDestroyCsrilu02Info(info_M));
    CHECK_CUSPARSE(hipsparseDestroyCsrsv2Info(info_L));
    CHECK_CUSPARSE(hipsparseDestroyCsrsv2Info(info_U));
    CHECK_CUDA(hipFree(dCsrValA) );
    CHECK_CUDA(hipFree(dCsrRowPtrA) );
    CHECK_CUDA(hipFree(dCsrColIndA) );
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dy));
    CHECK_CUDA(hipFree(dz));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
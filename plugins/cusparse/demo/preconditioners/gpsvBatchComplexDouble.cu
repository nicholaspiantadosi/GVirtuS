#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int algo = 0;

    hipDoubleComplex hds[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(7, 0)};
    hipDoubleComplex hdl[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(1, 0)};
    hipDoubleComplex hd[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0)};
    hipDoubleComplex hdu[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0)};
    hipDoubleComplex hdw[] = {make_hipDoubleComplex(2, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex hx[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(3, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(4, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0),
                      make_hipDoubleComplex(2, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(1, 0),
                      make_hipDoubleComplex(1, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(7, 0)};

    hipDoubleComplex hx_result[] = {make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(-0.436364, 0), make_hipDoubleComplex(-0.000000, 0), make_hipDoubleComplex(0.381818, 0),
                                    make_hipDoubleComplex(-0.000000, 0), make_hipDoubleComplex(3.363636, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(-2.036364, 0),
                                    make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(5.000000, 0), make_hipDoubleComplex(6.000000, 0),
                                    make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(7.000000, 0), make_hipDoubleComplex(8.000000, 0),
                                    make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0),
                                    make_hipDoubleComplex(2.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(3.000000, 0), make_hipDoubleComplex(4.000000, 0),
                                    make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(5.000000, 0), make_hipDoubleComplex(1.000000, 0),
                                    make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(6.000000, 0), make_hipDoubleComplex(7.000000, 0)};

    // Device memory management
    hipDoubleComplex *ddl, *dd, *ddu, *dx, *dds, *ddw;

    CHECK_CUDA( hipMalloc((void**) &dds,  m * batchCount * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddw,  m * batchCount * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(hipDoubleComplex)) );

    CHECK_CUDA( hipMemcpy(dds, hds, m * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddw, hdw, m * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseZgpsvInterleavedBatch_bufferSizeExt(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseZgpsvInterleavedBatch(handle, algo, m, dds, ddl, dd, ddu, ddw, dx, batchCount, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i].x - hx_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gpsvBatch test PASSED\n");
    else
        printf("gpsvBatch test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(dds));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));
    CHECK_CUDA(hipFree(ddw));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
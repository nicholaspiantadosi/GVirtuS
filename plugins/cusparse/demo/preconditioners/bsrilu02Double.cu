#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int blockSize = 2;
    const int mb = 2;
    const int nnzb = 4;

    double hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 4};
    int hBsrRowPtrA[] = {1, 3, 5};
    int hBsrColIndA[] = {1, 2, 1, 2};

    const int m = mb * blockSize;

    double hx[] = {1, 2, 3, 4};
    double hy[4];

    double hy_result[] = {0, 0.5, 0.5, 0};

    // Device memory management
    double *dBsrValA, *dx, *dy, *dz;
    int *dBsrRowPtrA, *dBsrColIndA;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dx,  m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dy,  m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dz,  m * sizeof(double)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    //hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    // Suppose that A is m x m sparse matrix represented by BSR format,
    // The number of block rows/columns is mb, and
    // the number of nonzero blocks is nnzb.
    // Assumption:
    // - handle is already created by hipsparseCreate(),
    // - (dBsrRowPtrA, dBsrColIndA, dBsrValA) is BSR of A on device memory,
    // - dx is right hand side vector on device memory.
    // - dy is solution vector on device memory.
    // - dz is intermediate result on device memory.
    // - dx, dy and dz are of size m.
    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    hipsparseMatDescr_t descr_U = 0;
    bsrilu02Info_t info_M = 0;
    bsrsv2Info_t   info_L = 0;
    bsrsv2Info_t   info_U = 0;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_U;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const double alpha = 1.;
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has unit diagonal
    // - matrix U is base-1
    // - matrix U is upper triangular
    // - matrix U has non-unit diagonal
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for bsrilu02 and two info's for bsrsv2
    hipsparseCreateBsrilu02Info(&info_M);
    hipsparseCreateBsrsv2Info(&info_L);
    hipsparseCreateBsrsv2Info(&info_U);

    // step 3: query how much memory used in bsrilu02 and bsrsv2, and allocate the buffer
    hipsparseDbsrilu02_bufferSize(handle, dir, mb, nnzb,
                                 descr_M, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M, &pBufferSize_M);
    hipsparseDbsrsv2_bufferSize(handle, dir, trans_L, mb, nnzb,
                               descr_L, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_L, &pBufferSize_L);
    hipsparseDbsrsv2_bufferSize(handle, dir, trans_U, mb, nnzb,
                               descr_U, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_U, &pBufferSize_U);

    pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete LU factorization on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on U
    // The lower(upper) triangular part of M has the same sparsity pattern as L(U),
    // we can do analysis of bsrilu0 and bsrsv2 simultaneously.

    hipsparseDbsrilu02_analysis(handle, dir, mb, nnzb, descr_M,
                               dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M,
                               policy_M, pBuffer);
    hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDbsrsv2_analysis(handle, dir, trans_L, mb, nnzb, descr_L,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize,
                             info_L, policy_L, pBuffer);

    hipsparseDbsrsv2_analysis(handle, dir, trans_U, mb, nnzb, descr_U,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize,
                             info_U, policy_U, pBuffer);

    // step 5: M = L * U
    hipsparseDbsrilu02(handle, dir, mb, nnzb, descr_M,
                      dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M, policy_M, pBuffer);
    status = hipsparseXbsrilu02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("block U(%d,%d) is not invertible\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseDbsrsv2_solve(handle, dir, trans_L, mb, nnzb, &alpha, descr_L,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_L,
                          dx, dz, policy_L, pBuffer);

    // step 7: solve U*y = z
    hipsparseDbsrsv2_solve(handle, dir, trans_U, mb, nnzb, &alpha, descr_U,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_U,
                          dz, dy, policy_U, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hy, dy, m * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hy[i] - hy_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsrilu02 test PASSED\n");
    else
        printf("bsrilu02 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_M));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_L));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_U));
    CHECK_CUSPARSE(hipsparseDestroyBsrilu02Info(info_M));
    CHECK_CUSPARSE(hipsparseDestroyBsrsv2Info(info_L));
    CHECK_CUSPARSE(hipsparseDestroyBsrsv2Info(info_U));
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
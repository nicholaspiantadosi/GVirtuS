#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;

    hipComplex hdl[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};
    hipComplex hd[] = {make_hipComplex(1, 0), make_hipComplex(4, 0), make_hipComplex(6, 0), make_hipComplex(9, 0)};
    hipComplex hdu[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(0, 0)};

    int ldb = 4;
    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(8, 0)};

    hipComplex hB_result[] = {make_hipComplex(1.000000, 0), make_hipComplex(0.500000, 0), make_hipComplex(-0.388889, 0), make_hipComplex(0.333333, 0),
                             make_hipComplex(0.000000, 0), make_hipComplex(0.000000, 0), make_hipComplex(-0.518519, 0), make_hipComplex(0.444444, 0),
                             make_hipComplex(0.000000, 0), make_hipComplex(0.000000, 0), make_hipComplex(0.055556, 0), make_hipComplex(0.666667, 0),
                             make_hipComplex(0.000000, 0), make_hipComplex(0.000000, 0), make_hipComplex(0.129630, 0), make_hipComplex(0.888889, 0)};

    // Device memory management
    hipComplex *ddl, *dd, *ddu, *dB;

    CHECK_CUDA( hipMalloc((void**) &ddl,  m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dB, ldb * n * sizeof(hipComplex)) );

    CHECK_CUDA( hipMemcpy(ddl, hdl, m * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * n * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseCgtsv2_bufferSizeExt(handle, m, n, ddl, dd, ddu, dB, ldb, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseCgtsv2(handle, m, n, ddl, dd, ddu, dB, ldb, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hB, dB, ldb * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (ldb * n); i++) {
        if((fabs(hB[i].x - hB_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gtsv2 test PASSED\n");
    else
        printf("gtsv2 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
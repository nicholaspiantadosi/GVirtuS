#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int blockSize = 2;
    const int mb = 2;
    const int nnzb = 4;

    hipComplex hBsrValA[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(4, 0)};
    int hBsrRowPtrA[] = {1, 3, 5};
    int hBsrColIndA[] = {1, 2, 1, 2};

    const int m = mb * blockSize;

    hipComplex hx[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0)};
    hipComplex hy[4];

    hipComplex hy_result[] = {make_hipComplex(1.526316, 0), make_hipComplex(0.500000, 0), make_hipComplex(-0.105263, 0), make_hipComplex(0.000000, 0)};

    // Device memory management
    hipComplex *dBsrValA, *dx, *dy, *dz;
    int *dBsrRowPtrA, *dBsrColIndA;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dx,  m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dy,  m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dz,  m * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    //hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    // Suppose that A is m x m sparse matrix represented by BSR format,
    // The number of block rows/columns is mb, and
    // the number of nonzero blocks is nnzb.
    // Assumption:
    // - handle is already created by hipsparseCreate(),
    // - (dBsrRowPtrA, dBsrColIndA, dBsrValA) is BSR of A on device memory,
    // - dx is right hand side vector on device memory,
    // - dy is solution vector on device memory.
    // - dz is intermediate result on device memory.
    // - dx, dy and dz are of size m.
    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    bsric02Info_t info_M  = 0;
    bsrsv2Info_t  info_L  = 0;
    bsrsv2Info_t  info_Lt = 0;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_Lt;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipComplex alpha = make_hipComplex(1, 0);
    const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for bsric02 and two info's for bsrsv2
    hipsparseCreateBsric02Info(&info_M);
    hipsparseCreateBsrsv2Info(&info_L);
    hipsparseCreateBsrsv2Info(&info_Lt);

    // step 3: query how much memory used in bsric02 and bsrsv2, and allocate the buffer
    hipsparseCbsric02_bufferSize(handle, dir, mb, nnzb,
                                descr_M, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M, &pBufferSize_M);
    hipsparseCbsrsv2_bufferSize(handle, dir, trans_L, mb, nnzb,
                               descr_L, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_L, &pBufferSize_L);
    hipsparseCbsrsv2_bufferSize(handle, dir, trans_Lt, mb, nnzb,
                               descr_L, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_Lt, &pBufferSize_Lt);

    pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_Lt));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete Cholesky on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of bsric02 and bsrsv2 simultaneously.

    hipsparseCbsric02_analysis(handle, dir, mb, nnzb, descr_M,
                              dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M,
                              policy_M, pBuffer);
    hipsparseStatus_t status = hipsparseXbsric02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseCbsrsv2_analysis(handle, dir, trans_L, mb, nnzb, descr_L,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize,
                             info_L, policy_L, pBuffer);

    hipsparseCbsrsv2_analysis(handle, dir, trans_Lt, mb, nnzb, descr_L,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize,
                             info_Lt, policy_Lt, pBuffer);

    // step 5: M = L * L'
    hipsparseCbsric02(handle, dir, mb, nnzb, descr_M,
                           dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_M, policy_M, pBuffer);
    status = hipsparseXbsric02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is not positive definite\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseCbsrsv2_solve(handle, dir, trans_L, mb, nnzb, &alpha, descr_L,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_L,
                          dx, dz, policy_L, pBuffer);

    // step 7: solve L'*y = z
    hipsparseCbsrsv2_solve(handle, dir, trans_Lt, mb, nnzb, &alpha, descr_L,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info_Lt,
                          dz, dy, policy_Lt, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hy, dy, m * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hy[i].x - hy_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsric02 test PASSED\n");
    else
        printf("bsric02 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_M));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_L));
    CHECK_CUSPARSE(hipsparseDestroyBsric02Info(info_M));
    CHECK_CUSPARSE(hipsparseDestroyBsrsv2Info(info_L));
    CHECK_CUSPARSE(hipsparseDestroyBsrsv2Info(info_Lt));
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int m = 4;
    const int n = 4;
    int batchCount = 2;
    int algo = 1;

    double hdl[] = {1, 1, 1, 1, 1, 1, 5, 1};
    double hd[] = {1, 4, 6, 9, 1, 2, 1, 1};
    double hdu[] = {1, 1, 7, 1, 1, 3, 1, 1};

    double hx[] = {1, 2, 1, 3,
                  1, 1, 1, 4,
                  1, 1, 5, 6,
                  1, 1, 7, 8,
                  1, 1, 1, 1,
                  2, 1, 3, 4,
                  1, 1, 5, 1,
                  1, 1, 6, 7};

    double hx_result[] = {1.000000, 0.717949, 0.000000, -0.871795,
                         0.000000, 10.128206, 1.000000, -6.128206,
                         1.000000, 1.000000, 5.000000, 6.000000,
                         1.000000, 1.000000, 7.000000, 8.000000,
                         1.000000, 1.000000, 1.000000, 1.000000,
                         2.000000, 1.000000, 3.000000, 4.000000,
                         1.000000, 1.000000, 5.000000, 1.000000,
                         1.000000, 1.000000, 6.000000, 7.000000};

    // Device memory management
    double *ddl, *dd, *ddu, *dx;

    CHECK_CUDA( hipMalloc((void**) &ddl,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dd,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &ddu,  m * batchCount * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dx, m * n * batchCount * sizeof(double)) );

    CHECK_CUDA( hipMemcpy(ddl, hdl, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dd, hd, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(ddu, hdu, m * batchCount * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dx, hx, m * n * batchCount * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSizeInBytes;
    void *pBuffer = 0;

    hipsparseDgtsvInterleavedBatch_bufferSizeExt(handle, algo, m, ddl, dd, ddu, dx, batchCount, &bufferSizeInBytes);

    hipMalloc((void**)&pBuffer, bufferSizeInBytes);

    hipsparseDgtsvInterleavedBatch(handle, algo, m, ddl, dd, ddu, dx, batchCount, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hx, dx, m * n * batchCount * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m * n * batchCount); i++) {
        if((fabs(hx[i] - hx_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gtsv2Batched test PASSED\n");
    else
        printf("gtsv2Batched test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dx));
    CHECK_CUDA(hipFree(ddl));
    CHECK_CUDA(hipFree(dd));
    CHECK_CUDA(hipFree(ddu));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
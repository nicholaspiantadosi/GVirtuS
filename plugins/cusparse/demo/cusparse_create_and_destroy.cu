
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>

const char * getErrorString(hipsparseStatus_t error)
{
    switch (error)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            return "The operation completed successfully.";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "The cuSPARSE library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSPARSE routine, or an error in the hardware setup.\n" \
				"To correct: call cusparseCreate() prior to the function call; and check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "Resource allocation failed inside the cuSPARSE library. This is usually caused by a cudaMalloc() failure.\n"\
					"To correct: prior to the function call, deallocate previously allocated memory as much as possible.";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "An unsupported value or parameter was passed to the function (a negative vector size, for example).\n"\
				"To correct: ensure that all the parameters being passed have valid values.";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision.\n"\
				"To correct: compile and run the application on a device with appropriate compute capability, which is 1.1 for 32-bit atomic operations and 1.3 for double precision.";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "An access to GPU memory space failed, which is usually caused by a failure to bind a texture.\n"\
				"To correct: prior to the function call, unbind any previously bound textures.";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "An internal cuSPARSE operation failed. This error is usually caused by a cudaMemcpyAsync() failure.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed. Also, check that the memory passed as a parameter to the routine is not being deallocated prior to the routine’s completion.";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function.\n"\
					"To correct: check that the fields in hipsparseMatDescr_t descrA were set correctly.";
    }

    return "<unknown>";
}

void CHECK_CUSPARSE(hipsparseStatus_t status)
{
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, getErrorString(status), status);
        exit(EXIT_FAILURE);
    }
}

int main(void)
{
    hipsparseHandle_t handle;
    printf("cusparseCreate\n");
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    printf("cusparseDestroy\n");
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    return 0;
}

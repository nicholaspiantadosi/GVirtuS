#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(void)
{
    int64_t size = 10;
    int64_t nnz = 5;
    int hIndices[] = {0, 2, 4, 6, 8};
    float hValues[] = {1, 2, 3, 4, 5};

    int *dIndices;
    float *dValues;

    CHECK_CUDA(hipMalloc((void**) &dIndices, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &dValues, nnz * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dIndices, hIndices, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dValues, hValues, nnz * sizeof(float), hipMemcpyHostToDevice) );

    hipsparseSpVecDescr_t spVecDescr;
    CHECK_CUSPARSE(hipsparseCreateSpVec(&spVecDescr, size, nnz, dIndices, dValues, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    int64_t size_toverify;
    int64_t nnz_toverify;
    int hIndices_toverify[nnz];
    float hValues_toverify[nnz];
    hipsparseIndexType_t idxType_toverify;
    hipsparseIndexBase_t idxBase_toverify;
    hipDataType valueType_toverify;

    int *dIndices_toverify;
    float *dValues_toverify;

    CHECK_CUDA(hipMalloc((void**) &dIndices_toverify, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &dValues_toverify, nnz * sizeof(float)));

    CHECK_CUSPARSE(hipsparseSpVecGet(spVecDescr, &size_toverify, &nnz_toverify, (void**)&dIndices_toverify, (void**)&dValues_toverify, &idxType_toverify, &idxBase_toverify, &valueType_toverify));

    CHECK_CUDA(hipMemcpy(hIndices_toverify, dIndices_toverify, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA(hipMemcpy(hValues_toverify, dValues_toverify, nnz * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    if (size_toverify != size) {
        correct = 0;
    }
    if (nnz_toverify != nnz) {
        correct = 0;
    }
    for (int i = 0; i < nnz; i++) {
        if((fabs(hIndices_toverify[i] - hIndices[i]) > 0.000001) || (fabs(hValues_toverify[i] - hValues[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (idxType_toverify != HIPSPARSE_INDEX_32I) {
        correct = 0;
    }
    if (idxBase_toverify != HIPSPARSE_INDEX_BASE_ZERO) {
        correct = 0;
    }
    if (valueType_toverify != HIP_R_32F) {
        correct = 0;
    }

    CHECK_CUSPARSE(hipsparseSpVecGetIndexBase(spVecDescr, &idxBase_toverify));
    if (idxBase_toverify != HIPSPARSE_INDEX_BASE_ZERO) {
        correct = 0;
    }

    float hValues2[] = {5, 4, 3, 2, 1};
    float *dValues2;
    CHECK_CUDA(hipMalloc((void**) &dValues2, nnz * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dValues2, hValues2, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUSPARSE(hipsparseSpVecSetValues(spVecDescr, dValues2));
    float *dValues2_toverify;
    CHECK_CUDA(hipMalloc((void**) &dValues2_toverify, nnz * sizeof(float)));
    CHECK_CUSPARSE(hipsparseSpVecGetValues(spVecDescr, (void**) &dValues2_toverify));
    float hValues2_toverify[nnz];
    CHECK_CUDA(hipMemcpy(hValues2_toverify, dValues2_toverify, nnz * sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < nnz; i++) {
        if((fabs(hValues2_toverify[i] - hValues2[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("spvec test PASSED\n");
    else
        printf("spvec test FAILED: wrong result\n");

    CHECK_CUSPARSE(hipsparseDestroySpVec(spVecDescr));
    CHECK_CUDA(hipFree(dIndices) );
    CHECK_CUDA(hipFree(dValues) );

    return EXIT_SUCCESS;
}

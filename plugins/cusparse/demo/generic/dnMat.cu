#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(void) {
    int64_t rows = 4;
    int64_t cols = 5;
    int64_t ld = cols;

    float hValues[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10};

    float *dValues;

    CHECK_CUDA(hipMalloc((void**) &dValues, rows * cols * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dValues, hValues, rows * cols * sizeof(float), hipMemcpyHostToDevice) );

    hipsparseDnMatDescr_t dnMatDescr;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&dnMatDescr, rows, cols, ld, dValues, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    int64_t rows_toverify;
    int64_t cols_toverify;
    int64_t ld_toverify;
    float hValues_toverify[rows * cols];
    float *dValues_toverify;
    hipDataType valueType_toverify;
    hipsparseOrder_t order_toverify;
    CHECK_CUDA(hipMalloc((void**) &dValues_toverify, rows * cols * sizeof(float)));

    CHECK_CUSPARSE(hipsparseDnMatGet(dnMatDescr, &rows_toverify, &cols_toverify, &ld_toverify, (void**)&dValues_toverify, &valueType_toverify, &order_toverify));

    CHECK_CUDA(hipMemcpy(hValues_toverify, dValues_toverify, rows * cols * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    if (rows_toverify != rows) {
        correct = 0;
    }
    if (cols_toverify != cols) {
        correct = 0;
    }
    if (ld_toverify != ld) {
        correct = 0;
    }
    for (int i = 0; i < rows * cols; i++) {
        if((fabs(hValues_toverify[i] - hValues[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (valueType_toverify != HIP_R_32F) {
        correct = 0;
    }
    if (order_toverify != HIPSPARSE_ORDER_ROW) {
        correct = 0;
    }

    float hValues2[] = {11, 2, 3, 4, 5, 6, 7, 8, 9, 10, 12, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float *dValues2;
    CHECK_CUDA(hipMalloc((void**) &dValues2, rows * cols * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dValues2, hValues2, rows * cols * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUSPARSE(hipsparseDnMatSetValues(dnMatDescr, dValues2));
    float hValues2_toverify[rows * cols];
    float *dValues2_toverify;
    CHECK_CUDA(hipMalloc((void**) &dValues2_toverify, rows * cols * sizeof(float)));
    CHECK_CUSPARSE(hipsparseDnMatGetValues(dnMatDescr, (void**)&dValues2_toverify));
    CHECK_CUDA(hipMemcpy(hValues2_toverify, dValues2_toverify, rows * cols * sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < rows * cols; i++) {
        if((fabs(hValues2_toverify[i] - hValues2[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("dnMat test PASSED\n");
    else
        printf("dnMat test FAILED: wrong result\n");

    CHECK_CUSPARSE(hipsparseDestroyDnMat(dnMatDescr));
    CHECK_CUDA(hipFree(dValues) );
    CHECK_CUDA(hipFree(dValues2) );

    return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(void) {
    int64_t rows = 4;
    int64_t cols = 5;
    int64_t nnz = 9;

    float hCscValues[] = {1, 4, 2, 3, 5, 7, 8, 9, 6};
    int hCscColOffsetsA[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndA[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};

    float *dCscValues;
    int *dCscColOffsetsA, *dCscRowIndA;

    CHECK_CUDA(hipMalloc((void**) &dCscValues, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**) &dCscColOffsetsA, (cols + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &dCscRowIndA, nnz * sizeof(int)));

    CHECK_CUDA(hipMemcpy(dCscValues, hCscValues, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCscColOffsetsA, hCscColOffsetsA, (cols + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCscRowIndA, hCscRowIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );

    hipsparseSpMatDescr_t spMatDescr;
    CHECK_CUSPARSE(hipsparseCreateCsc(&spMatDescr, rows, cols, nnz, dCscColOffsetsA, dCscRowIndA, dCscValues, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    float hCscValues2[] = {10, 4, 2, 3, 5, 7, 8, 9, 6};
    int hCscColOffsetsA2[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndA2[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    float *dCscValues2;
    int *dCscColOffsetsA2, *dCscRowIndA2;
    CHECK_CUDA(hipMalloc((void**) &dCscValues2, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**) &dCscColOffsetsA2, (cols + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &dCscRowIndA2, nnz * sizeof(int)));
    CHECK_CUDA(hipMemcpy(dCscValues2, hCscValues2, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCscColOffsetsA2, hCscColOffsetsA2, (cols + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCscRowIndA2, hCscRowIndA2, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUSPARSE(hipsparseCscSetPointers(spMatDescr, dCscColOffsetsA2, dCscRowIndA2, dCscValues2));

    int64_t rows_toverify;
    int64_t cols_toverify;
    int64_t nnz_toverify;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(spMatDescr, &rows_toverify, &cols_toverify, &nnz_toverify));

    int correct = 1;
    if (rows_toverify != rows) {
        correct = 0;
    }
    if (cols_toverify != cols) {
        correct = 0;
    }
    if (nnz_toverify != nnz) {
        correct = 0;
    }

    hipsparseFormat_t format;
    CHECK_CUSPARSE(hipsparseSpMatGetFormat(spMatDescr, &format));
    if (format != HIPSPARSE_FORMAT_CSC) {
        correct = 0;
    }

    hipsparseIndexBase_t idxBase_toverify;
    CHECK_CUSPARSE(hipsparseSpMatGetIndexBase(spMatDescr, &idxBase_toverify));
    if (idxBase_toverify != HIPSPARSE_INDEX_BASE_ZERO) {
        correct = 0;
    }

    float hCscValues3[] = {11, 4, 2, 3, 5, 7, 8, 9, 6};
    float *dCscValues3;
    float hCscValues3_toverify[nnz];
    float *dCscValues3_toverify;
    CHECK_CUDA(hipMalloc((void**) &dCscValues3, nnz * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dCscValues3, hCscValues3, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUSPARSE(hipsparseSpMatSetValues(spMatDescr, dCscValues3));
    CHECK_CUDA(hipMalloc((void**) &dCscValues3_toverify, nnz * sizeof(float)));
    CHECK_CUSPARSE(hipsparseSpMatGetValues(spMatDescr, (void**)&dCscValues3_toverify));
    CHECK_CUDA(hipMemcpy(hCscValues3_toverify, dCscValues3_toverify, nnz * sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCscValues3_toverify[i] - hCscValues3[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csc test PASSED\n");
    else
        printf("csc test FAILED: wrong result\n");

    CHECK_CUSPARSE(hipsparseDestroySpMat(spMatDescr));
    CHECK_CUDA(hipFree(dCscValues) );
    CHECK_CUDA(hipFree(dCscRowIndA) );
    CHECK_CUDA(hipFree(dCscColOffsetsA) );
    CHECK_CUDA(hipFree(dCscRowIndA2) );
    CHECK_CUDA(hipFree(dCscColOffsetsA2) );
    CHECK_CUDA(hipFree(dCscValues2) );

    return EXIT_SUCCESS;
}

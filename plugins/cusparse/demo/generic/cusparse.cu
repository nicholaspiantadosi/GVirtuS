#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "utilities2.h"

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, double *, int, int, int, double *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, double, double *, int *, int *, double *, double, double *);
void bsrnnz(hipsparseHandle_t, hipsparseMatDescr_t, int *, int *, int, int, int, int, int, int, int *, int &);
void csr2bsr(hipsparseHandle_t, hipsparseMatDescr_t, double *, int *, int *, int, int, int, int, int , int , int, int *,  double *, int *);
void bsrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, int, int, int, double, double *, int *, int *, int, double *, double, double *);

// Interfacce per calcoli su host
void sommaArrayCompPerCompCPU(double *, double *, int, double *);
void prodottoScalareCPU(double *, double, int, double *);
void prodottoScalareArrayCPU(double *, double *, int, double, double *);
void prodottoMatriceVettoreCPU(double *, double *, int, int, double, double *);
void aAxpbySequential(double *, double *, int, int, double, double, double *);

int main(int argn, char *argv[])
{
	// Inizio esecuzione
	clock_t tStart = clock();

	// Variabili generiche
	int m, n, nnz, blockDim, mb=0, nb=0, nnzb=0, csrOperation;
	bool flag_print = true;

    // Variabili su host
    double *matrix_host;
	double *csr_values_result;
	int *csr_columns_result, *csr_offsets_result;
	double *bsr_values_result;
	int *bsr_columns_result, *bsr_offsets_result;
	double *x_host, *y_result_csr, *y_result_bsr;
	double alpha, beta;
	double *matrix_host_sequential;
	double *y_result_sequential;

	// Imposto il seed in base all'orario
	srand(time(0));

	printf("\n");
	printf("################################################################################################################################\n");
    printf("##########            Calcolo del prodotto di una matrice sparsa per vettore mediante la libreria CUSPARSE            ##########\n"); 
	printf("################################################################################################################################\n\n");

	printf("Se NON vengono forniti dati in input uso valori default (il vettore e la matrice avranno elementi con valori casuali):\n");

    // Controlli su input
    if(argn < 8)
	{ 
		printf("Numero di parametri insufficiente --> Uso dei valori di default\n");
		m = 4;
		n = 5;
        nnz = 9;
		blockDim = 2;
		alpha = 3;
		beta = 2;
		csrOperation = 1;
	}
	else
	{
		printf("Numero di parametri corretto\n");
		m = atoi(argv[1]);
		n = atoi(argv[2]);
        nnz = atoi(argv[3]);
		blockDim = atoi(argv[4]);
		alpha = atoi(argv[5]);
		beta = atoi(argv[6]);
		csrOperation = atoi(argv[7]);
	}

	printf("\n");
	printf("********************************************************************************************************************************\n");
	if (csrOperation == 1)
		printf("**********                                       OPERAZIONE SELEZIONATA --> CSR                                       **********\n"); 
	else
		printf("**********                                       OPERAZIONE SELEZIONATA --> BSR                                       **********\n"); 
	printf("********************************************************************************************************************************\n\n");

	printf("\tM=%d righe\n", m);
	printf("\tN=%d colonne\n", n);
	printf("\tnnz=%d valori diversi da 0\n", nnz);
	printf("\tblockDim=%d dimensione blocco per formato BSR\n", blockDim);
	printf("\talpha=%f per operazione bsrmv\n", alpha);
	printf("\tbeta=%f per operazione bsrmv\n", beta);
	printf("\tcsrOperation=%d per effettuare operazione csrmv (0 per bsrmv, 1 per csrmv)\n", csrOperation);
	if (beta > 0) {
		printf("\tè stato indicato beta > 0, l'array di output y verrà inizializzato con valori random\n");

	}
	printf("\n");

	// Controllo su numero di valori diversi da zero richiesti
    if (nnz > (m * n / 2)) {
		printf("Per rappresentare la matrice in un formato sparso, i valori devono essere al massimo pari alla metà della dimensione.\n");
		return 2;
	}

    // Allocazione memoria sull'host
	matrix_host = (double *)malloc((m*n+1)*sizeof(double));
	csr_values_result=(double *)malloc(nnz * sizeof(double));
	csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
	csr_columns_result=(int *)malloc(nnz * sizeof(int));
	x_host = (double *)malloc(n*sizeof(double));
	if (csrOperation == 1)
		y_result_csr = (double *)malloc(m*sizeof(double));
	else
		y_result_bsr = (double *)malloc(m*sizeof(double));
	matrix_host_sequential = (double *)malloc((m*n+1)*sizeof(double));
	y_result_sequential = (double *)malloc(m*sizeof(double));

	// Inizializzazione variabili sull'host
    initializeMatrixRandomSparse(matrix_host, m, n, nnz);
	initializeArrayRandom(x_host, n);
	if (beta > 0)
		if (csrOperation == 1)
			initializeArrayRandom(y_result_csr, m);
		else
			initializeArrayRandom(y_result_bsr, m);
	else
		if (csrOperation == 1)
			initializeArrayToZero(y_result_csr, m);
		else
			initializeArrayToZero(y_result_bsr, m);
	if (csrOperation == 1)
		copyArray(y_result_csr, y_result_sequential, m);
	else
		copyArray(y_result_bsr, y_result_sequential, m);

	// Swap formato matrice per calcolo sequenziale su host
	swapMatrix(matrix_host, m, n, matrix_host_sequential);

    // Stampa variabili generate
	if (m <= 40 && n <= 40)
	{
		printf("Matrice sparsa:\n");
		stampaMatrix(matrix_host, m, n);
		printf("\n");

		printf("Array x:\n");
		stampaArrayF(x_host, n);
		printf("\n");

		printf("Matrice in formato denso per calcolo sequenziale:\n");
		stampaMatrixF1(matrix_host_sequential, m, n);
		printf("\n");

		if (beta > 0) {
			printf("Array y:\n");
			stampaArrayF(y_result_csr, m);
			printf("\n");
		}
	}	
	else
    {
		printf("Numero eccessivo di valori, la matrice non verrà visualizzata a video...\n\n");
		flag_print = false;
    }

	// Dichiarazione dell'handle per CUSPARSE
	hipsparseHandle_t handle;

	// Creazione dell'handle per CUSPARSE
	CHECK_CUSPARSE(hipsparseCreate(&handle));

	// Creazione della struttura della matrice con relative proprietà e forma
	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	// Tempo inizializzazioni
	clock_t tInit = clock();

	// Conversione matrice in formato CSR
	mat2csr(handle, descr, matrix_host, m, n, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

	if (flag_print) 
	{
		//Stampa matrice convertita in formato CSR
		printf("Matrice sparsa convertita in formato CSR\n");
		printf("\t csrValA:\t");
		stampaArrayF(csr_values_result, nnz);
		printf("\t csrRowPtrA:\t");
		stampaArray(csr_offsets_result, (m + 1));
		printf("\t csrColIndA:\t");
		stampaArray(csr_columns_result, nnz);
		printf("\n");
	}

	if (csrOperation == 1) 
	{
		// Operazione csrmv corrispondente al seguente prodotto y = alpha * A * x + beta * y
		csrmv(handle, descr, m, n, nnz, alpha, csr_values_result, csr_offsets_result, csr_columns_result, x_host, beta, y_result_csr);

		if (flag_print) 
		{
			//Stampa array risultato dall'operazione vsrmv tra matrice in formato CSR, vettore x_host, alpha e beta
			printf("Vettore risultato dall'operazione csrmv\n");
			stampaArrayF(y_result_csr, m);
			printf("\n");
		}
	}
	else
	{
		// Calcolo mb e nb a partire da blockDim
		mb = (m + blockDim - 1) / blockDim;
		nb = (n + blockDim - 1) / blockDim;

		// Allocazione variabili su host
		bsr_offsets_result=(int *)malloc((mb + 1) * sizeof(int));

		// Calcolo bsrRowPtrC e blocchi diversi da zero
		bsrnnz(handle, descr, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, bsr_offsets_result, nnzb);

		// Allocazione variabili su host sulla base del numero di blocchi diversi da zero
		bsr_columns_result=(int *)malloc(nnzb * sizeof(int));
		bsr_values_result=(double *)malloc((blockDim * blockDim) * nnzb * sizeof(double));

		// Conversione da formato CSR a BSR
		csr2bsr(handle, descr, csr_values_result, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, nnzb, bsr_offsets_result, bsr_values_result, bsr_columns_result);

		if (flag_print) 
		{
			// Stampa matrice convertita in formato BSR
			printf("Matrice sparsa convertita in formato BSR\n");
			printf("\t bsrValC:\t");
			stampaArrayF(bsr_values_result, (blockDim * blockDim) * nnzb);
			printf("\t bsrRowPtrC:\t");
			stampaArray(bsr_offsets_result, (mb + 1));
			printf("\t bsrColIndC:\t");
			stampaArray(bsr_columns_result, nnzb);
			printf("\n");
		}

		// Operazione bsrmv corrispondente al seguente prodotto y = alpha * A * x + beta * y
		bsrmv(handle, descr, m, n, nnz, mb, nb, nnzb, alpha, bsr_values_result, bsr_offsets_result, bsr_columns_result, blockDim, x_host, beta, y_result_bsr);

		if (flag_print) 
		{
			//Stampa array risultato dall'operazione bsrmv tra matrice in formato BSR, vettore x_host, alpha e beta
			printf("Vettore risultato dall'operazione bsrmv\n");
			stampaArrayF(y_result_bsr, m);
			printf("\n");
		}
	}

	// Tempo operazioni CUSPARSE
	clock_t tCusparseOperations = clock();

	// Operazione y = alpha * A * x + beta * y effettuata in modalità sequenziale
	aAxpbySequential(matrix_host_sequential, x_host, m, n, alpha, beta, y_result_sequential);

	// Tempo operazione sequenziale
	clock_t tSequentialOperation = clock();

	if (flag_print) 
	{
		//Stampa array risultato dall'operazione [y = alpha * A * x + beta * y] sequenziale tra matrice in formato denso, vettore x_host, alpha e beta
		printf("Vettore risultato dall'operazione [y = alpha * A * x + beta * y] sequenziale\n");
		stampaArrayF(y_result_sequential, m);
		printf("\n");
	}

	// Verifica se i due array risultato (uno calcolato con cusparse sul device e uno calcolato su host) coincidono
	if (csrOperation == 1)
		equalArrayD(y_result_csr, y_result_sequential, m);
	else
		equalArrayD(y_result_bsr, y_result_sequential, m);

	//Libera la memoria sull'host
	if (csrOperation == 1) {
		free(csr_values_result);
		free(csr_offsets_result);
		free(csr_columns_result);
	} else {
		free(bsr_values_result);
		free(bsr_offsets_result);
		free(bsr_columns_result);
	}
	free(matrix_host);
	free(matrix_host_sequential);
	// free(x_host);
	// free(y_result_csr);
	// free(y_result_bsr);
	// free(y_result_sequential);

	// Termina l'handle per CUSPARSE
	CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

	// Tempo liberazione memoria
	clock_t tFinish = clock();

	// Log tempi esecuzione
	printf("\nTempo inizializzazione:\t\t%fms", (double)(tInit - tStart)/(CLOCKS_PER_SEC/1000));
	printf("\nTempo operazioni cusparse:\t%fms", (double)(tCusparseOperations - tInit)/(CLOCKS_PER_SEC/1000));
	printf("\nTempo operazione sequenziale:\t%fms", (double)(tSequentialOperation - tCusparseOperations)/(CLOCKS_PER_SEC/1000));
	printf("\nTempo liberazione memoria:\t%fms", (double)(tFinish - tSequentialOperation)/(CLOCKS_PER_SEC/1000));
	printf("\nTempo totale:\t\t\t%fms\n\n", (double)(tFinish - tStart)/(CLOCKS_PER_SEC/1000));
}

// Function che consente di ottenere la matrice in formato CSR
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - matrix - matrice sparsa
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//     output:
//         - csrValA - array contenente i valori
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga
//         - csrColIndA - array contenente gli indici di colonne
void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, double * matrix, int m, int n, int nnz, double * csrValA, int * csrRowPtrA, int * csrColIndA)
{
	int nnz_total = 0;

	// Variabili su device
	double *matrix_device;
    double *csr_values_device;
	int *csr_columns_device, *csr_offsets_device;
	int *nnz_per_row;

	// Allocazione memoria su device
	CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

	// Inizializzazione variabili su device
	CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(double), hipMemcpyHostToDevice));

	// Calcolo valori diversi da zero
	CHECK_CUSPARSE(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

	// Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseSnnz()
	if (nnz != nnz_total) {
		printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
		exit(EXIT_FAILURE);
	}

	// Conversione matrice in formato CSR
	CHECK_CUSPARSE(hipsparseDdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
	CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(double), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

	//Libera la memoria sul device
	CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
	CHECK_CUDA(hipFree(nnz_per_row));
}

// Function che calcola il prodotto matrice per vettore tramite csrmv (y = alpha * A * x + beta * y)
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - alpha - scalare che viene utilizzato nel primo prodotto csrmv (alpha * A * x)
//         - csrValA - array contenente i valori della matrice in formato CSR
//         - csrRowPtrA - array contenente i valori di indice di riga della matrice in formato CSR
//         - csrColIndA - array contenente i valori di colonna della matrice in formato CSR
//         - x - array che viene utilizzato nel primo prodotto csrmv (alpha * A * x)
//         - beta - scalare che viene utilizzato nel secondo prodotto csrmv (beta * y)
//         - y - array che viene utilizzato nel secondo prodotto csrmv (beta * y)
//     output:
//         - y - array risultato dell'operazione csrmv (y = alpha * A * x + beta * y)
void csrmv(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int n, int nnz, double alpha, double * csrValA, int * csrRowPtrA, int * csrColIndA, double * x, double beta, double * y)
{
	// Variabili su device
	double *x_device, *y_device;
	double *csrValA_device;
	int *csrRowPtrA_device, *csrColIndA_device;

	// Allocazione variabili su device
	CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**)&x_device, n * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**)&y_device, m * sizeof(double)));

	// Copia valori da host a device
	CHECK_CUDA(hipMemcpy(x_device, x, n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

	// Operazione y = alpha * A * x + beta * y
	// la matrice A è rappresentata in formato CSR dagli array csrValA_device, csrRowPtrA_device e csrColIndA_device
	CHECK_CUSPARSE(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, &alpha, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, x_device, &beta, y_device));

	// Copia valori da device a host
	CHECK_CUDA(hipMemcpy(y, y_device, m * sizeof(double), hipMemcpyDeviceToHost));

	//Libera la memoria sul device
	CHECK_CUDA(hipFree(csrValA_device));
	CHECK_CUDA(hipFree(csrRowPtrA_device));
	CHECK_CUDA(hipFree(csrColIndA_device));
	CHECK_CUDA(hipFree(x_device));
	CHECK_CUDA(hipFree(y_device));

}

// Function che consente di ottenere il numero dei blocchi con elementi diversi da zero e l'array bsrRowPtrC inizializzato con i valori di indice per i blocchi di riga
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga, della matrice in formato CSR
//         - csrColIndA - array contenente gli indici di colonne, della matrice in formato CSR
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//     output:
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//         - nnzb - blocchi con valori diversi da zero
void bsrnnz(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int * bsrRowPtrC, int &nnzb)
{
	// Variabili su host
	int base;
	int *nnzTotalBsr = &nnzb;
	
	// Variabili su device
	int *bsrRowPtrC_device;
	int *csrRowPtrA_device, *csrColIndA_device;

	// Allocazione variabili su device
	CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

	// Copia variabili da host a device
	CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

	// Calcolo del numero dei blocchi diversi da zero per il formato BSR
	CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrRowPtrC_device, nnzTotalBsr));

	// Controllo sul valore dei blocchi calcolato, se null lo calcolo sulla base degli indici dei blocchi e il numero dei blocchi per riga
	if (NULL != nnzTotalBsr)
	{
		nnzb = *nnzTotalBsr;
	}
	else
	{
		CHECK_CUDA(hipMemcpy(&nnzb, bsrRowPtrC_device + mb, sizeof(int), hipMemcpyDeviceToHost));
		CHECK_CUDA(hipMemcpy(&base, bsrRowPtrC_device, sizeof(int), hipMemcpyDeviceToHost));
		nnzb -= base;
	}

	// Copia risultato da device a host
	CHECK_CUDA(hipMemcpy(bsrRowPtrC, bsrRowPtrC_device, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost));

	//Libera la memoria sul device
	CHECK_CUDA(hipFree(bsrRowPtrC_device));
	CHECK_CUDA(hipFree(csrRowPtrA_device));
	CHECK_CUDA(hipFree(csrColIndA_device));

}

// Function che consente di ottenere il numero dei blocchi con elementi diversi da zero e l'array bsrRowPtrC inizializzato con i valori di indice per i blocchi di riga
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - csrValA - array contenente i valori della matrice in formato CSR
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga, della matrice in formato CSR
//         - csrColIndA - array contenente gli indici di colonne, della matrice in formato CSR
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//         - nnzb - blocchi con valori diversi da zero
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//     output:
//         - bsrValC - array contenente i valori della matrice in formato BSR
//         - bsrColIndC - array contenente i valori di colonna dei blocchi della matrice in formato BSR
void csr2bsr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, double * csrValA, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int nnzb, int * bsrRowPtrC, double * bsrValC, int * bsrColIndC)
{
	// Variabili su device
	double *bsrValC_device;
	int *bsrRowPtrC_device, *bsrColIndC_device;
	double *csrValA_device;
	int *csrRowPtrA_device, *csrColIndA_device;

	// Allocazione variabili su device sulla base del numero di blocchi diversi da zero
	CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**)&bsrColIndC_device, nnzb * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**)&bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

	// Copia da host a device
	CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));

	// Conversione da CSR a BSR
	CHECK_CUSPARSE(hipsparseDcsr2bsr(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device));

	// Copia risultato da device a host
	CHECK_CUDA(hipMemcpy(bsrValC, bsrValC_device, ((blockDim * blockDim) * nnzb) * sizeof(double), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipMemcpy(bsrColIndC, bsrColIndC_device, nnzb * sizeof(int), hipMemcpyDeviceToHost));

	//Libera la memoria sul device
	CHECK_CUDA(hipFree(bsrValC_device));
	CHECK_CUDA(hipFree(bsrColIndC_device));
	CHECK_CUDA(hipFree(bsrRowPtrC_device));
	CHECK_CUDA(hipFree(csrValA_device));
	CHECK_CUDA(hipFree(csrRowPtrA_device));
	CHECK_CUDA(hipFree(csrColIndA_device));

}

// Function che calcola il prodotto matrice per vettore tramite bsrmv (y = alpha * A * x + beta * y)
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//         - nnzb - blocchi con valori diversi da zero
//         - alpha - scalare che viene utilizzato nel primo prodotto bsrmv (alpha * A * x)
//         - bsrValC - array contenente i valori della matrice in formato BSR
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//         - bsrColIndC - array contenente i valori di colonna dei blocchi della matrice in formato BSR
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - x - array che viene utilizzato nel primo prodotto bsrmv (alpha * A * x)
//         - beta - scalare che viene utilizzato nel secondo prodotto bsrmv (beta * y)
//         - y - array che viene utilizzato nel secondo prodotto bsrmv (beta * y)
//     output:
//         - y - array risultato dell'operazione bsrmv (y = alpha * A * x + beta * y)
void bsrmv(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int n, int nnz, int mb, int nb, int nnzb, double alpha, double * bsrValC, int * bsrRowPtrC, int * bsrColIndC, int blockDim, double * x, double beta, double * y)
{
	// Variabili su device
	double *x_device, *y_device;
	double *bsrValC_device;
	int *bsrRowPtrC_device, *bsrColIndC_device;

	// Allocazione variabili su device
	CHECK_CUDA(hipMalloc((void**) &bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**) &bsrRowPtrC_device, (mb + 1) * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**) &bsrColIndC_device, nnzb * sizeof(int)));
	CHECK_CUDA(hipMalloc((void**)&x_device, (nb*blockDim) * sizeof(double)));
	CHECK_CUDA(hipMalloc((void**)&y_device, (mb*blockDim) * sizeof(double)));

	// Copia valori da host a device
	CHECK_CUDA(hipMemcpy(x_device, x, n * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(bsrValC_device, bsrValC, (blockDim * blockDim) * nnzb * sizeof(double), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(bsrColIndC_device, bsrColIndC, nnzb * sizeof(int), hipMemcpyHostToDevice));

	// Operazione y = alpha * A * x + beta * y
	// la matrice A è rappresentata in formato BSR dagli array bsrValC_device, bsrRowPtrC_device e bsrColIndC_device
	CHECK_CUSPARSE(hipsparseDbsrmv(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device, blockDim, x_device, &beta, y_device));

	// Copia valori da device a host
	CHECK_CUDA(hipMemcpy(y, y_device, (mb*blockDim) * sizeof(double), hipMemcpyDeviceToHost));

	//Libera la memoria sul device
	CHECK_CUDA(hipFree(bsrValC_device));
	CHECK_CUDA(hipFree(bsrColIndC_device));
	CHECK_CUDA(hipFree(bsrRowPtrC_device));
	CHECK_CUDA(hipFree(x_device));
	CHECK_CUDA(hipFree(y_device));

}

// Function che calcola la somma di due vettori
//     input:
//         - a - primo vettore
//         - b - secondo vettore
//         - n - dimensione di entrambi i vettori
//     output:
//         - c - risultato ottenuto sommando componente per componente i due vettori
void sommaArrayCompPerCompCPU(double * a, double * b, int n, double * c)
{
    int i;
    for(i=0; i<n; i++)
        c[i]=a[i]+b[i];
}

// Function che calcola il prodotto tra un vettore e uno scalare
//     input:
//         - a - primo vettore
//         - b - scalare
//         - n - dimensione del vettore
//     output:
//         - ris - vettore risultato in cui l'i-esimo componente è il prodotto dell'i-esimo componente di a per b
void prodottoScalareCPU(double * a, double b, int n, double * ris)
{
    int i;
    for(i=0;i<n;i++)
        ris[i] = a[i] * b;
}

// Function che calcola il prodotto scalare tra due vettori
//     input:
//         - a - primo vettore
//         - b - secondo vettore
//         - n - dimensione di entrambi i vettori
//         - alpha - se > -1 il risultato viene moltiplicato per alpha
//     output:
//         - ris - risultato ottenuto dalla sommatoria dei prodotti dei singoli elementi dei vettori, moltiplicato per alpha (se > -1)
void prodottoScalareArrayCPU(double * a, double * b, int n, double alpha, double * ris)
{
    int i;
    for(i=0;i<n;i++)
        *ris+=(a[i]*b[i]);
	if (alpha > -1)
		*ris*=alpha;
}

// Function che calcola il prodotto di una matrice (dimensione m x n) per un vettore (dimensione n)
//     input:
//         - matrix - matrice
//         - vector - vettore
//         - m - numero di righe della matrice
//         - n - numero di colonne della matrice corrispondente con la dimensione del vettore
//         - alpha - valore per cui deve essere moltiplicato ogni elemento di out
//     output:
//         - out - vettore risultato di dimensione m in cui l'elemento i contiene il prodotto dell'i-esima riga della matrice e il vettore
void prodottoMatriceVettoreCPU(double * matrix, double * vector, int m, int n, double alpha, double * out)
{
    for(int i=0; i<m;i++)
        prodottoScalareArrayCPU(&matrix[i*n], vector, n, alpha, &out[i]);
}

// Function che calcola il prodotto [y = alpha * A * x + beta * y] in modalità sequenziale
//     input:
//         - matrix - matrice
//         - vector - vettore
//         - m - numero di righe della matrice
//         - n - numero di colonne della matrice corrispondente con la dimensione del vettore
//         - alpha - valore per cui deve essere moltiplicato la prima parte dell'operazione
//         - beta - valore per cui deve essere moltiplicato la seconda parte dell'operazione
//     output:
//         - out - vettore risultato di dimensione m in cui l'elemento è la somma dell'elemento stesso per beta, più il prodotto tra la matrice e il vettore scalato di un fattore alpha
void aAxpbySequential(double * matrix, double * vector, int m, int n, double alpha, double beta, double * out)
{
	// Dichiarazione variabili di appoggio su host
	double *out1, *out2;

	// Allocazione su host
	out1 = (double *)malloc(m*sizeof(double));
	out2 = (double *)malloc(m*sizeof(double));

	// Inizializzazione a 0 di tutti i vettori
	initializeArrayToZero(out1, m);
	initializeArrayToZero(out2, m);

	// Prima parte dell'operazione [alpha * A * x]
	prodottoMatriceVettoreCPU(matrix, vector, m, n, alpha, out1);

	// Seconda parte dell'operazione [beta * y]
	prodottoScalareCPU(out, beta, m, out2);

	// Somma dei vettori risultanti delle due precedenti operazioni
	sommaArrayCompPerCompCPU(out1, out2, m, out);

	// Libera memoria
	free(out1);
	free(out2);
}

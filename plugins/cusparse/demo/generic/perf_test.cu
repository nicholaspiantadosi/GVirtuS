#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

void initializeMatrixZero(double *matrix, int M, int N)
{
    int i, j, k=0;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < M; j++)
        {
            matrix[k++]=0;
        }
    }
}

void initializeMatrixRandomSparse(double *matrix, int M, int N, int nnz)
{
	initializeMatrixZero(matrix, M, N);
	int i=0;
	double random_number;
	for (i = 0; i < nnz;) {
		int index = (int) (M * N * ((double) rand() / (RAND_MAX + 1.0)));
		if (matrix[index]) { 
			continue;
		}
		random_number = (double) rand() / ( (double) RAND_MAX / 100 ) + 1;
		matrix[index] = random_number;
		++i;
	}
}

void initializeMatrixRandom(double *matrix, int M, int N)
{
	int i,j,k=0;
	double random_number;
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < M; j++)
		{
			random_number=(double)rand()/((double)RAND_MAX/(100)) + 1;
			matrix[k++]= random_number;
		}
	}
}

int main(int argn, char *argv[])
{
    // Host problem definition
    int m = 1000;
    int n = 1000;
    int nnz = 500;
    int lda = m;
    int ldb = m;
    int ldc = m;
    double alpha = 2.5f;
    double beta = 1.5f;

    double *hA = (double *)malloc(m*n*sizeof(double));
    initializeMatrixRandomSparse(hA, m, n, nnz);

    double *hB = (double *)malloc(m*n*sizeof(double));
    initializeMatrixRandom(hB, m, n);

    double *hC = (double *)malloc(m*n*sizeof(double));
    initializeMatrixRandom(hC, m, n);

    double threshold = 2;

    // Device memory management
    double *dA, *dB, *dC;
    double *dCsrValC;
    int *dCsrRowPtrC, *dCsrColIndC;

    clock_t tStart = clock();

    CHECK_CUDA(hipMalloc((void**) &dA,  m * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&dCsrRowPtrC, sizeof(int) * (m + 1)));

    CHECK_CUDA(hipMalloc((void**) &dB,  m * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &dC,  m * n * sizeof(double)));

    CHECK_CUDA(hipMemcpy(dA, hA, m * n * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dB, hB, m * n * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dC, hC, m * n * sizeof(double), hipMemcpyHostToDevice) );

    clock_t tCopied2Device = clock();

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrC = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL ));

    size_t pBufferSize;
    void *pBuffer = 0;

    CHECK_CUSPARSE(hipsparseDpruneDense2csr_bufferSizeExt(handle, m, n, dA, lda, &threshold, descrC, NULL, dCsrRowPtrC, NULL, &pBufferSize));

//    if(pBufferSize == 0) {
        pBufferSize = 512;
//    }

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSize));

    int nnzc;
    int *nnzTotalDevHostPtr = &nnzc;

    CHECK_CUSPARSE(hipsparseDpruneDense2csrNnz(handle, m, n, dA, lda, &threshold, descrC, dCsrRowPtrC, nnzTotalDevHostPtr, pBuffer));

    nnzc = *nnzTotalDevHostPtr;

    CHECK_CUDA(hipMalloc((void**)&dCsrValC, sizeof(double) * nnzc));
    CHECK_CUDA(hipMalloc((void**)&dCsrColIndC, sizeof(int) * nnzc));

    CHECK_CUSPARSE(hipsparseDpruneDense2csr(handle, m, n, dA, lda, &threshold, descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC, pBuffer));

    clock_t tDense2Csr = clock();

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;

    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, m, n, nnz,
                                      dCsrRowPtrC, dCsrColIndC, dCsrValC,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, m, n, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, m, n, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) );
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) );
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) );

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) );

    clock_t tSPMM = clock();

    // device result check

    double hCsrValC[nnzc];
    int hCsrRowPtrC[m + 1];
    int hCsrColIndC[nnzc];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzc * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzc * sizeof(int), hipMemcpyDeviceToHost) );

    clock_t tCopied2Host = clock();

    // step 6: free resources

    // device memory deallocation
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) );
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValC) );
    CHECK_CUDA(hipFree(dCsrRowPtrC) );
    CHECK_CUDA(hipFree(dCsrColIndC) );
    CHECK_CUDA(hipFree(dA) );
    CHECK_CUDA(hipFree(dB) );
    CHECK_CUDA(hipFree(dC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    clock_t tEnd = clock();

    printf("\nTime to copy to device:\t\t\t\t%fms", (double)(tCopied2Device - tStart)/(CLOCKS_PER_SEC/1000));
    printf("\nTime to convert matrix from dense to sparse:\t%fms", (double)(tDense2Csr - tCopied2Device)/(CLOCKS_PER_SEC/1000));
    printf("\nTime to execute SPMM operation:\t\t\t%fms", (double)(tSPMM - tDense2Csr)/(CLOCKS_PER_SEC/1000));
    printf("\nTime to copy result to host:\t\t\t%fms", (double)(tCopied2Host - tSPMM)/(CLOCKS_PER_SEC/1000));
    printf("\nTime to free resources:\t\t\t\t%fms", (double)(tEnd - tCopied2Host)/(CLOCKS_PER_SEC/1000));
    printf("\nTotal time:\t\t\t\t\t%fms\n", (double)(tEnd - tStart)/(CLOCKS_PER_SEC/1000));

    return EXIT_SUCCESS;
}

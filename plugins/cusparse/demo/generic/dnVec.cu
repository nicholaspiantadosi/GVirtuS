#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(void) {
    int64_t size = 10;

    float hValues[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};

    float *dValues;

    CHECK_CUDA(hipMalloc((void**) &dValues, size * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dValues, hValues, size * sizeof(float), hipMemcpyHostToDevice) );

    hipsparseDnVecDescr_t dnVecDescr;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dnVecDescr, size, dValues, HIP_R_32F));

    int64_t size_toverify;
    float hValues_toverify[size];
    float *dValues_toverify;
    hipDataType valueType_toverify;
    CHECK_CUDA(hipMalloc((void**) &dValues_toverify, size * sizeof(float)));

    CHECK_CUSPARSE(hipsparseDnVecGet(dnVecDescr, &size_toverify, (void**)&dValues_toverify, &valueType_toverify));

    CHECK_CUDA(hipMemcpy(hValues_toverify, dValues_toverify, size * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    if (size_toverify != size) {
        correct = 0;
    }
    for (int i = 0; i < size; i++) {
        if((fabs(hValues_toverify[i] - hValues[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (valueType_toverify != HIP_R_32F) {
        correct = 0;
    }

    float hValues2[] = {11, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float *dValues2;
    CHECK_CUDA(hipMalloc((void**) &dValues2, size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dValues2, hValues2, size * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUSPARSE(hipsparseDnVecSetValues(dnVecDescr, dValues2));
    float hValues2_toverify[size];
    float *dValues2_toverify;
    CHECK_CUDA(hipMalloc((void**) &dValues2_toverify, size * sizeof(float)));
    CHECK_CUSPARSE(hipsparseDnVecGetValues(dnVecDescr, (void**)&dValues2_toverify));
    CHECK_CUDA(hipMemcpy(hValues2_toverify, dValues2_toverify, size * sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < size; i++) {
        if((fabs(hValues2_toverify[i] - hValues2[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("dnVec test PASSED\n");
    else
        printf("dnVec test FAILED: wrong result\n");

    CHECK_CUSPARSE(hipsparseDestroyDnVec(dnVecDescr));
    CHECK_CUDA(hipFree(dValues) );

    return EXIT_SUCCESS;
}

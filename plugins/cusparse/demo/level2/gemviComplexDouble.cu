#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition - x = [1.0, 0.0, 0.0, 2.0, 3.0, 0.0, 4.0]
    hipDoubleComplex hX[] = { make_hipDoubleComplex(1,0), make_hipDoubleComplex(2,0), make_hipDoubleComplex(3,0), make_hipDoubleComplex(4,0) };
    int hXind[] = {0, 3, 4, 6};
    hipDoubleComplex hA[] = { make_hipDoubleComplex(1,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(2,0), make_hipDoubleComplex(3,0),
                       make_hipDoubleComplex(0,0), make_hipDoubleComplex(4,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(0,0),
                       make_hipDoubleComplex(5,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(6,0), make_hipDoubleComplex(7,0),
                       make_hipDoubleComplex(0,0), make_hipDoubleComplex(8,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(9,0)};
    int m = 4;
    int n = 4;
    int nnz = 9;
    int lda = m;
    hipDoubleComplex alpha = make_hipDoubleComplex(1,0);
    hipDoubleComplex beta = make_hipDoubleComplex(0,0);
    hipDoubleComplex hY[] = { make_hipDoubleComplex(0,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(0,0), make_hipDoubleComplex(0,0) };
    hipDoubleComplex hY_result[] = { make_hipDoubleComplex(1,0), make_hipDoubleComplex(16,0), make_hipDoubleComplex(2,0), make_hipDoubleComplex(21,0) };

    // Device memory management
    hipDoubleComplex *dA;
    hipDoubleComplex *dX, *dY;
    int *dXind;

    CHECK_CUDA( hipMalloc((void**) &dA,  m * n * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dX, n * sizeof(hipDoubleComplex)) );
    CHECK_CUDA( hipMalloc((void**) &dXind, n * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dY, m * sizeof(hipDoubleComplex)) );
    
    CHECK_CUDA( hipMemcpy(dA, hA, m * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dX, hX, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dXind, hXind, n * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dY, hY, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    int pBufferSize = 0;
    void* dBuffer = NULL;

    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseZgemvi_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,m, n, nnz, &pBufferSize));
    CHECK_CUSPARSE(hipsparseZgemvi(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, &alpha, dA, lda, nnz, dX, dXind, &beta, dY, HIPSPARSE_INDEX_BASE_ZERO, dBuffer));

    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if (hY[i].x != hY_result[i].x) { // direct cuDoubleComplexing point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("gemvi test PASSED\n");
    else
        printf("gemvi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dX) );
    CHECK_CUDA( hipFree(dXind) );
    CHECK_CUDA( hipFree(dY) );
    return EXIT_SUCCESS;

}
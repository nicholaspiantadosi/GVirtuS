#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, hipComplex *, int, int, int, hipComplex *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, hipComplex, hipComplex *, int *, int *, hipComplex *, hipComplex, hipComplex *);
void bsrnnz(hipsparseHandle_t, hipsparseMatDescr_t, int *, int *, int, int, int, int, int, int, int *, int &);
void csr2bsr(hipsparseHandle_t, hipsparseMatDescr_t, hipComplex *, int *, int *, int, int, int, int, int , int , int, int *,  hipComplex *, int *);
void bsrxmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, int, int, int, int, hipComplex*, hipComplex*, int*, int*, int*, int*, int, hipComplex*, hipComplex*, hipComplex*);

int main(int argn, char *argv[])
{
    // Variabili generiche
    int m, n, nnz, blockDim, mb=0, nb=0, nnzb=0, sizeOfMask=0;

    // Variabili su host
    hipComplex *matrix_host;
    hipComplex *csr_values_result;
    int *csr_columns_result, *csr_offsets_result;
    hipComplex *bsr_values_result;
    int *bsr_mask, *bsr_columns_result, *bsr_offsets_result, *bsr_offsets_result_start, *bsr_offsets_result_end;
    hipComplex *x_host, *y_result_bsr;
    hipComplex alpha, beta;
    hipComplex *matrix_host_sequential;
    hipComplex *y_result_sequential;

    srand(time(0));

    m = 4;
    n = 5;
    nnz = 9;
    blockDim = 2;
    alpha = make_hipComplex(3, 0);
    beta = make_hipComplex(2, 0);
    sizeOfMask = 1;

    // Allocazione memoria sull'host
    matrix_host = (hipComplex *)malloc((m*n+1)*sizeof(hipComplex));
    csr_values_result=(hipComplex *)malloc(nnz * sizeof(hipComplex));
    csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
    csr_columns_result=(int *)malloc(nnz * sizeof(int));
    x_host = (hipComplex *)malloc(n*sizeof(hipComplex));
    y_result_bsr = (hipComplex *)malloc(m*sizeof(hipComplex));
    matrix_host_sequential = (hipComplex *)malloc((m*n+1)*sizeof(hipComplex));
    y_result_sequential = (hipComplex *)malloc(m*sizeof(hipComplex));

    // Inizializzazione variabili sull'host
    initializeMatrixRandomSparseC(matrix_host, m, n, nnz);
    initializeArrayRandomC(x_host, n);
    if (beta.x > 0)
        initializeArrayRandomC(y_result_bsr, m);
    else
        initializeArrayToZeroC(y_result_bsr, m);
    copyArrayC(y_result_bsr, y_result_sequential, m);

    // Swap formato matrice per calcolo sequenziale su host
    swapMatrixC(matrix_host, m, n, matrix_host_sequential);

    printf("Matrice sparsa:\n");
    stampaMatrixC(matrix_host, m, n);
    printf("\n");

    printf("Array x:\n");
    stampaArrayC(x_host, n);
    printf("\n");

    printf("Matrice in formato denso per calcolo sequenziale:\n");
    stampaMatrixC1D(matrix_host_sequential, m, n);
    printf("\n");

    printf("Array y:\n");
    stampaArrayC(y_result_bsr, m);
    printf("\n");

    // Dichiarazione dell'handle per CUSPARSE
    hipsparseHandle_t handle;

    // Creazione dell'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Creazione della struttura della matrice con relative proprietà e forma
    hipsparseMatDescr_t descr = 0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Conversione matrice in formato CSR
    mat2csr(handle, descr, matrix_host, m, n, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

    //Stampa matrice convertita in formato CSR
    printf("Matrice sparsa convertita in formato CSR\n");
    printf("\t csrValA:\t");
    stampaArrayC(csr_values_result, nnz);
    printf("\t csrRowPtrA:\t");
    stampaArray(csr_offsets_result, (m + 1));
    printf("\t csrColIndA:\t");
    stampaArray(csr_columns_result, nnz);
    printf("\n");

    // Calcolo mb e nb a partire da blockDim
    mb = (m + blockDim - 1) / blockDim;
    nb = (n + blockDim - 1) / blockDim;

    // Allocazione variabili su host
    bsr_offsets_result=(int *)malloc((mb + 1) * sizeof(int));
    bsr_offsets_result_start=(int *)malloc((mb) * sizeof(int));
    bsr_offsets_result_end=(int *)malloc((mb) * sizeof(int));

    // Calcolo bsrRowPtrC e blocchi diversi da zero
    bsrnnz(handle, descr, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, bsr_offsets_result, nnzb);

    // Allocazione variabili su host sulla base del numero di blocchi diversi da zero
    bsr_columns_result=(int *)malloc(nnzb * sizeof(int));
    bsr_mask=(int *)malloc((sizeOfMask) * sizeof(int));
    bsr_values_result=(hipComplex *)malloc((blockDim * blockDim) * nnzb * sizeof(hipComplex));

    // Conversione da formato CSR a BSR
    csr2bsr(handle, descr, csr_values_result, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, nnzb, bsr_offsets_result, bsr_values_result, bsr_columns_result);

    initializeArrayTo2(bsr_mask, sizeOfMask);
    for(int i=0;i<mb;i++)
    {
        bsr_offsets_result_start[i]=bsr_offsets_result[i];
        bsr_offsets_result_end[mb - i - 1]=bsr_offsets_result[mb - i];
    }

    // Stampa matrice convertita in formato BSRX
    printf("Matrice sparsa convertita in formato BSRX\n");
    printf("\t bsrVal:\t");
    stampaArrayC(bsr_values_result, (blockDim * blockDim) * nnzb);
    printf("\t bsrMaskPtr:\t");
    stampaArray(bsr_mask, sizeOfMask);
    printf("\t bsrRowPtr:\t");
    stampaArray(bsr_offsets_result_start, (mb));
    printf("\t bsrRowPtrEnd:\t");
    stampaArray(bsr_offsets_result_end, (mb));
    printf("\t bsrColInd:\t");
    stampaArray(bsr_columns_result, nnzb);
    printf("\n");

    bsrxmv(handle, descr, m, n, nnz, sizeOfMask, mb, nb, nnzb, &alpha, bsr_values_result, bsr_mask, bsr_offsets_result_start, bsr_offsets_result_end, bsr_columns_result, blockDim, x_host, &beta, y_result_bsr);

    //Stampa array risultato dall'operazione bsrxmv tra matrice in formato BSR, vettore x_host, alpha e beta
    printf("Vettore risultato dall'operazione bsrxmv\n");
    stampaArrayC(y_result_bsr, m);
    printf("\n");

    //Libera la memoria sull'host
    free(csr_values_result);
    free(csr_offsets_result);
    free(csr_columns_result);
    free(bsr_values_result);
    free(bsr_mask);
    free(bsr_offsets_result);
    free(bsr_offsets_result_start);
    free(bsr_offsets_result_end);
    free(bsr_columns_result);
    free(matrix_host);
    free(matrix_host_sequential);

    // Termina l'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

}

void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipComplex * matrix, int m, int n, int nnz, hipComplex * csrValA, int * csrRowPtrA, int * csrColIndA)
{
    int nnz_total = 0;

    // Variabili su device
    hipComplex *matrix_device;
    hipComplex *csr_values_device;
    int *csr_columns_device, *csr_offsets_device;
    int *nnz_per_row;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(hipComplex), hipMemcpyHostToDevice));

    // Calcolo valori diversi da zero
    CHECK_CUSPARSE(hipsparseCnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

    // Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseCnnz()
    if (nnz != nnz_total) {
        printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
        exit(EXIT_FAILURE);
    }

    // Conversione matrice in formato CSR
    CHECK_CUSPARSE(hipsparseCdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(hipComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
    CHECK_CUDA(hipFree(nnz_per_row));
}

void bsrnnz(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int * bsrRowPtrC, int &nnzb)
{
    // Variabili su host
    int base;
    int *nnzTotalBsr = &nnzb;

    // Variabili su device
    int *bsrRowPtrC_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia variabili da host a device
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Calcolo del numero dei blocchi diversi da zero per il formato BSR
    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrRowPtrC_device, nnzTotalBsr));

    // Controllo sul valore dei blocchi calcolato, se null lo calcolo sulla base degli indici dei blocchi e il numero dei blocchi per riga
    if (NULL != nnzTotalBsr)
    {
        nnzb = *nnzTotalBsr;
    }
    else
    {
        CHECK_CUDA(hipMemcpy(&nnzb, bsrRowPtrC_device + mb, sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(&base, bsrRowPtrC_device, sizeof(int), hipMemcpyDeviceToHost));
        nnzb -= base;
    }

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrRowPtrC, bsrRowPtrC_device, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

void csr2bsr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipComplex * csrValA, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int nnzb, int * bsrRowPtrC, hipComplex * bsrValC, int * bsrColIndC)
{
    // Variabili su device
    hipComplex *bsrValC_device;
    int *bsrRowPtrC_device, *bsrColIndC_device;
    hipComplex *csrValA_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device sulla base del numero di blocchi diversi da zero
    CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrColIndC_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia da host a device
    CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Conversione da CSR a BSR
    CHECK_CUSPARSE(hipsparseCcsr2bsr(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrValC, bsrValC_device, ((blockDim * blockDim) * nnzb) * sizeof(hipComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(bsrColIndC, bsrColIndC_device, nnzb * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrValC_device));
    CHECK_CUDA(hipFree(bsrColIndC_device));
    //CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrValA_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

void bsrxmv(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int n, int nnz, int sizeOfMask, int mb, int nb, int nnzb, hipComplex* alpha, hipComplex* bsrVal, int* bsrMaskPtr, int* bsrRowPtr, int* bsrEndPtr, int* bsrColInd, int blockDim, hipComplex* x, hipComplex* beta, hipComplex* y)
{
    // Variabili su device
    hipComplex *x_device, *y_device;
    hipComplex *bsrVal_device;
    int *bsrMaskPtr_device, *bsrRowPtr_device, *bsrEndPtr_device, *bsrColInd_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &bsrVal_device, (blockDim * blockDim) * nnzb * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &bsrMaskPtr_device, sizeOfMask * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrRowPtr_device, mb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrEndPtr_device, mb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrColInd_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&x_device, (nb*blockDim) * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**)&y_device, (mb*blockDim) * sizeof(hipComplex)));

    // Copia valori da host a device
    CHECK_CUDA(hipMemcpy(x_device, x, n * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrVal_device, bsrVal, (blockDim * blockDim) * nnzb * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrMaskPtr_device, bsrMaskPtr, sizeOfMask * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtr_device, bsrRowPtr, mb * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrEndPtr_device, bsrEndPtr, mb * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrColInd_device, bsrColInd, nnzb * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseCbsrxmv(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, sizeOfMask, mb, nb, nnzb, alpha, descr, bsrVal_device, bsrMaskPtr_device, bsrRowPtr_device, bsrEndPtr_device, bsrColInd_device, blockDim, x_device, beta, y_device));

    CHECK_CUDA(hipMemcpy(y, y_device, (mb*blockDim) * sizeof(hipComplex), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(bsrVal_device));
    CHECK_CUDA(hipFree(bsrMaskPtr_device));
    CHECK_CUDA(hipFree(bsrColInd_device));
    CHECK_CUDA(hipFree(bsrRowPtr_device));
    CHECK_CUDA(hipFree(bsrEndPtr_device));
    CHECK_CUDA(hipFree(x_device));
    CHECK_CUDA(hipFree(y_device));

}
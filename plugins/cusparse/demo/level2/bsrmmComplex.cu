#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    hipComplex hBsrValA[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                            make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                            make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0),
                            make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(4, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(8, 0)};

    hipComplex hC[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(1, 0), make_hipComplex(0, 0),
                      make_hipComplex(0, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                      make_hipComplex(4, 0), make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                      make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex hC_result[] = {make_hipComplex(10, 0), make_hipComplex(8, 0), make_hipComplex(27, 0), make_hipComplex(28, 0),
                             make_hipComplex(12, 0), make_hipComplex(2, 0), make_hipComplex(31, 0), make_hipComplex(16, 0),
                             make_hipComplex(32, 0), make_hipComplex(5, 0), make_hipComplex(72, 0), make_hipComplex(24, 0),
                             make_hipComplex(44, 0), make_hipComplex(7, 0), make_hipComplex(98, 0), make_hipComplex(32, 0)};

    hipComplex alpha = make_hipComplex(1, 0);
    hipComplex beta = make_hipComplex(1, 0);

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    hipComplex *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    hipComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    // perform C:=alpha*A*B + beta*C
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseCbsrmm(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, kb, nnzb, &alpha, descrA, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, dB, ldb, &beta, dC, ldc);

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if (hC[i].x != hC_result[i].x) { // direct cuComplexing point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
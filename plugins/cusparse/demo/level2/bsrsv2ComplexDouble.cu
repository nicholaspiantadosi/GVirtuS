#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, hipDoubleComplex *, int, int, int, hipDoubleComplex *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, hipDoubleComplex, hipDoubleComplex *, int *, int *, hipDoubleComplex *, hipDoubleComplex, hipDoubleComplex *);
void bsrnnz(hipsparseHandle_t, hipsparseMatDescr_t, int *, int *, int, int, int, int, int, int, int *, int &);
void csr2bsr(hipsparseHandle_t, hipsparseMatDescr_t, hipDoubleComplex *, int *, int *, int, int, int, int, int , int , int, int *,  hipDoubleComplex *, int *);
void bsrsv2(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, hipDoubleComplex *, int *, int *, int, hipDoubleComplex *, hipDoubleComplex *);

int main(int argn, char *argv[])
{
    // Variabili generiche
    int m, nnz, blockDim, mb=0, nnzb=0;

    // Variabili su host
    hipDoubleComplex *matrix_host;
    hipDoubleComplex *csr_values_result;
    int *csr_columns_result, *csr_offsets_result;
    hipDoubleComplex *bsr_values_result;
    int *bsr_columns_result, *bsr_offsets_result;
    hipDoubleComplex *x_host, *y_result_bsr;
    hipDoubleComplex *matrix_host_sequential;

    srand(time(0));

    m = 4;
    nnz = 5;
    blockDim = 2;

    // Allocazione memoria sull'host
    matrix_host = (hipDoubleComplex *)malloc((m*m)*sizeof(hipDoubleComplex));
    csr_values_result=(hipDoubleComplex *)malloc(nnz * sizeof(hipDoubleComplex));
    csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
    csr_columns_result=(int *)malloc(nnz * sizeof(int));
    x_host = (hipDoubleComplex *)malloc(m*sizeof(hipDoubleComplex));
    y_result_bsr = (hipDoubleComplex *)malloc(m*sizeof(hipDoubleComplex));
    matrix_host_sequential = (hipDoubleComplex *)malloc((m*m+1)*sizeof(hipDoubleComplex));

    // Inizializzazione variabili sull'host
    initializeMatrixLowerTriangularSparseRandomZ(matrix_host, m, nnz);
    initializeArrayRandomZ(x_host, m);
    initializeArrayToZeroZ(y_result_bsr, m);

    // Swap formato matrice per calcolo sequenziale su host
    swapMatrixZ(matrix_host, m, m, matrix_host_sequential);

    printf("Matrice sparsa:\n");
    stampaMatrixZ(matrix_host, m, m);
    printf("\n");

    printf("Array x:\n");
    stampaArrayZ(x_host, m);
    printf("\n");

    printf("Matrice in formato denso per calcolo sequenziale:\n");
    stampaMatrixZ1D(matrix_host_sequential, m, m);
    printf("\n");

    // Dichiarazione dell'handle per CUSPARSE
    hipsparseHandle_t handle;

    // Creazione dell'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Creazione della struttura della matrice con relative proprietà e forma
    hipsparseMatDescr_t descr = 0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_UNIT);

    // Conversione matrice in formato CSR
    mat2csr(handle, descr, matrix_host, m, m, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

    //Stampa matrice convertita in formato CSR
    printf("Matrice sparsa convertita in formato CSR\n");
    printf("\t csrValA:\t");
    stampaArrayZ(csr_values_result, nnz);
    printf("\t csrRowPtrA:\t");
    stampaArray(csr_offsets_result, (m + 1));
    printf("\t csrColIndA:\t");
    stampaArray(csr_columns_result, nnz);
    printf("\n");

    // Calcolo mb a partire da blockDim
    mb = (m + blockDim - 1) / blockDim;

    // Allocazione variabili su host
    bsr_offsets_result=(int *)malloc((mb + 1) * sizeof(int));

    // Calcolo bsrRowPtrC e blocchi diversi da zero
    bsrnnz(handle, descr, csr_offsets_result, csr_columns_result, m, m, nnz, blockDim, mb, mb, bsr_offsets_result, nnzb);

    // Allocazione variabili su host sulla base del numero di blocchi diversi da zero
    bsr_columns_result=(int *)malloc(nnzb * sizeof(int));
    bsr_values_result=(hipDoubleComplex *)malloc((blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex));

    // Conversione da formato CSR a BSR
    csr2bsr(handle, descr, csr_values_result, csr_offsets_result, csr_columns_result, m, m, nnz, blockDim, mb, mb, nnzb, bsr_offsets_result, bsr_values_result, bsr_columns_result);

    // Stampa matrice convertita in formato BSR
    printf("Matrice sparsa convertita in formato BSR\n");
    printf("\t bsrValC:\t");
    stampaArrayZ(bsr_values_result, (blockDim * blockDim) * nnzb);
    printf("\t bsrRowPtrC:\t");
    stampaArray(bsr_offsets_result, (mb + 1));
    printf("\t bsrColIndC:\t");
    stampaArray(bsr_columns_result, nnzb);
    printf("\n");

    bsrsv2(handle, descr, m, mb, nnzb, bsr_values_result, bsr_offsets_result, bsr_columns_result, blockDim, x_host, y_result_bsr);

    printf("Vettore risultato dall'operazione \n");
    stampaArrayZ(y_result_bsr, m);
    printf("\n");

    //Libera la memoria sull'host
    free(csr_values_result);
    free(csr_offsets_result);
    free(csr_columns_result);
    free(bsr_values_result);
    free(bsr_offsets_result);
    free(bsr_columns_result);
    free(matrix_host);
    free(matrix_host_sequential);

    // Termina l'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}

void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipDoubleComplex * matrix, int m, int n, int nnz, hipDoubleComplex * csrValA, int * csrRowPtrA, int * csrColIndA)
{
    int nnz_total = 0;

    // Variabili su device
    hipDoubleComplex *matrix_device;
    hipDoubleComplex *csr_values_device;
    int *csr_columns_device, *csr_offsets_device;
    int *nnz_per_row;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // Calcolo valori diversi da zero
    CHECK_CUSPARSE(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

    // Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseZnnz()
    if (nnz != nnz_total) {
        printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
        exit(EXIT_FAILURE);
    }

    // Conversione matrice in formato CSR
    CHECK_CUSPARSE(hipsparseZdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
    CHECK_CUDA(hipFree(nnz_per_row));
}

void bsrnnz(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int * bsrRowPtrC, int &nnzb)
{
    // Variabili su host
    int base;
    int *nnzTotalBsr = &nnzb;

    // Variabili su device
    int *bsrRowPtrC_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia variabili da host a device
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Calcolo del numero dei blocchi diversi da zero per il formato BSR
    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrRowPtrC_device, nnzTotalBsr));

    // Controllo sul valore dei blocchi calcolato, se null lo calcolo sulla base degli indici dei blocchi e il numero dei blocchi per riga
    if (NULL != nnzTotalBsr)
    {
        nnzb = *nnzTotalBsr;
    }
    else
    {
        CHECK_CUDA(hipMemcpy(&nnzb, bsrRowPtrC_device + mb, sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(&base, bsrRowPtrC_device, sizeof(int), hipMemcpyDeviceToHost));
        nnzb -= base;
    }

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrRowPtrC, bsrRowPtrC_device, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

void csr2bsr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipDoubleComplex * csrValA, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int nnzb, int * bsrRowPtrC, hipDoubleComplex * bsrValC, int * bsrColIndC)
{
    // Variabili su device
    hipDoubleComplex *bsrValC_device;
    int *bsrRowPtrC_device, *bsrColIndC_device;
    hipDoubleComplex *csrValA_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device sulla base del numero di blocchi diversi da zero
    CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrColIndC_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia da host a device
    CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Conversione da CSR a BSR
    CHECK_CUSPARSE(hipsparseZcsr2bsr(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrValC, bsrValC_device, ((blockDim * blockDim) * nnzb) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(bsrColIndC, bsrColIndC_device, nnzb * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrValC_device));
    CHECK_CUDA(hipFree(bsrColIndC_device));
    //CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrValA_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

// Suppose that L is m x m sparse matrix represented by BSR format,
// The number of block rows/columns is mb, and
// the number of nonzero blocks is nnzb.
// L is lower triangular with unit diagonal.
// Assumption:
// - dimension of matrix L is m(=mb*blockDim),
// - matrix L has nnz(=nnzb*blockDim*blockDim) nonzero elements,
// - handle is already created by hipsparseCreate(),
// - (d_bsrRowPtr, d_bsrColInd, d_bsrVal) is BSR of L on device memory,
// - d_x is right hand side vector on device memory.
// - d_y is solution vector on device memory.
// - d_x and d_y are of size m.
void bsrsv2(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int mb, int nnzb, hipDoubleComplex * bsrVal, int * bsrRowPtr, int * bsrColInd, int blockDim, hipDoubleComplex * x, hipDoubleComplex * y) {

    hipDoubleComplex *bsrVal_device;
    int *bsrRowPtr_device, *bsrColInd_device;
    hipDoubleComplex *x_device, *y_device;

    CHECK_CUDA(hipMalloc((void**) &bsrVal_device, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &bsrRowPtr_device, (mb + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrColInd_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &x_device, (mb*blockDim) * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &y_device, (mb*blockDim) * sizeof(hipDoubleComplex)));

    CHECK_CUDA(hipMemcpy(x_device, x, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrVal_device, bsrVal, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtr_device, bsrRowPtr, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrColInd_device, bsrColInd, nnzb * sizeof(int), hipMemcpyHostToDevice));

    bsrsv2Info_t info = 0;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipDoubleComplex alpha = make_hipDoubleComplex(1., 0);
    const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

// step 1: create a descriptor which contains
// - matrix L is base-1
// - matrix L is lower triangular
// - matrix L has unit diagonal, specified by parameter HIPSPARSE_DIAG_TYPE_UNIT
//   (L may not have all diagonal elements.)

// step 2: create a empty info structure
    hipsparseCreateBsrsv2Info(&info);

// step 3: query how much memory used in bsrsv2, and allocate the buffer
    hipsparseZbsrsv2_bufferSize(handle, dir, trans, mb, nnzb, descr,
                               bsrVal_device, bsrRowPtr_device, bsrColInd_device, blockDim, info, &pBufferSize);

// pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

// step 4: perform analysis
    hipsparseZbsrsv2_analysis(handle, dir, trans, mb, nnzb, descr,
                             bsrVal_device, bsrRowPtr_device, bsrColInd_device, blockDim,
                             info, policy, pBuffer);
// L has unit diagonal, so no structural zero is reported.
    hipsparseStatus_t status = hipsparseXbsrsv2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

// step 5: solve L*y = x
    hipsparseZbsrsv2_solve(handle, dir, trans, mb, nnzb, &alpha, descr,
                          bsrVal_device, bsrRowPtr_device, bsrColInd_device, blockDim, info,
                          x_device, y_device, policy, pBuffer);
// L has unit diagonal, so no numerical zero is reported.
    status = hipsparseXbsrsv2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    CHECK_CUDA(hipMemcpy(y, y_device, (mb*blockDim) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

// step 6: free resources
    CHECK_CUDA(hipFree(bsrVal_device));
    CHECK_CUDA(hipFree(bsrColInd_device));
    CHECK_CUDA(hipFree(bsrRowPtr_device));
    CHECK_CUDA(hipFree(x_device));
    CHECK_CUDA(hipFree(y_device));
    hipFree(pBuffer);
    hipsparseDestroyBsrsv2Info(info);
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(handle);
}
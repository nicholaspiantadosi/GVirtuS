#include <hip/hip_runtime.h>  // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>      // hipsparseSpMV
#include <stdio.h>         // printf
#include <stdlib.h>        // EXIT_FAILURE
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}
//#ifndef USE_FLOAT
//#define DTYPE HIP_R_16F
//typedef half dtype;
//#else
#define DTYPE HIP_R_32F
typedef float dtype;
//#endif

int main() {
    // Host problem definition
    const int A_num_rows = 4;
    const int A_num_cols = 4;
    const int A_num_nnz  = 9;
    int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    dtype hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                              6.0f, 7.0f, 8.0f, 9.0f };
    dtype hX[]            = { 1.0f, 2.0f, 3.0f, 4.0f };
    dtype yTemp[]         = { 19.0f, 0.0f, 0.0f, 0.0f };
    const dtype result[]  = { 19.0f, 8.0f, 51.0f, 52.0f };
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    dtype *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_nnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_values, A_num_nnz * sizeof(dtype)) )
    CHECK_CUDA( hipMalloc((void**) &dX, A_num_cols * sizeof(dtype)) )
    CHECK_CUDA( hipMalloc((void**) &dY, A_num_rows * sizeof(dtype)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_num_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                           A_num_nnz * sizeof(dtype), hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_rows * sizeof(dtype),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, yTemp, A_num_rows * sizeof(dtype),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = 0;
    void*  dBuffer    = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
#ifdef USE_SPMV
    float alpha = 1.0f;
    float beta  = 0.0f;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_num_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, DTYPE) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, DTYPE) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, DTYPE) )
    // allocate an external buffer if needed
    //printf("\nalpha: %f\n", alpha);
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, DTYPE,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    //printf("\nbufferSize: %d\n", bufferSize);
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, DTYPE,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
#else
    dtype alpha = 1.0f;
    dtype beta  = 0.0f;
    hipsparseMatDescr_t descrA;
    CHECK_CUSPARSE( hipsparseCreateMatDescr(&descrA) );
    CHECK_CUSPARSE( hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    CHECK_CUSPARSE( cusparseCsrmvEx_bufferSize(
            handle,
            CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            A_num_rows, A_num_cols, A_num_nnz,
            &alpha, DTYPE,
            descrA, dA_values, DTYPE, dA_csrOffsets, dA_columns,
            dX, DTYPE,
            &beta, DTYPE,
            dY, DTYPE,
            DTYPE,
            &bufferSize) );
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) );
    CHECK_CUSPARSE( cusparseCsrmvEx(
            handle,
            CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            A_num_rows, A_num_cols, A_num_nnz,
            &alpha, DTYPE,
            descrA, dA_values, DTYPE, dA_csrOffsets, dA_columns,
            dX, DTYPE,
            &beta, DTYPE,
            dY, DTYPE,
            DTYPE,
            dBuffer) );  // this is line 121
#endif
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    dtype hY[A_num_rows];
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(dtype),
                           hipMemcpyDeviceToHost) )

    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        if (((float)hY[i]) != ((float)result[i])) {
            correct = 0;
            printf("hY[%d] = %f, result[%d] = %f\n", i, (float)hY[i], i, (float)result[i]);
            //break;
        }
    }
    if (correct)
        printf("spmv_example test PASSED\n");
    else
        printf("spmv_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return EXIT_SUCCESS;
}
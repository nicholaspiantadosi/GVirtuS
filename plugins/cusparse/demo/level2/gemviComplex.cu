#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition - x = [1.0, 0.0, 0.0, 2.0, 3.0, 0.0, 4.0]
    hipComplex hX[] = { make_hipComplex(1,0), make_hipComplex(2,0), make_hipComplex(3,0), make_hipComplex(4,0) };
    int hXind[] = {0, 3, 4, 6};
    hipComplex hA[] = { make_hipComplex(1,0), make_hipComplex(0,0), make_hipComplex(2,0), make_hipComplex(3,0),
                       make_hipComplex(0,0), make_hipComplex(4,0), make_hipComplex(0,0), make_hipComplex(0,0),
                       make_hipComplex(5,0), make_hipComplex(0,0), make_hipComplex(6,0), make_hipComplex(7,0),
                       make_hipComplex(0,0), make_hipComplex(8,0), make_hipComplex(0,0), make_hipComplex(9,0)};
    int m = 4;
    int n = 4;
    int nnz = 9;
    int lda = m;
    hipComplex alpha = make_hipComplex(1,0);
    hipComplex beta = make_hipComplex(0,0);
    hipComplex hY[] = { make_hipComplex(0,0), make_hipComplex(0,0), make_hipComplex(0,0), make_hipComplex(0,0) };
    hipComplex hY_result[] = { make_hipComplex(1,0), make_hipComplex(16,0), make_hipComplex(2,0), make_hipComplex(21,0) };

    // Device memory management
    hipComplex *dA;
    hipComplex *dX, *dY;
    int *dXind;

    CHECK_CUDA( hipMalloc((void**) &dA,  m * n * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dX, n * sizeof(hipComplex)) );
    CHECK_CUDA( hipMalloc((void**) &dXind, n * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dY, m * sizeof(hipComplex)) );
    
    CHECK_CUDA( hipMemcpy(dA, hA, m * n * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dX, hX, n * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dXind, hXind, n * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dY, hY, m * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    int pBufferSize = 0;
    void* dBuffer = NULL;

    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseCgemvi_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,m, n, nnz, &pBufferSize));
    CHECK_CUSPARSE(hipsparseCgemvi(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, &alpha, dA, lda, nnz, dX, dXind, &beta, dY, HIPSPARSE_INDEX_BASE_ZERO, dBuffer));

    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, m * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if (hY[i].x != hY_result[i].x) { // direct cuComplexing point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("gemvi test PASSED\n");
    else
        printf("gemvi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) );
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dX) );
    CHECK_CUDA( hipFree(dXind) );
    CHECK_CUDA( hipFree(dY) );
    return EXIT_SUCCESS;

}
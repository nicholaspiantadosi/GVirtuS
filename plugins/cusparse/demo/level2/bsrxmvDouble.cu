#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, double *, int, int, int, double *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, double, double *, int *, int *, double *, double, double *);
void bsrnnz(hipsparseHandle_t, hipsparseMatDescr_t, int *, int *, int, int, int, int, int, int, int *, int &);
void csr2bsr(hipsparseHandle_t, hipsparseMatDescr_t, double *, int *, int *, int, int, int, int, int , int , int, int *,  double *, int *);
void bsrxmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, int, int, int, int, double*, double*, int*, int*, int*, int*, int, double*, double*, double*);

int main(int argn, char *argv[])
{
    // Variabili generiche
    int m, n, nnz, blockDim, mb=0, nb=0, nnzb=0, sizeOfMask=0;

    // Variabili su host
    double *matrix_host;
    double *csr_values_result;
    int *csr_columns_result, *csr_offsets_result;
    double *bsr_values_result;
    int *bsr_mask, *bsr_columns_result, *bsr_offsets_result, *bsr_offsets_result_start, *bsr_offsets_result_end;
    double *x_host, *y_result_bsr;
    double alpha, beta;
    double *matrix_host_sequential;
    double *y_result_sequential;

    srand(time(0));

    m = 4;
    n = 5;
    nnz = 9;
    blockDim = 2;
    alpha = 3;
    beta = 2;
    sizeOfMask = 1;

    // Allocazione memoria sull'host
    matrix_host = (double *)malloc((m*n+1)*sizeof(double));
    csr_values_result=(double *)malloc(nnz * sizeof(double));
    csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
    csr_columns_result=(int *)malloc(nnz * sizeof(int));
    x_host = (double *)malloc(n*sizeof(double));
    y_result_bsr = (double *)malloc(m*sizeof(double));
    matrix_host_sequential = (double *)malloc((m*n+1)*sizeof(double));
    y_result_sequential = (double *)malloc(m*sizeof(double));

    // Inizializzazione variabili sull'host
    initializeMatrixRandomSparseD(matrix_host, m, n, nnz);
    initializeArrayRandomD(x_host, n);
    if (beta > 0)
        initializeArrayRandomD(y_result_bsr, m);
    else
        initializeArrayToZeroD(y_result_bsr, m);
    copyArrayD(y_result_bsr, y_result_sequential, m);

    // Swap formato matrice per calcolo sequenziale su host
    swapMatrixD(matrix_host, m, n, matrix_host_sequential);

    printf("Matrice sparsa:\n");
    stampaMatrixD(matrix_host, m, n);
    printf("\n");

    printf("Array x:\n");
    stampaArrayD(x_host, n);
    printf("\n");

    printf("Matrice in formato denso per calcolo sequenziale:\n");
    stampaMatrixD1D(matrix_host_sequential, m, n);
    printf("\n");

    printf("Array y:\n");
    stampaArrayD(y_result_bsr, m);
    printf("\n");

    // Dichiarazione dell'handle per CUSPARSE
    hipsparseHandle_t handle;

    // Creazione dell'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Creazione della struttura della matrice con relative proprietà e forma
    hipsparseMatDescr_t descr = 0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Conversione matrice in formato CSR
    mat2csr(handle, descr, matrix_host, m, n, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

    //Stampa matrice convertita in formato CSR
    printf("Matrice sparsa convertita in formato CSR\n");
    printf("\t csrValA:\t");
    stampaArrayD(csr_values_result, nnz);
    printf("\t csrRowPtrA:\t");
    stampaArray(csr_offsets_result, (m + 1));
    printf("\t csrColIndA:\t");
    stampaArray(csr_columns_result, nnz);
    printf("\n");

    // Calcolo mb e nb a partire da blockDim
    mb = (m + blockDim - 1) / blockDim;
    nb = (n + blockDim - 1) / blockDim;

    // Allocazione variabili su host
    bsr_offsets_result=(int *)malloc((mb + 1) * sizeof(int));
    bsr_offsets_result_start=(int *)malloc((mb) * sizeof(int));
    bsr_offsets_result_end=(int *)malloc((mb) * sizeof(int));

    // Calcolo bsrRowPtrC e blocchi diversi da zero
    bsrnnz(handle, descr, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, bsr_offsets_result, nnzb);

    // Allocazione variabili su host sulla base del numero di blocchi diversi da zero
    bsr_columns_result=(int *)malloc(nnzb * sizeof(int));
    bsr_mask=(int *)malloc((sizeOfMask) * sizeof(int));
    bsr_values_result=(double *)malloc((blockDim * blockDim) * nnzb * sizeof(double));

    // Conversione da formato CSR a BSR
    csr2bsr(handle, descr, csr_values_result, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, nnzb, bsr_offsets_result, bsr_values_result, bsr_columns_result);

    initializeArrayTo2(bsr_mask, sizeOfMask);
    for(int i=0;i<mb;i++)
    {
        bsr_offsets_result_start[i]=bsr_offsets_result[i];
        bsr_offsets_result_end[mb - i - 1]=bsr_offsets_result[mb - i];
    }

    // Stampa matrice convertita in formato BSRX
    printf("Matrice sparsa convertita in formato BSRX\n");
    printf("\t bsrVal:\t");
    stampaArrayD(bsr_values_result, (blockDim * blockDim) * nnzb);
    printf("\t bsrMaskPtr:\t");
    stampaArray(bsr_mask, sizeOfMask);
    printf("\t bsrRowPtr:\t");
    stampaArray(bsr_offsets_result_start, (mb));
    printf("\t bsrRowPtrEnd:\t");
    stampaArray(bsr_offsets_result_end, (mb));
    printf("\t bsrColInd:\t");
    stampaArray(bsr_columns_result, nnzb);
    printf("\n");

    bsrxmv(handle, descr, m, n, nnz, sizeOfMask, mb, nb, nnzb, &alpha, bsr_values_result, bsr_mask, bsr_offsets_result_start, bsr_offsets_result_end, bsr_columns_result, blockDim, x_host, &beta, y_result_bsr);

    //Stampa array risultato dall'operazione bsrxmv tra matrice in formato BSR, vettore x_host, alpha e beta
    printf("Vettore risultato dall'operazione bsrxmv\n");
    stampaArrayD(y_result_bsr, m);
    printf("\n");

    //Libera la memoria sull'host
    free(csr_values_result);
    free(csr_offsets_result);
    free(csr_columns_result);
    free(bsr_values_result);
    free(bsr_mask);
    free(bsr_offsets_result);
    free(bsr_offsets_result_start);
    free(bsr_offsets_result_end);
    free(bsr_columns_result);
    free(matrix_host);
    free(matrix_host_sequential);

    // Termina l'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

}

void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, double * matrix, int m, int n, int nnz, double * csrValA, int * csrRowPtrA, int * csrColIndA)
{
    int nnz_total = 0;

    // Variabili su device
    double *matrix_device;
    double *csr_values_device;
    int *csr_columns_device, *csr_offsets_device;
    int *nnz_per_row;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(double), hipMemcpyHostToDevice));

    // Calcolo valori diversi da zero
    CHECK_CUSPARSE(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

    // Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseSnnz()
    if (nnz != nnz_total) {
        printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
        exit(EXIT_FAILURE);
    }

    // Conversione matrice in formato CSR
    CHECK_CUSPARSE(hipsparseDdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
    CHECK_CUDA(hipFree(nnz_per_row));
}

void bsrnnz(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int * bsrRowPtrC, int &nnzb)
{
    // Variabili su host
    int base;
    int *nnzTotalBsr = &nnzb;

    // Variabili su device
    int *bsrRowPtrC_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia variabili da host a device
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Calcolo del numero dei blocchi diversi da zero per il formato BSR
    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrRowPtrC_device, nnzTotalBsr));

    // Controllo sul valore dei blocchi calcolato, se null lo calcolo sulla base degli indici dei blocchi e il numero dei blocchi per riga
    if (NULL != nnzTotalBsr)
    {
        nnzb = *nnzTotalBsr;
    }
    else
    {
        CHECK_CUDA(hipMemcpy(&nnzb, bsrRowPtrC_device + mb, sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(&base, bsrRowPtrC_device, sizeof(int), hipMemcpyDeviceToHost));
        nnzb -= base;
    }

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrRowPtrC, bsrRowPtrC_device, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

void csr2bsr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, double * csrValA, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int nnzb, int * bsrRowPtrC, double * bsrValC, int * bsrColIndC)
{
    // Variabili su device
    double *bsrValC_device;
    int *bsrRowPtrC_device, *bsrColIndC_device;
    double *csrValA_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device sulla base del numero di blocchi diversi da zero
    CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrColIndC_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia da host a device
    CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Conversione da CSR a BSR
    CHECK_CUSPARSE(hipsparseDcsr2bsr(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrValC, bsrValC_device, ((blockDim * blockDim) * nnzb) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(bsrColIndC, bsrColIndC_device, nnzb * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrValC_device));
    CHECK_CUDA(hipFree(bsrColIndC_device));
    //CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrValA_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

void bsrxmv(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int n, int nnz, int sizeOfMask, int mb, int nb, int nnzb, double* alpha, double* bsrVal, int* bsrMaskPtr, int* bsrRowPtr, int* bsrEndPtr, int* bsrColInd, int blockDim, double* x, double* beta, double* y)
{
    // Variabili su device
    double *x_device, *y_device;
    double *bsrVal_device;
    int *bsrMaskPtr_device, *bsrRowPtr_device, *bsrEndPtr_device, *bsrColInd_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &bsrVal_device, (blockDim * blockDim) * nnzb * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &bsrMaskPtr_device, sizeOfMask * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrRowPtr_device, mb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrEndPtr_device, mb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrColInd_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&x_device, (nb*blockDim) * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&y_device, (mb*blockDim) * sizeof(double)));

    // Copia valori da host a device
    CHECK_CUDA(hipMemcpy(x_device, x, n * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrVal_device, bsrVal, (blockDim * blockDim) * nnzb * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrMaskPtr_device, bsrMaskPtr, sizeOfMask * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtr_device, bsrRowPtr, mb * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrEndPtr_device, bsrEndPtr, mb * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrColInd_device, bsrColInd, nnzb * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseDbsrxmv(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, sizeOfMask, mb, nb, nnzb, alpha, descr, bsrVal_device, bsrMaskPtr_device, bsrRowPtr_device, bsrEndPtr_device, bsrColInd_device, blockDim, x_device, beta, y_device));

    CHECK_CUDA(hipMemcpy(y, y_device, (mb*blockDim) * sizeof(double), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(bsrVal_device));
    CHECK_CUDA(hipFree(bsrMaskPtr_device));
    CHECK_CUDA(hipFree(bsrColInd_device));
    CHECK_CUDA(hipFree(bsrRowPtr_device));
    CHECK_CUDA(hipFree(bsrEndPtr_device));
    CHECK_CUDA(hipFree(x_device));
    CHECK_CUDA(hipFree(y_device));

}
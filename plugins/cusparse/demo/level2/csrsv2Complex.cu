#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, hipComplex *, int, int, int, hipComplex *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, hipComplex, hipComplex *, int *, int *, hipComplex *, hipComplex, hipComplex *);
void csrsv2(hipsparseHandle_t, hipsparseMatDescr_t, int, int, hipComplex *, int *, int *, hipComplex *, hipComplex *);

int main(int argn, char *argv[])
{
    // Variabili generiche
    int m, nnz;

    // Variabili su host
    hipComplex *matrix_host;
    hipComplex *csr_values_result;
    int *csr_columns_result, *csr_offsets_result;
    hipComplex *x_host, *y_result_csr;
    hipComplex *matrix_host_sequential;

    srand(time(0));

    m = 4;
    nnz = 5;

    // Allocazione memoria sull'host
    matrix_host = (hipComplex *)malloc((m*m)*sizeof(hipComplex));
    csr_values_result=(hipComplex *)malloc(nnz * sizeof(hipComplex));
    csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
    csr_columns_result=(int *)malloc(nnz * sizeof(int));
    x_host = (hipComplex *)malloc(m*sizeof(hipComplex));
    y_result_csr = (hipComplex *)malloc(m*sizeof(hipComplex));
    matrix_host_sequential = (hipComplex *)malloc((m*m+1)*sizeof(hipComplex));

    // Inizializzazione variabili sull'host
    initializeMatrixLowerTriangularSparseRandomC(matrix_host, m, nnz);
    initializeArrayRandomC(x_host, m);
    initializeArrayToZeroC(y_result_csr, m);

    // Swap formato matrice per calcolo sequenziale su host
    swapMatrixC(matrix_host, m, m, matrix_host_sequential);

    printf("Matrice sparsa:\n");
    stampaMatrixC(matrix_host, m, m);
    printf("\n");

    printf("Array x:\n");
    stampaArrayC(x_host, m);
    printf("\n");

    printf("Matrice in formato denso per calcolo sequenziale:\n");
    stampaMatrixC1D(matrix_host_sequential, m, m);
    printf("\n");

    // Dichiarazione dell'handle per CUSPARSE
    hipsparseHandle_t handle;

    // Creazione dell'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Creazione della struttura della matrice con relative proprietà e forma
    hipsparseMatDescr_t descr = 0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_UNIT);

    // Conversione matrice in formato CSR
    mat2csr(handle, descr, matrix_host, m, m, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

    //Stampa matrice convertita in formato CSR
    printf("Matrice sparsa convertita in formato CSR\n");
    printf("\t csrValA:\t");
    stampaArrayC(csr_values_result, nnz);
    printf("\t csrRowPtrA:\t");
    stampaArray(csr_offsets_result, (m + 1));
    printf("\t csrColIndA:\t");
    stampaArray(csr_columns_result, nnz);
    printf("\n");

    csrsv2(handle, descr, m, nnz, csr_values_result, csr_offsets_result, csr_columns_result, x_host, y_result_csr);

    printf("Vettore risultato dall'operazione \n");
    stampaArrayC(y_result_csr, m);
    printf("\n");

    //Libera la memoria sull'host
    free(csr_values_result);
    free(csr_offsets_result);
    free(csr_columns_result);
    free(matrix_host);
    free(matrix_host_sequential);

    // Termina l'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
}

void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipComplex * matrix, int m, int n, int nnz, hipComplex * csrValA, int * csrRowPtrA, int * csrColIndA)
{
    int nnz_total = 0;

    // Variabili su device
    hipComplex *matrix_device;
    hipComplex *csr_values_device;
    int *csr_columns_device, *csr_offsets_device;
    int *nnz_per_row;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(hipComplex), hipMemcpyHostToDevice));

    // Calcolo valori diversi da zero
    CHECK_CUSPARSE(hipsparseCnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

    // Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseSnnz()
    if (nnz != nnz_total) {
        printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
        exit(EXIT_FAILURE);
    }

    // Conversione matrice in formato CSR
    CHECK_CUSPARSE(hipsparseCdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(hipComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
    CHECK_CUDA(hipFree(nnz_per_row));
}

// Suppose that L is m x m sparse matrix represented by CSR format,
// L is lower triangular with unit diagonal.
// Assumption:
// - dimension of matrix L is m,
// - matrix L has nnz number zero elements,
// - handle is already created by hipsparseCreate(),
// - (d_csrRowPtr, d_csrColInd, d_csrVal) is CSR of L on device memory,
// - d_x is right hand side vector on device memory,
// - d_y is solution vector on device memory.
void csrsv2(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int nnz, hipComplex * csrVal, int * csrRowPtr, int * csrColInd, hipComplex * x, hipComplex * y) {

    hipComplex *d_csrVal;
    int *d_csrRowPtr, *d_csrColInd;
    hipComplex *d_x, *d_y;

    CHECK_CUDA(hipMalloc((void**) &d_csrVal, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &d_csrRowPtr, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &d_csrColInd, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &d_x, m * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &d_y, m * sizeof(hipComplex)));

    CHECK_CUDA(hipMemcpy(d_x, x, m * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, y, m * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrVal, csrVal, nnz * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrRowPtr, csrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice));

    csrsv2Info_t info = 0;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipComplex alpha = make_hipComplex(1., 0);
    const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    // step 1: create a descriptor which contains
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has unit diagonal, specified by parameter HIPSPARSE_DIAG_TYPE_UNIT
    //   (L may not have all diagonal elements.)
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_UNIT);

    // step 2: create a empty info structure
    hipsparseCreateCsrsv2Info(&info);

    // step 3: query how much memory used in csrsv2, and allocate the buffer
    hipsparseCcsrsv2_bufferSize(handle, trans, m, nnz, descr,
                               d_csrVal, d_csrRowPtr, d_csrColInd, info, &pBufferSize);
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);

    // step 4: perform analysis
    hipsparseCcsrsv2_analysis(handle, trans, m, nnz, descr,
                             d_csrVal, d_csrRowPtr, d_csrColInd,
                             info, policy, pBuffer);
    // L has unit diagonal, so no structural zero is reported.
    hipsparseStatus_t status = hipsparseXcsrsv2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // step 5: solve L*y = x
    hipsparseCcsrsv2_solve(handle, trans, m, nnz, &alpha, descr,
                          d_csrVal, d_csrRowPtr, d_csrColInd, info,
                          d_x, d_y, policy, pBuffer);
    // L has unit diagonal, so no numerical zero is reported.
    status = hipsparseXcsrsv2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    CHECK_CUDA(hipMemcpy(y, d_y, m * sizeof(hipComplex), hipMemcpyDeviceToHost));

    // step 6: free resources
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    hipFree(pBuffer);
    hipsparseDestroyCsrsv2Info(info);
}
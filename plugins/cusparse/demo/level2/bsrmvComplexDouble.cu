#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

// Interfacce che usano funcion CUSPARSE
void mat2csr(hipsparseHandle_t, hipsparseMatDescr_t, hipDoubleComplex *, int, int, int, hipDoubleComplex *, int *, int *);
void csrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, hipDoubleComplex, hipDoubleComplex *, int *, int *, hipDoubleComplex *, hipDoubleComplex, hipDoubleComplex *);
void bsrnnz(hipsparseHandle_t, hipsparseMatDescr_t, int *, int *, int, int, int, int, int, int, int *, int &);
void csr2bsr(hipsparseHandle_t, hipsparseMatDescr_t, hipDoubleComplex *, int *, int *, int, int, int, int, int , int , int, int *,  hipDoubleComplex *, int *);
void bsrmv(hipsparseHandle_t, hipsparseMatDescr_t, int, int, int, int, int, int, hipDoubleComplex, hipDoubleComplex *, int *, int *, int, hipDoubleComplex *, hipDoubleComplex, hipDoubleComplex *);

int main(int argn, char *argv[])
{
    // Variabili generiche
    int m, n, nnz, blockDim, mb=0, nb=0, nnzb=0;

    // Variabili su host
    hipDoubleComplex *matrix_host;
    hipDoubleComplex *csr_values_result;
    int *csr_columns_result, *csr_offsets_result;
    hipDoubleComplex *bsr_values_result;
    int *bsr_columns_result, *bsr_offsets_result;
    hipDoubleComplex *x_host, *y_result_bsr;
    hipDoubleComplex alpha, beta;
    hipDoubleComplex *matrix_host_sequential;
    hipDoubleComplex *y_result_sequential;

    srand(time(0));

    m = 4;
    n = 5;
    nnz = 9;
    blockDim = 2;
    alpha = make_hipDoubleComplex(3, 0);
    beta = make_hipDoubleComplex(2, 0);

    // Allocazione memoria sull'host
    matrix_host = (hipDoubleComplex *)malloc((m*n+1)*sizeof(hipDoubleComplex));
    csr_values_result=(hipDoubleComplex *)malloc(nnz * sizeof(hipDoubleComplex));
    csr_offsets_result=(int *)malloc((m + 1) * sizeof(int));
    csr_columns_result=(int *)malloc(nnz * sizeof(int));
    x_host = (hipDoubleComplex *)malloc(n*sizeof(hipDoubleComplex));
    y_result_bsr = (hipDoubleComplex *)malloc(m*sizeof(hipDoubleComplex));
    matrix_host_sequential = (hipDoubleComplex *)malloc((m*n+1)*sizeof(hipDoubleComplex));
    y_result_sequential = (hipDoubleComplex *)malloc(m*sizeof(hipDoubleComplex));

    // Inizializzazione variabili sull'host
    initializeMatrixRandomSparseZ(matrix_host, m, n, nnz);
    initializeArrayRandomZ(x_host, n);
    if (beta.x > 0)
        initializeArrayRandomZ(y_result_bsr, m);
    else
        initializeArrayToZeroZ(y_result_bsr, m);
    copyArrayZ(y_result_bsr, y_result_sequential, m);

    // Swap formato matrice per calcolo sequenziale su host
    swapMatrixZ(matrix_host, m, n, matrix_host_sequential);

    printf("Matrice sparsa:\n");
    stampaMatrixZ(matrix_host, m, n);
    printf("\n");

    printf("Array x:\n");
    stampaArrayZ(x_host, n);
    printf("\n");

    printf("Matrice in formato denso per calcolo sequenziale:\n");
    stampaMatrixZ1D(matrix_host_sequential, m, n);
    printf("\n");

    printf("Array y:\n");
    stampaArrayZ(y_result_bsr, m);
    printf("\n");

    // Dichiarazione dell'handle per CUSPARSE
    hipsparseHandle_t handle;

    // Creazione dell'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Creazione della struttura della matrice con relative proprietà e forma
    hipsparseMatDescr_t descr = 0;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Conversione matrice in formato CSR
    mat2csr(handle, descr, matrix_host, m, n, nnz, csr_values_result, csr_offsets_result, csr_columns_result);

    //Stampa matrice convertita in formato CSR
    printf("Matrice sparsa convertita in formato CSR\n");
    printf("\t csrValA:\t");
    stampaArrayZ(csr_values_result, nnz);
    printf("\t csrRowPtrA:\t");
    stampaArray(csr_offsets_result, (m + 1));
    printf("\t csrColIndA:\t");
    stampaArray(csr_columns_result, nnz);
    printf("\n");

    // Calcolo mb e nb a partire da blockDim
    mb = (m + blockDim - 1) / blockDim;
    nb = (n + blockDim - 1) / blockDim;

    // Allocazione variabili su host
    bsr_offsets_result=(int *)malloc((mb + 1) * sizeof(int));

    // Calcolo bsrRowPtrC e blocchi diversi da zero
    bsrnnz(handle, descr, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, bsr_offsets_result, nnzb);

    // Allocazione variabili su host sulla base del numero di blocchi diversi da zero
    bsr_columns_result=(int *)malloc(nnzb * sizeof(int));
    bsr_values_result=(hipDoubleComplex *)malloc((blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex));

    // Conversione da formato CSR a BSR
    csr2bsr(handle, descr, csr_values_result, csr_offsets_result, csr_columns_result, m, n, nnz, blockDim, mb, nb, nnzb, bsr_offsets_result, bsr_values_result, bsr_columns_result);

    // Stampa matrice convertita in formato BSR
    printf("Matrice sparsa convertita in formato BSR\n");
    printf("\t bsrValC:\t");
    stampaArrayZ(bsr_values_result, (blockDim * blockDim) * nnzb);
    printf("\t bsrRowPtrC:\t");
    stampaArray(bsr_offsets_result, (mb + 1));
    printf("\t bsrColIndC:\t");
    stampaArray(bsr_columns_result, nnzb);
    printf("\n");

    // Operazione bsrmv corrispondente al seguente prodotto y = alpha * A * x + beta * y
    bsrmv(handle, descr, m, n, nnz, mb, nb, nnzb, alpha, bsr_values_result, bsr_offsets_result, bsr_columns_result, blockDim, x_host, beta, y_result_bsr);

    //Stampa array risultato dall'operazione bsrmv tra matrice in formato BSR, vettore x_host, alpha e beta
    printf("Vettore risultato dall'operazione bsrmv\n");
    stampaArrayZ(y_result_bsr, m);
    printf("\n");

    //Libera la memoria sull'host
    free(csr_values_result);
    free(csr_offsets_result);
    free(csr_columns_result);
    free(bsr_values_result);
    free(bsr_offsets_result);
    free(bsr_columns_result);
    free(matrix_host);
    free(matrix_host_sequential);

    // Termina l'handle per CUSPARSE
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

}

// Function che consente di ottenere la matrice in formato CSR
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - matrix - matrice sparsa
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//     output:
//         - csrValA - array contenente i valori
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga
//         - csrColIndA - array contenente gli indici di colonne
void mat2csr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipDoubleComplex * matrix, int m, int n, int nnz, hipDoubleComplex * csrValA, int * csrRowPtrA, int * csrColIndA)
{
    int nnz_total = 0;

    // Variabili su device
    hipDoubleComplex *matrix_device;
    hipDoubleComplex *csr_values_device;
    int *csr_columns_device, *csr_offsets_device;
    int *nnz_per_row;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &matrix_device, m * n * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_values_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csr_offsets_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csr_columns_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &nnz_per_row, m * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(matrix_device, matrix, m * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // Calcolo valori diversi da zero
    CHECK_CUSPARSE(hipsparseZnnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, descr, matrix_device, m, nnz_per_row, &nnz_total));

    // Controllo su valori diversi da zero richiesti in input rispetto a quelli calcolati tramite hipsparseSnnz()
    if (nnz != nnz_total) {
        printf("I valori diversi da zero richiesti in input sono diversi rispetto a quelli rilevati: richiesti %d valori ma sono stati rilevati %d valori diversi da zero!\n\n", nnz, nnz_total);
        exit(EXIT_FAILURE);
    }

    // Conversione matrice in formato CSR
    CHECK_CUSPARSE(hipsparseZdense2csr(handle, m, n, descr, matrix_device, m, nnz_per_row, csr_values_device, csr_offsets_device, csr_columns_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(csrValA, csr_values_device, nnz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrRowPtrA, csr_offsets_device, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(csrColIndA, csr_columns_device, nnz * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(csr_values_device));
    CHECK_CUDA(hipFree(csr_offsets_device));
    CHECK_CUDA(hipFree(csr_columns_device));
    CHECK_CUDA(hipFree(matrix_device));
    CHECK_CUDA(hipFree(nnz_per_row));
}

// Function che consente di ottenere il numero dei blocchi con elementi diversi da zero e l'array bsrRowPtrC inizializzato con i valori di indice per i blocchi di riga
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga, della matrice in formato CSR
//         - csrColIndA - array contenente gli indici di colonne, della matrice in formato CSR
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//     output:
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//         - nnzb - blocchi con valori diversi da zero
void bsrnnz(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int * bsrRowPtrC, int &nnzb)
{
    // Variabili su host
    int base;
    int *nnzTotalBsr = &nnzb;

    // Variabili su device
    int *bsrRowPtrC_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia variabili da host a device
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Calcolo del numero dei blocchi diversi da zero per il formato BSR
    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrRowPtrC_device, nnzTotalBsr));

    // Controllo sul valore dei blocchi calcolato, se null lo calcolo sulla base degli indici dei blocchi e il numero dei blocchi per riga
    if (NULL != nnzTotalBsr)
    {
        nnzb = *nnzTotalBsr;
    }
    else
    {
        CHECK_CUDA(hipMemcpy(&nnzb, bsrRowPtrC_device + mb, sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(&base, bsrRowPtrC_device, sizeof(int), hipMemcpyDeviceToHost));
        nnzb -= base;
    }

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrRowPtrC, bsrRowPtrC_device, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

// Function che consente di ottenere il numero dei blocchi con elementi diversi da zero e l'array bsrRowPtrC inizializzato con i valori di indice per i blocchi di riga
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - csrValA - array contenente i valori della matrice in formato CSR
//         - csrRowPtrA - array contenente gli indici per i quali bisogna considerare il nuovo indice di riga, della matrice in formato CSR
//         - csrColIndA - array contenente gli indici di colonne, della matrice in formato CSR
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//         - nnzb - blocchi con valori diversi da zero
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//     output:
//         - bsrValC - array contenente i valori della matrice in formato BSR
//         - bsrColIndC - array contenente i valori di colonna dei blocchi della matrice in formato BSR
void csr2bsr(hipsparseHandle_t handle, hipsparseMatDescr_t descr, hipDoubleComplex * csrValA, int * csrRowPtrA, int * csrColIndA, int m, int n, int nnz, int blockDim, int mb, int nb, int nnzb, int * bsrRowPtrC, hipDoubleComplex * bsrValC, int * bsrColIndC)
{
    // Variabili su device
    hipDoubleComplex *bsrValC_device;
    int *bsrRowPtrC_device, *bsrColIndC_device;
    hipDoubleComplex *csrValA_device;
    int *csrRowPtrA_device, *csrColIndA_device;

    // Allocazione variabili su device sulla base del numero di blocchi diversi da zero
    CHECK_CUDA(hipMalloc((void**) &csrValA_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &csrRowPtrA_device, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &csrColIndA_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrColIndC_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**)&bsrRowPtrC_device, (mb + 1) * sizeof(int)));

    // Copia da host a device
    CHECK_CUDA(hipMemcpy(csrValA_device, csrValA, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrRowPtrA_device, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(csrColIndA_device, csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Conversione da CSR a BSR
    CHECK_CUSPARSE(hipsparseZcsr2bsr(handle, HIPSPARSE_DIRECTION_COLUMN, m, n, descr, csrValA_device, csrRowPtrA_device, csrColIndA_device, blockDim, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(bsrValC, bsrValC_device, ((blockDim * blockDim) * nnzb) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(bsrColIndC, bsrColIndC_device, nnzb * sizeof(int), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrValC_device));
    CHECK_CUDA(hipFree(bsrColIndC_device));
    //CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(csrValA_device));
    CHECK_CUDA(hipFree(csrRowPtrA_device));
    CHECK_CUDA(hipFree(csrColIndA_device));

}

// Function che calcola il prodotto matrice per vettore tramite bsrmv (y = alpha * A * x + beta * y)
//     input:
//         - handle - handle di CUSPARSE
//         - descr - proprietà e forma della matrice
//         - m - numero di righe
//         - n - numero di colonne
//         - nnz - numero di valori diversi da zero
//         - mb - numero di righe dei blocchi calcolati come segue: mb = (m + blockDim - 1) / blockDim
//         - nb - numero di colonne dei blocchi calcolati come segue: nb = (n + blockDim - 1) / blockDim;
//         - nnzb - blocchi con valori diversi da zero
//         - alpha - scalare che viene utilizzato nel primo prodotto bsrmv (alpha * A * x)
//         - bsrValC - array contenente i valori della matrice in formato BSR
//         - bsrRowPtrC - array contenente i valori di indice per i blocchi di riga della matrice in formato BSR
//         - bsrColIndC - array contenente i valori di colonna dei blocchi della matrice in formato BSR
//         - blockDim - dimensione che deve avere il blocco nel formato BSR
//         - x - array che viene utilizzato nel primo prodotto bsrmv (alpha * A * x)
//         - beta - scalare che viene utilizzato nel secondo prodotto bsrmv (beta * y)
//         - y - array che viene utilizzato nel secondo prodotto bsrmv (beta * y)
//     output:
//         - y - array risultato dell'operazione bsrmv (y = alpha * A * x + beta * y)
void bsrmv(hipsparseHandle_t handle, hipsparseMatDescr_t descr, int m, int n, int nnz, int mb, int nb, int nnzb, hipDoubleComplex alpha, hipDoubleComplex * bsrValC, int * bsrRowPtrC, int * bsrColIndC, int blockDim, hipDoubleComplex * x, hipDoubleComplex beta, hipDoubleComplex * y)
{
    // Variabili su device
    hipDoubleComplex *x_device, *y_device;
    hipDoubleComplex *bsrValC_device;
    int *bsrRowPtrC_device, *bsrColIndC_device;

    // Allocazione variabili su device
    CHECK_CUDA(hipMalloc((void**) &bsrValC_device, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &bsrRowPtrC_device, (mb + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &bsrColIndC_device, nnzb * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&x_device, (nb*blockDim) * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**)&y_device, (mb*blockDim) * sizeof(hipDoubleComplex)));

    // Copia valori da host a device
    CHECK_CUDA(hipMemcpy(x_device, x, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrValC_device, bsrValC, (blockDim * blockDim) * nnzb * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrRowPtrC_device, bsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(bsrColIndC_device, bsrColIndC, nnzb * sizeof(int), hipMemcpyHostToDevice));

    // Operazione y = alpha * A * x + beta * y
    // la matrice A è rappresentata in formato BSR dagli array bsrValC_device, bsrRowPtrC_device e bsrColIndC_device
    CHECK_CUSPARSE(hipsparseZbsrmv(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, nnzb, &alpha, descr, bsrValC_device, bsrRowPtrC_device, bsrColIndC_device, blockDim, x_device, &beta, y_device));

    // Copia valori da device a host
    CHECK_CUDA(hipMemcpy(y, y_device, (mb*blockDim) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(bsrValC_device));
    CHECK_CUDA(hipFree(bsrColIndC_device));
    CHECK_CUDA(hipFree(bsrRowPtrC_device));
    CHECK_CUDA(hipFree(x_device));
    CHECK_CUDA(hipFree(y_device));

}
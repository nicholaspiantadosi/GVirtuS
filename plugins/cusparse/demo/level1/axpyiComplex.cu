#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void generateSparseVectorFormat(hipComplex *, int, hipComplex *, int *);
void axpyi(hipsparseHandle_t, int, int, hipComplex *, hipComplex *, int *, hipComplex *);

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    hipComplex alpha = make_hipComplex(2, 1);
    hipComplex *xVal = (hipComplex *)malloc(nnz*sizeof(hipComplex));
    int *xInd = (int *)malloc(nnz*sizeof(int));
    hipComplex *y_result = (hipComplex *)malloc(n*sizeof(hipComplex));

    hipComplex *vector = (hipComplex *)malloc(n*sizeof(hipComplex));

    initializeArrayToZerocuComplex(vector, n, nnz);
    initializeArrayToZerocuComplex(y_result, n);

    printf("Array x:\n");
    stampaArrayC(vector, n);

    generateSparseVectorFormat(vector, n, xVal, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayC(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    axpyi(handle, n, nnz, &alpha, xVal, xInd, y_result);

    printf("Vettore risultato:\n");
    stampaArrayC(y_result, n);

    free(vector);
    free(y_result);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void axpyi(hipsparseHandle_t handle, int n, int nnz, hipComplex * alpha, hipComplex * xVal, int * xInd, hipComplex * y) {

    // Variabili su device
    hipComplex * xVal_device;
    int * xInd_device;
    hipComplex * y_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(hipComplex)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(hipComplex), hipMemcpyHostToDevice));

    // Calcolo prodotto tramite function axpyi
    CHECK_CUSPARSE(hipsparseCaxpyi(handle, nnz, alpha, xVal_device, xInd_device, y_device, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(y, y_device, n * sizeof(hipComplex), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));

}

void generateSparseVectorFormat(hipComplex * vector_host, int n, hipComplex * sparse_values, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i].x > 0) {
            sparse_values[j] = vector_host[i];
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
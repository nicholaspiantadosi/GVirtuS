#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void generateSparseVectorFormat(float *, int, float *, int *);
void sctr(hipsparseHandle_t, int, int, float *, int *, float *);

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    float *xVal = (float *)malloc(nnz*sizeof(float));
    int *xInd = (int *)malloc(nnz*sizeof(int));
    float *y_result = (float *)malloc(n*sizeof(float));

    float *vector = (float *)malloc(n*sizeof(float));

    initializeArrayRandomSparse(vector, n, nnz);
    initializeArrayToZero(y_result, n);

    printf("Array x:\n");
    stampaArrayF(vector, n);

    generateSparseVectorFormat(vector, n, xVal, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayF(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    sctr(handle, n, nnz, xVal, xInd, y_result);

    printf("Vettore risultato:\n");
    stampaArrayF(y_result, n);

    free(vector);
    free(y_result);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void sctr(hipsparseHandle_t handle, int n, int nnz, float * xVal, int * xInd, float * y) {

    // Variabili su device
    float * xVal_device;
    int * xInd_device;
    float * y_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(float)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseSsctr(handle, nnz, xVal_device, xInd_device, y_device, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(y, y_device, n * sizeof(float), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));

}

void generateSparseVectorFormat(float * vector_host, int n, float * sparse_values, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i] > 0) {
            sparse_values[j] = vector_host[i];
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
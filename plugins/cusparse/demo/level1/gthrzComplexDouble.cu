#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void gthrz(hipsparseHandle_t, int, int, hipDoubleComplex *, hipDoubleComplex *, int *);
void generateSparseIndex(hipDoubleComplex *, int, int *);

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    hipDoubleComplex *y = (hipDoubleComplex *)malloc(n*sizeof(hipDoubleComplex));
    hipDoubleComplex *xVal = (hipDoubleComplex *)malloc(nnz*sizeof(hipDoubleComplex));
    int *xInd = (int *)malloc(nnz*sizeof(int));

    initializeArrayRandomSparsecuDoubleComplex(y, n, nnz);
    initializeArrayToZerocuDoubleComplex(xVal, nnz);

    printf("Array y:\n");
    stampaArrayZ(y, n);

    generateSparseIndex(y, n, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayZ(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    gthrz(handle, n, nnz, y, xVal, xInd);

    printf("Vettore risultato:\n");
    stampaArrayZ(xVal, nnz);

    printf("Array y:\n");
    stampaArrayZ(y, n);

    free(y);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void gthrz(hipsparseHandle_t handle, int n, int nnz, hipDoubleComplex * y, hipDoubleComplex * xVal, int * xInd) {

    // Variabili su device
    hipDoubleComplex * y_device;
    hipDoubleComplex * xVal_device;
    int * xInd_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Scrivi i valori sparsi in vettore denso
    CHECK_CUSPARSE(hipsparseZgthrz(handle, nnz, y_device, xVal_device, xInd_device, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(xVal, xVal_device, nnz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(y, y_device, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));
}

void generateSparseIndex(hipDoubleComplex * vector_host, int n, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i].x > 0) {
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
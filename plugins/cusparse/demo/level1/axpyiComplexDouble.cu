#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void generateSparseVectorFormat(hipDoubleComplex *, int, hipDoubleComplex *, int *);
void axpyi(hipsparseHandle_t, int, int, hipDoubleComplex *, hipDoubleComplex *, int *, hipDoubleComplex *);

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    hipDoubleComplex alpha = make_hipDoubleComplex(2, 1);
    hipDoubleComplex *xVal = (hipDoubleComplex *)malloc(nnz*sizeof(hipDoubleComplex));
    int *xInd = (int *)malloc(nnz*sizeof(int));
    hipDoubleComplex *y_result = (hipDoubleComplex *)malloc(n*sizeof(hipDoubleComplex));

    hipDoubleComplex *vector = (hipDoubleComplex *)malloc(n*sizeof(hipDoubleComplex));

    initializeArrayToZerocuDoubleComplex(vector, n, nnz);
    initializeArrayToZerocuDoubleComplex(y_result, n);

    printf("Array x:\n");
    stampaArrayZ(vector, n);

    generateSparseVectorFormat(vector, n, xVal, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayZ(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    axpyi(handle, n, nnz, &alpha, xVal, xInd, y_result);

    printf("Vettore risultato:\n");
    stampaArrayZ(y_result, n);

    free(vector);
    free(y_result);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void axpyi(hipsparseHandle_t handle, int n, int nnz, hipDoubleComplex * alpha, hipDoubleComplex * xVal, int * xInd, hipDoubleComplex * y) {

    // Variabili su device
    hipDoubleComplex * xVal_device;
    int * xInd_device;
    hipDoubleComplex * y_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(hipDoubleComplex)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    // Calcolo prodotto tramite function axpyi
    CHECK_CUSPARSE(hipsparseZaxpyi(handle, nnz, alpha, xVal_device, xInd_device, y_device, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(y, y_device, n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));

}

void generateSparseVectorFormat(hipDoubleComplex * vector_host, int n, hipDoubleComplex * sparse_values, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i].x > 0) {
            sparse_values[j] = vector_host[i];
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
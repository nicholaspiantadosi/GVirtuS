#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void gthr(hipsparseHandle_t, int, int, hipComplex *, hipComplex *, int *);
void generateSparseIndex(hipComplex *, int, int *);

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    hipComplex *y = (hipComplex *)malloc(n*sizeof(hipComplex));
    hipComplex *xVal = (hipComplex *)malloc(nnz*sizeof(hipComplex));
    int *xInd = (int *)malloc(nnz*sizeof(int));

    initializeArrayRandomSparsecuComplex(y, n, nnz);
    initializeArrayToZerocuComplex(xVal, nnz);

    printf("Array y:\n");
    stampaArrayC(y, n);

    generateSparseIndex(y, n, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayC(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    gthr(handle, n, nnz, y, xVal, xInd);

    printf("Vettore risultato:\n");
    stampaArrayC(xVal, nnz);

    free(y);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void gthr(hipsparseHandle_t handle, int n, int nnz, hipComplex * y, hipComplex * xVal, int * xInd) {

    // Variabili su device
    hipComplex * y_device;
    hipComplex * xVal_device;
    int * xInd_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(hipComplex)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Scrivi i valori sparsi in vettore denso
    CHECK_CUSPARSE(hipsparseCgthr(handle, nnz, y_device, xVal_device, xInd_device, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(xVal, xVal_device, nnz * sizeof(hipComplex), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));
}

void generateSparseIndex(hipComplex * vector_host, int n, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i].x > 0) {
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
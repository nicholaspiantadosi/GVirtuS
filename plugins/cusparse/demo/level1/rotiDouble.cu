#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

void generateSparseVectorFormat(double *, int, double *, int *);
void roti(hipsparseHandle_t , int , int , double * , int * , double * , double * , double * );

int main(void)
{
    srand(time(0));

    int n = 10;
    int nnz = 4;
    double c = 2;
    double s = 1;
    double *xVal = (double *)malloc(nnz*sizeof(double));
    int *xInd = (int *)malloc(nnz*sizeof(int));
    double *y_result = (double *)malloc(n*sizeof(double));

    double *vector = (double *)malloc(n*sizeof(double));

    initializeArrayRandomSparseDouble(vector, n, nnz);
    initializeArrayToZeroDouble(y_result, n);

    printf("Array x:\n");
    stampaArrayD(vector, n);

    generateSparseVectorFormat(vector, n, xVal, xInd);

    printf("Array x in formato sparso:\n");
    printf("\txVal: ");
    stampaArrayD(xVal, nnz);
    printf("\n\txInd: ");
    stampaArray(xInd, nnz);
    printf("\n");

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    roti(handle, n, nnz, xVal, xInd, y_result, &c, &s);

    printf("Vettore risultato:\n");
    stampaArrayD(y_result, n);

    printf("Vettore x:\n");
    stampaArrayD(xVal, nnz);

    free(vector);
    free(y_result);
    free(xVal);
    free(xInd);
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}

void roti(hipsparseHandle_t handle, int n, int nnz, double * xVal, int * xInd, double * y, double * c, double * s) {

    // Variabili su device
    double * xVal_device;
    int * xInd_device;
    double * y_device;

    // Allocazione memoria su device
    CHECK_CUDA(hipMalloc((void**) &xVal_device, nnz * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &xInd_device, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &y_device, n * sizeof(double)));

    // Inizializzazione variabili su device
    CHECK_CUDA(hipMemcpy(xVal_device, xVal, nnz * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(xInd_device, xInd, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(y_device, y, n * sizeof(double), hipMemcpyHostToDevice));

    // Calcolo prodotto tramite function axpyi
    CHECK_CUSPARSE(hipsparseDroti(handle, nnz, xVal_device, xInd_device, y_device, c, s, HIPSPARSE_INDEX_BASE_ZERO));

    // Copia risultato da device a host
    CHECK_CUDA(hipMemcpy(y, y_device, n * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(xVal, xVal_device, nnz * sizeof(double), hipMemcpyDeviceToHost));

    //Libera la memoria sul device
    CHECK_CUDA(hipFree(xVal_device));
    CHECK_CUDA(hipFree(xInd_device));
    CHECK_CUDA(hipFree(y_device));

}

void generateSparseVectorFormat(double * vector_host, int n, double * sparse_values, int * sparse_col_index) {
    int i = 0;
    int j = 0;
    for (i = 0; i < n; i++) {
        if (vector_host[i] > 0) {
            sparse_values[j] = vector_host[i];
            sparse_col_index[j] = i;
            ++j;
        }
    }
}
#include<stdio.h>
#include<hipsparse.h>

const char * getErrorString(hipsparseStatus_t error)
{
    switch (error)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            return "The operation completed successfully.";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "The cuSPARSE library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSPARSE routine, or an error in the hardware setup.\n" \
				"To correct: call hipsparseCreate() prior to the function call; and check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "Resource allocation failed inside the cuSPARSE library. This is usually caused by a hipMalloc() failure.\n"\
					"To correct: prior to the function call, deallocate previously allocated memory as much as possible.";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "An unsupported value or parameter was passed to the function (a negative vector size, for example).\n"\
				"To correct: ensure that all the parameters being passed have valid values.";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision.\n"\
				"To correct: compile and run the application on a device with appropriate compute capability, which is 1.1 for 32-bit atomic operations and 1.3 for double precision.";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "An access to GPU memory space failed, which is usually caused by a failure to bind a texture.\n"\
				"To correct: prior to the function call, unbind any previously bound textures.";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "An internal cuSPARSE operation failed. This error is usually caused by a hipMemcpyAsync() failure.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed. Also, check that the memory passed as a parameter to the routine is not being deallocated prior to the routine’s completion.";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function.\n"\
					"To correct: check that the fields in hipsparseMatDescr_t descrA were set correctly.";
    }

    return "<unknown>";
}

void CHECK_CUSPARSE(hipsparseStatus_t status)
{
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, getErrorString(status), status);
        exit(EXIT_FAILURE);
    }
}

int main(void)
{
    printf("hipsparseCreateCsrsv2Info\n");
    csrsv2Info_t csrsv2Info;
    CHECK_CUSPARSE(hipsparseCreateCsrsv2Info(&csrsv2Info));
    printf("csrsv2Info: %d\n", csrsv2Info);

    printf("hipsparseDestroyCsrsv2Info\n");
    CHECK_CUSPARSE(hipsparseDestroyCsrsv2Info(csrsv2Info));
    printf("csrsv2Info destroyed\n\n");

    printf("hipsparseCreateCsrsm2Info\n");
    csrsm2Info_t csrsm2Info;
    CHECK_CUSPARSE(hipsparseCreateCsrsm2Info(&csrsm2Info));
    printf("csrsm2Info: %d\n", csrsm2Info);

    printf("hipsparseDestroyCsrsm2Info\n");
    CHECK_CUSPARSE(hipsparseDestroyCsrsm2Info(csrsm2Info));
    printf("csrsm2Info destroyed\n\n");

    printf("hipsparseCreateCsric02Info\n");
    csric02Info_t csric02Info;
    CHECK_CUSPARSE(hipsparseCreateCsric02Info(&csric02Info));
    printf("csric02Info: %d\n", csric02Info);

    printf("hipsparseDestroyCsric02Info\n");
    CHECK_CUSPARSE(hipsparseDestroyCsric02Info(csric02Info));
    printf("csric02Info destroyed\n\n");

    printf("hipsparseCreateCsrilu02Info\n");
    csrilu02Info_t csrilu02Info;
    CHECK_CUSPARSE(hipsparseCreateCsrilu02Info(&csrilu02Info));
    printf("csrilu02Info: %d\n", csrilu02Info);

    printf("hipsparseDestroyCsrilu02Info\n");
    CHECK_CUSPARSE(hipsparseDestroyCsrilu02Info(csrilu02Info));
    printf("csrilu02Info destroyed\n\n");

    printf("hipsparseCreateBsrsv2Info\n");
    bsrsv2Info_t bsrsv2Info;
    CHECK_CUSPARSE(hipsparseCreateBsrsv2Info(&bsrsv2Info));
    printf("bsrsv2Info: %d\n", bsrsv2Info);

    printf("hipsparseDestroyBsrsv2Info\n");
    CHECK_CUSPARSE(hipsparseDestroyBsrsv2Info(bsrsv2Info));
    printf("bsrsv2Info destroyed\n\n");

    printf("hipsparseCreateBsrsm2Info\n");
    bsrsm2Info_t bsrsm2Info;
    CHECK_CUSPARSE(hipsparseCreateBsrsm2Info(&bsrsm2Info));
    printf("bsrsm2Info: %d\n", bsrsm2Info);

    printf("hipsparseDestroyBsrsm2Info\n");
    CHECK_CUSPARSE(hipsparseDestroyBsrsm2Info(bsrsm2Info));
    printf("bsrsm2Info destroyed\n\n");

    printf("hipsparseCreateBsric02Info\n");
    bsric02Info_t bsric02Info;
    CHECK_CUSPARSE(hipsparseCreateBsric02Info(&bsric02Info));
    printf("bsric02Info: %d\n", bsric02Info);

    printf("hipsparseDestroyBsric02Info\n");
    CHECK_CUSPARSE(hipsparseDestroyBsric02Info(bsric02Info));
    printf("bsric02Info destroyed\n\n");

    printf("hipsparseCreateBsrilu02Info\n");
    bsrilu02Info_t bsrilu02Info;
    CHECK_CUSPARSE(hipsparseCreateBsrilu02Info(&bsrilu02Info));
    printf("bsrilu02Info: %d\n", bsrilu02Info);

    printf("hipsparseDestroyBsrilu02Info\n");
    CHECK_CUSPARSE(hipsparseDestroyBsrilu02Info(bsrilu02Info));
    printf("bsrilu02Info destroyed\n\n");

    printf("hipsparseCreateCsrgemm2Info\n");
    csrgemm2Info_t csrgemm2Info;
    CHECK_CUSPARSE(hipsparseCreateCsrgemm2Info(&csrgemm2Info));
    printf("csrgemm2Info: %d\n", csrgemm2Info);

    printf("hipsparseDestroyCsrgemm2Info\n");
    CHECK_CUSPARSE(hipsparseDestroyCsrgemm2Info(csrgemm2Info));
    printf("csrgemm2Info destroyed\n\n");

    printf("hipsparseCreatePruneInfo\n");
    pruneInfo_t pruneInfo;
    CHECK_CUSPARSE(hipsparseCreatePruneInfo(&pruneInfo));
    printf("pruneInfo: %d\n", pruneInfo);

    printf("hipsparseDestroyPruneInfo\n");
    CHECK_CUSPARSE(hipsparseDestroyPruneInfo(pruneInfo));
    printf("pruneInfo destroyed\n\n");

    return 0;
}
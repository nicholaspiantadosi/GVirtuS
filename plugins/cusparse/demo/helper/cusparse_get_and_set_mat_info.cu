
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<hipsparse.h>

const char * getErrorString(hipsparseStatus_t error)
{
    switch (error)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            return "The operation completed successfully.";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "The cuSPARSE library was not initialized. This is usually caused by the lack of a prior call, an error in the CUDA Runtime API called by the cuSPARSE routine, or an error in the hardware setup.\n" \
				"To correct: call cusparseCreate() prior to the function call; and check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "Resource allocation failed inside the cuSPARSE library. This is usually caused by a cudaMalloc() failure.\n"\
					"To correct: prior to the function call, deallocate previously allocated memory as much as possible.";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "An unsupported value or parameter was passed to the function (a negative vector size, for example).\n"\
				"To correct: ensure that all the parameters being passed have valid values.";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "The function requires a feature absent from the device architecture; usually caused by the lack of support for atomic operations or double precision.\n"\
				"To correct: compile and run the application on a device with appropriate compute capability, which is 1.1 for 32-bit atomic operations and 1.3 for double precision.";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "An access to GPU memory space failed, which is usually caused by a failure to bind a texture.\n"\
				"To correct: prior to the function call, unbind any previously bound textures.";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "The GPU program failed to execute. This is often caused by a launch failure of the kernel on the GPU, which can be caused by multiple reasons.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed.";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "An internal cuSPARSE operation failed. This error is usually caused by a cudaMemcpyAsync() failure.\n"\
					"To correct: check that the hardware, an appropriate version of the driver, and the cuSPARSE library are correctly installed. Also, check that the memory passed as a parameter to the routine is not being deallocated prior to the routine’s completion.";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "The matrix type is not supported by this function. This is usually caused by passing an invalid matrix descriptor to the function.\n"\
					"To correct: check that the fields in hipsparseMatDescr_t descrA were set correctly.";
    }

    return "<unknown>";
}

void CHECK_CUSPARSE(hipsparseStatus_t status)
{
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, getErrorString(status), status);
        exit(EXIT_FAILURE);
    }
}

int main(void)
{
    printf("cusparseCreateMatDescr\n");
    hipsparseMatDescr_t descrA;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    printf("descrA: %d\n\n", descrA);

    printf("cusparseGetMatDiagType\n");
    hipsparseDiagType_t diagType = hipsparseGetMatDiagType(descrA);
    printf("diagType: %d\n", diagType);
    printf("hipsparseSetMatDiagType to 1\n");
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_UNIT));
    printf("cusparseGetMatDiagType\n");
    printf("diagType: %d\n", hipsparseGetMatDiagType(descrA));
    printf("hipsparseSetMatDiagType to 0\n");
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    printf("cusparseGetMatDiagType\n");
    printf("diagType: %d\n\n", hipsparseGetMatDiagType(descrA));

    printf("cusparseGetMatFillMode\n");
    hipsparseFillMode_t fillMode = hipsparseGetMatFillMode(descrA);
    printf("fillMode: %d\n", fillMode);
    printf("hipsparseSetMatFillMode to 1\n");
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_UPPER));
    printf("cusparseGetMatFillMode\n");
    printf("fillMode: %d\n", hipsparseGetMatFillMode(descrA));
    printf("hipsparseSetMatFillMode to 0\n");
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER));
    printf("cusparseGetMatFillMode\n");
    printf("fillMode: %d\n\n", hipsparseGetMatFillMode(descrA));

    printf("cusparseGetMatIndexBase\n");
    hipsparseIndexBase_t indexBase = hipsparseGetMatIndexBase(descrA);
    printf("indexBase: %d\n", indexBase);
    printf("hipsparseSetMatIndexBase to 1\n");
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    printf("cusparseGetMatIndexBase\n");
    printf("indexBase: %d\n", hipsparseGetMatIndexBase(descrA));
    printf("hipsparseSetMatIndexBase to 0\n");
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    printf("cusparseGetMatIndexBase\n");
    printf("indexBase: %d\n\n", hipsparseGetMatIndexBase(descrA));

    printf("cusparseGetMatType\n");
    hipsparseMatrixType_t matrixType = hipsparseGetMatType(descrA);
    printf("matrixType: %d\n", matrixType);
    printf("hipsparseSetMatType to 1\n");
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
    printf("cusparseGetMatType\n");
    printf("matrixType: %d\n", hipsparseGetMatType(descrA));
    printf("hipsparseSetMatType to 2\n");
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_HERMITIAN));
    printf("cusparseGetMatType\n");
    printf("matrixType: %d\n", hipsparseGetMatType(descrA));
    printf("hipsparseSetMatType to 3\n");
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
    printf("cusparseGetMatType\n");
    printf("matrixType: %d\n", hipsparseGetMatType(descrA));
    printf("hipsparseSetMatType to 0\n");
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    printf("cusparseGetMatType\n");
    printf("matrixType: %d\n", hipsparseGetMatType(descrA));

    printf("cusparseDestroyMatDescr\n");
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    printf("descrA destroyed\n");

    return 0;
}

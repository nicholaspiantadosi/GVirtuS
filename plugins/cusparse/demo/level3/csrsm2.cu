#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int m = 4;
    const int nrhs = 4;
    const int nnz = 9;
    const int ldb = m; // leading dimension of B and C

    float hCsrValA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    float hB[] = {1, 2, 0, 3,
                  0, 0, 0, 4,
                  0, 0, 5, 6,
                  0, 0, 7, 8};

    float hC[] = {0, 0, 0, 0,
                  0, 0, 0, 0,
                  0, 0, 0, 0,
                  0, 0, 0, 0};
    float hC_result[] = {0, 0, 0, 0,
                  0, 0, 0, 0,
                  0, 0, 0, 0,
                  0, 0, 0, 0};

    float alpha = 1;

    // Device memory management
    float *dCsrValA;
    int *dCsrRowPtrA, *dCsrColIndA;
    float *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  m * nrhs * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dC,  m * nrhs * sizeof(float)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, m * nrhs * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, m * nrhs * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    csrsm2Info_t info = 0;
    hipsparseCreateCsrsm2Info(&info);
    size_t pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    int algo = 0;

    hipsparseScsrsm2_bufferSizeExt(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz,  &alpha, descrA,
                               dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseScsrsm2_analysis(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXcsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // WHERE IS C???
    hipsparseScsrsm2_solve(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXcsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, m * nrhs * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m * nrhs; i++) {
        if((fabs(hC[i] - hC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csrm2 test PASSED\n");
    else
        printf("csrm2 test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
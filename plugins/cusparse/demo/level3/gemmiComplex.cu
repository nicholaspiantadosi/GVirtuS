#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition

    int m = 4;
    int n = 5;
    int k = 5;
    int nnz = 9;
    hipComplex alpha = make_hipComplex(1, 0);
    hipComplex hA[] = {
            make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0),
            make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(0, 0), make_hipComplex(9, 0), make_hipComplex(0, 0) };
    int lda = m;
    hipComplex hCscValB[] = {make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(9, 0),
                            make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(6, 0) };
    int hCscColPtrB[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndB[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    hipComplex beta = make_hipComplex(1, 0);
    hipComplex hC[] = {
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0) };
    int ldc = m;
    hipComplex hC_result[] = {
            make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(27, 0), make_hipComplex(3, 0), make_hipComplex(4, 0),
            make_hipComplex(0, 0), make_hipComplex(16, 0), make_hipComplex(12, 0), make_hipComplex(54, 0), make_hipComplex(0, 0),
            make_hipComplex(75, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(35, 0),
            make_hipComplex(0, 0), make_hipComplex(36, 0), make_hipComplex(0, 0), make_hipComplex(82, 0), make_hipComplex(0, 0) };

    // Device memory management
    hipComplex *dA, *dC;
    hipComplex *dCscValB;
    int *dCscColPtrB, *dCscRowIndB;

    CHECK_CUDA( hipMalloc((void**) &dA,  lda * k * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCscValB, nnz * sizeof(hipComplex)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrB, (k + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndB, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * n * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dA, hA, lda * k * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscValB, hCscValB, nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrB, hCscColPtrB, (k + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndB, hCscRowIndB, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * n * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    CHECK_CUSPARSE(hipsparseCgemmi(handle, m, n, k, nnz, &alpha, dA, lda, dCscValB, dCscColPtrB, dCscRowIndB, &beta, dC, ldc));

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * n; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gemmi test PASSED\n");
    else
        printf("gemmi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dCscValB) );
    CHECK_CUDA( hipFree(dCscColPtrB) );
    CHECK_CUDA( hipFree(dCscRowIndB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;

}
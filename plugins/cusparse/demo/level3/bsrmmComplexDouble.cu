#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    hipDoubleComplex hBsrValA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                            make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0),
                            make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0),
                            make_hipDoubleComplex(6, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(4, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0)};

    hipDoubleComplex hC[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(4, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(6, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex hC_result[] = {make_hipDoubleComplex(10, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(27, 0), make_hipDoubleComplex(28, 0),
                             make_hipDoubleComplex(12, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(31, 0), make_hipDoubleComplex(16, 0),
                             make_hipDoubleComplex(32, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(72, 0), make_hipDoubleComplex(24, 0),
                             make_hipDoubleComplex(44, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(98, 0), make_hipDoubleComplex(32, 0)};

    hipDoubleComplex alpha = make_hipDoubleComplex(1, 0);
    hipDoubleComplex beta = make_hipDoubleComplex(1, 0);

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    hipDoubleComplex *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    hipDoubleComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    // perform C:=alpha*A*B + beta*C
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseZbsrmm(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, kb, nnzb, &alpha, descrA, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, dB, ldb, &beta, dC, ldc);

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if (hC[i].x != hC_result[i].x) { // direct cuDoubleComplexing point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
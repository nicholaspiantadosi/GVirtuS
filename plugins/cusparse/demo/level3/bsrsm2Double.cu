#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    double hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 4};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    double hB[] = {1, 2, 0, 3, 0, 0, 0, 4, 0, 0, 5, 6, 0, 0, 7, 8};

    double hC[] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    double hC_result[] = {1,0.500000,-0.833333,-0.250000,0,0,0,1,0,0,0.833333,1.500000,0,0,1.166667,2};

    double alpha = 1;

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    double *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    double *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(double)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    bsrsm2Info_t info = 0;
    hipsparseCreateBsrsm2Info(&info);
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;

    hipsparseDbsrsm2_bufferSize(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                               dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseDbsrsm2_analysis(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXbsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDbsrsm2_solve(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, &alpha, descrA,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, dB, ldb, dC, ldc, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXbsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if((fabs(hC[i] - hC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    hipComplex hBsrValA[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                            make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                            make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0),
                            make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(4, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipComplex hB[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(8, 0)};

    hipComplex hC[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                      make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};
    hipComplex hC_result[] = {make_hipComplex(1, 0),make_hipComplex(0.500000, 0),make_hipComplex(-0.833333, 0),make_hipComplex(-0.250000, 0),
                             make_hipComplex(0, 0),make_hipComplex(0, 0),make_hipComplex(0, 0),make_hipComplex(1, 0),
                             make_hipComplex(0, 0),make_hipComplex(0, 0),make_hipComplex(0.833333, 0),make_hipComplex(1.500000, 0),
                             make_hipComplex(0, 0),make_hipComplex(0, 0),make_hipComplex(1.166667, 0),make_hipComplex(2, 0)};

    hipComplex alpha = make_hipComplex(1, 0);

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    hipComplex *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    hipComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    bsrsm2Info_t info = 0;
    hipsparseCreateBsrsm2Info(&info);
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;

    hipsparseCbsrsm2_bufferSize(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                               dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseCbsrsm2_analysis(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXbsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseCbsrsm2_solve(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, &alpha, descrA,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, dB, ldb, dC, ldc, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXbsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition

    int m = 4;
    int n = 5;
    int k = 5;
    int nnz = 9;
    float alpha = 1;
    float hA[] = {
            1, 0, 2, 3, 0,
            0, 4, 0, 0, 0,
            5, 0, 6, 0, 7,
            0, 8, 0, 9, 0 };
    int lda = m;
    float hCscValB[] = {1, 5, 4, 2, 3, 9, 7, 8, 6};
    int hCscColPtrB[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndB[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    float beta = 1;
    float hC[] = {
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0 };
    int ldc = m;
    float hC_result[] = {
            1, 0, 27, 3, 4,
            0, 16, 12, 54, 0,
            75, 0, 0, 0, 35,
            0, 36, 0, 82, 0 };

    // Device memory management
    float *dA, *dC;
    float *dCscValB;
    int *dCscColPtrB, *dCscRowIndB;

    CHECK_CUDA( hipMalloc((void**) &dA,  lda * k * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCscValB, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrB, (k + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndB, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * n * sizeof(float)));

    CHECK_CUDA( hipMemcpy(dA, hA, lda * k * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscValB, hCscValB, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrB, hCscColPtrB, (k + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndB, hCscRowIndB, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * n * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    CHECK_CUSPARSE(hipsparseSgemmi(handle, m, n, k, nnz, &alpha, dA, lda, dCscValB, dCscColPtrB, dCscRowIndB, &beta, dC, ldc));

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * n; i++) {
        if((fabs(hC[i] - hC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gemmi test PASSED\n");
    else
        printf("gemmi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dCscValB) );
    CHECK_CUDA( hipFree(dCscColPtrB) );
    CHECK_CUDA( hipFree(dCscRowIndB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;

}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition

    int m = 4;
    int n = 5;
    int k = 5;
    int nnz = 9;
    hipDoubleComplex alpha = make_hipDoubleComplex(1, 0);
    hipDoubleComplex hA[] = {
            make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(0, 0) };
    int lda = m;
    hipDoubleComplex hCscValB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(9, 0),
                            make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(6, 0) };
    int hCscColPtrB[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndB[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    hipDoubleComplex beta = make_hipDoubleComplex(1, 0);
    hipDoubleComplex hC[] = {
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0) };
    int ldc = m;
    hipDoubleComplex hC_result[] = {
            make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(27, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(16, 0), make_hipDoubleComplex(12, 0), make_hipDoubleComplex(54, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(75, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(35, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(36, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(82, 0), make_hipDoubleComplex(0, 0) };

    // Device memory management
    hipDoubleComplex *dA, *dC;
    hipDoubleComplex *dCscValB;
    int *dCscColPtrB, *dCscRowIndB;

    CHECK_CUDA( hipMalloc((void**) &dA,  lda * k * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCscValB, nnz * sizeof(hipDoubleComplex)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrB, (k + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndB, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * n * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dA, hA, lda * k * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscValB, hCscValB, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrB, hCscColPtrB, (k + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndB, hCscRowIndB, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    CHECK_CUSPARSE(hipsparseZgemmi(handle, m, n, k, nnz, &alpha, dA, lda, dCscValB, dCscColPtrB, dCscRowIndB, &beta, dC, ldc));

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * n; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gemmi test PASSED\n");
    else
        printf("gemmi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dCscValB) );
    CHECK_CUDA( hipFree(dCscColPtrB) );
    CHECK_CUDA( hipFree(dCscRowIndB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;

}
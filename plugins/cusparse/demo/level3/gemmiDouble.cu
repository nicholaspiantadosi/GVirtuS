#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{
    // Host problem definition

    int m = 4;
    int n = 5;
    int k = 5;
    int nnz = 9;
    double alpha = 1;
    double hA[] = {
            1, 0, 2, 3, 0,
            0, 4, 0, 0, 0,
            5, 0, 6, 0, 7,
            0, 8, 0, 9, 0 };
    int lda = m;
    double hCscValB[] = {1, 5, 4, 2, 3, 9, 7, 8, 6};
    int hCscColPtrB[] = {0, 2, 4, 6, 7, 9};
    int hCscRowIndB[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    double beta = 1;
    double hC[] = {
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0,
            0, 0, 0, 0, 0 };
    int ldc = m;
    double hC_result[] = {
            1, 0, 27, 3, 4,
            0, 16, 12, 54, 0,
            75, 0, 0, 0, 35,
            0, 36, 0, 82, 0 };

    // Device memory management
    double *dA, *dC;
    double *dCscValB;
    int *dCscColPtrB, *dCscRowIndB;

    CHECK_CUDA( hipMalloc((void**) &dA,  lda * k * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCscValB, nnz * sizeof(double)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrB, (k + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndB, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * n * sizeof(double)));

    CHECK_CUDA( hipMemcpy(dA, hA, lda * k * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscValB, hCscValB, nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrB, hCscColPtrB, (k + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndB, hCscRowIndB, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * n * sizeof(double), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    CHECK_CUSPARSE(hipsparseDgemmi(handle, m, n, k, nnz, &alpha, dA, lda, dCscValB, dCscColPtrB, dCscRowIndB, &beta, dC, ldc));

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * n; i++) {
        if((fabs(hC[i] - hC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gemmi test PASSED\n");
    else
        printf("gemmi test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dA) );
    CHECK_CUDA( hipFree(dCscValB) );
    CHECK_CUDA( hipFree(dCscColPtrB) );
    CHECK_CUDA( hipFree(dCscRowIndB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;

}
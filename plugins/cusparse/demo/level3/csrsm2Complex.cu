#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int m = 4;
    const int nrhs = 4;
    const int nnz = 9;
    const int ldb = m; // leading dimension of B and C

    hipComplex hCsrValA[] = {
            make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0),
            make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(9, 0)};

    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    hipComplex hB[] = {
            make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(6, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(8, 0)};

    hipComplex hC[] = {
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};
    hipComplex hC_result[] = {
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
            make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};

    hipComplex alpha = make_hipComplex(1, 0);

    // Device memory management
    hipComplex *dCsrValA;
    int *dCsrRowPtrA, *dCsrColIndA;
    hipComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  m * nrhs * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  m * nrhs * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, m * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, m * nrhs * sizeof(hipComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    csrsm2Info_t info = 0;
    hipsparseCreateCsrsm2Info(&info);
    size_t pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    int algo = 0;

    hipsparseCcsrsm2_bufferSizeExt(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz,  &alpha, descrA,
                               dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseCcsrsm2_analysis(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXcsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // WHERE IS C???
    hipsparseCcsrsm2_solve(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXcsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, m * nrhs * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m * nrhs; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csrm2 test PASSED\n");
    else
        printf("csrm2 test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
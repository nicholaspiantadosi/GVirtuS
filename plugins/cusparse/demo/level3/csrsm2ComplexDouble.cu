#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int m = 4;
    const int nrhs = 4;
    const int nnz = 9;
    const int ldb = m; // leading dimension of B and C

    hipDoubleComplex hCsrValA[] = {
            make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(5, 0),
            make_hipDoubleComplex(6, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(9, 0)};
    
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    hipDoubleComplex hB[] = {
            make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0)};

    hipDoubleComplex hC[] = {
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};
    hipDoubleComplex hC_result[] = {
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
            make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};

    hipDoubleComplex alpha = make_hipDoubleComplex(1, 0);

    // Device memory management
    hipDoubleComplex *dCsrValA;
    int *dCsrRowPtrA, *dCsrColIndA;
    hipDoubleComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  m * nrhs * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  m * nrhs * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, m * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, m * nrhs * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    csrsm2Info_t info = 0;
    hipsparseCreateCsrsm2Info(&info);
    size_t pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    int algo = 0;

    hipsparseZcsrsm2_bufferSizeExt(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz,  &alpha, descrA,
                               dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseZcsrsm2_analysis(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                             dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXcsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // WHERE IS C???
    hipsparseZcsrsm2_solve(handle, algo, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nrhs, nnz, &alpha, descrA,
                          dCsrValA, dCsrRowPtrA, dCsrColIndA, dB, ldb, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXcsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, m * nrhs * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < m * nrhs; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dCsrValA) );
    CHECK_CUDA( hipFree(dCsrRowPtrA) );
    CHECK_CUDA( hipFree(dCsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
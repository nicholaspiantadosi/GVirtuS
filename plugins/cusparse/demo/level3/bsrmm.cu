#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    float hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 4};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    float hB[] = {1, 2, 0, 3, 0, 0, 0, 4, 0, 0, 5, 6, 0, 0, 7, 8};

    float hC[] = {0, 0, 1, 0, 0, 2, 3, 0, 4, 5, 0, 0, 6, 7, 0, 0};
    float hC_result[] = {10, 8, 27, 28, 12, 2, 31, 16, 32, 5, 72, 24, 44, 7, 98, 32};

    float alpha = 1;
    float beta = 1;

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    float *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    float *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(float)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    // perform C:=alpha*A*B + beta*C
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, kb, nnzb, &alpha, descrA, dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, dB, ldb, &beta, dC, ldc);

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if (hC[i] != hC_result[i]) { // direct floating point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("bsrmm test PASSED\n");
    else
        printf("bsrmm test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
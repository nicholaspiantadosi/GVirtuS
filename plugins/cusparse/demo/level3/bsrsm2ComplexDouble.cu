#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "../level2/utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int blockSize = 2;
    const int mb = 2;
    const int kb = 2;
    const int nnzb = 4;

    hipDoubleComplex hBsrValA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                            make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0),
                            make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0),
                            make_hipDoubleComplex(6, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(4, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipDoubleComplex hB[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0)};

    hipDoubleComplex hC[] = {make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
                      make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0)};
    hipDoubleComplex hC_result[] = {make_hipDoubleComplex(1, 0),make_hipDoubleComplex(0.500000, 0),make_hipDoubleComplex(-0.833333, 0),make_hipDoubleComplex(-0.250000, 0),
                             make_hipDoubleComplex(0, 0),make_hipDoubleComplex(0, 0),make_hipDoubleComplex(0, 0),make_hipDoubleComplex(1, 0),
                             make_hipDoubleComplex(0, 0),make_hipDoubleComplex(0, 0),make_hipDoubleComplex(0.833333, 0),make_hipDoubleComplex(1.500000, 0),
                             make_hipDoubleComplex(0, 0),make_hipDoubleComplex(0, 0),make_hipDoubleComplex(1.166667, 0),make_hipDoubleComplex(2, 0)};

    hipDoubleComplex alpha = make_hipDoubleComplex(1, 0);

    // A is mb*kb, B is k*n and C is m*n
    const int m = mb*blockSize;
    const int k = kb*blockSize;
    const int ldb = k; // leading dimension of B
    const int ldc = m; // leading dimension of C

    // Device memory management
    hipDoubleComplex *dBsrValA;
    int *dBsrRowPtrA, *dBsrColIndA;
    hipDoubleComplex *dB, *dC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockSize * blockSize) * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dB,  ldb * m * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dC,  ldc * m * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockSize * blockSize) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB, hB, ldb * m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC, hC, ldc * m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);

    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    bsrsm2Info_t info = 0;
    hipsparseCreateBsrsm2Info(&info);
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;

    hipsparseZbsrsm2_bufferSize(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                               dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseZbsrsm2_analysis(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, descrA,
                             dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    hipsparseStatus_t status = hipsparseXbsrsm2_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseZbsrsm2_solve(handle, HIPSPARSE_DIRECTION_COLUMN, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, m, nnzb, &alpha, descrA,
                          dBsrValA, dBsrRowPtrA, dBsrColIndA, blockSize, info, dB, ldb, dC, ldc, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);

    status = hipsparseXbsrsm2_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is missing\n", numerical_zero, numerical_zero);
    }

    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, ldc * m * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < ldc * m; i++) {
        if((fabs(hC[i].x - hC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("bsrm2 test PASSED\n");
    else
        printf("bsrm2 test FAILED: wrong result\n");

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // device memory deallocation
    CHECK_CUDA( hipFree(dBsrValA) );
    CHECK_CUDA( hipFree(dBsrRowPtrA) );
    CHECK_CUDA( hipFree(dBsrColIndA) );
    CHECK_CUDA( hipFree(dB) );
    CHECK_CUDA( hipFree(dC) );
    return EXIT_SUCCESS;
}
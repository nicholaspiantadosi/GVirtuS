#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 3;
    int n = 2;
    int nnz = 4;

    int hCscColPtr[] = {0, 2, 4};
    int hCscRowInd[] = {1, 0, 2, 0};

    int hCscRowInd_result[] = {0, 1, 0, 2};

    // Device memory management
    int *dCscColPtr, *dCscRowInd, *dp;

    CHECK_CUDA( hipMalloc((void**) &dCscColPtr, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dp, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCscColPtr, hCscColPtr, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowInd, hCscRowInd, nnz * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    size_t pBufferSizeInBytes = 0;
    void *pBuffer = NULL;

    CHECK_CUSPARSE(hipsparseXcscsort_bufferSizeExt(handle, m, n, nnz, dCscColPtr, dCscRowInd, &pBufferSizeInBytes));

    if (pBufferSizeInBytes == 0) {
        pBufferSizeInBytes = 1280;
    }

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSizeInBytes * sizeof(char)));

    CHECK_CUSPARSE(hipsparseCreateIdentityPermutation(handle, nnz, dp));

    CHECK_CUSPARSE(hipsparseXcscsort(handle, m, n, nnz, descrA, dCscColPtr, dCscRowInd, dp, pBuffer));

    // device result check
    CHECK_CUDA( hipMemcpy(hCscRowInd, dCscRowInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCscRowInd[i] - hCscRowInd_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("cscsort test PASSED\n");
    else
        printf("cscsort test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCscColPtr) );
    CHECK_CUDA(hipFree(dCscRowInd) );
    CHECK_CUDA(hipFree(dp) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
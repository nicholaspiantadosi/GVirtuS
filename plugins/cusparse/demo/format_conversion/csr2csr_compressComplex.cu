#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "utilities.h"

int main(){
    int m = 6, n = 5;
    hipsparseHandle_t  handle;
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    hipsparseMatDescr_t descrX;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrX));
    int total_nnz = 13;

    hipComplex *csrValX;
    int *csrRowPtrX;
    int *csrColIndX;
    CHECK_CUDA( hipMalloc((void**) &csrValX, sizeof(hipComplex) * total_nnz) );
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrX, sizeof(int) * (m+1))) ;
    CHECK_CUDA( hipMalloc((void**) &csrColIndX, sizeof(int) * total_nnz)) ;

    hipComplex hCsrVal[] = {make_hipComplex(1, 0), make_hipComplex(3, 0), make_hipComplex(-4, 0), make_hipComplex(5, 0), make_hipComplex(2, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(6, 0), make_hipComplex(9, 0), make_hipComplex(3.5, 0), make_hipComplex(5.5, 0), make_hipComplex(6.5, 0), make_hipComplex(-9.9, 0)};
    int hCsrRowPtrX[] = {0, 2, 4, 7, 9, 11, 13};
    int hCsrColIndX[] = {0, 1, 1, 2, 0, 3, 4, 2, 4, 3, 4, 0, 2};

    CHECK_CUDA( hipMemcpy(csrValX, hCsrVal, total_nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrRowPtrX, hCsrRowPtrX, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrColIndX, hCsrColIndX, total_nnz * sizeof(int), hipMemcpyHostToDevice) );

    hipComplex tol = make_hipComplex(3.5, 0);
    int *nnzPerRowY;
    int testNNZTotal;
    CHECK_CUDA( hipMalloc((void**) &nnzPerRowY,  m * sizeof(int)));
    CHECK_CUSPARSE( hipsparseCnnz_compress(handle, m, descrX, csrValX,
                                         csrRowPtrX, nnzPerRowY,
                                         &testNNZTotal, tol));
    hipComplex *csrValY;
    int *csrRowPtrY;
    int *csrColIndY;
    CHECK_CUDA( hipMalloc((void**) &csrValY, sizeof(hipComplex) * testNNZTotal));
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrY, sizeof(int) * (m+1)));
    CHECK_CUDA( hipMalloc((void**) &csrColIndY, sizeof(int) * testNNZTotal));

    CHECK_CUSPARSE( hipsparseCcsr2csr_compress( handle, m, n, descrX, csrValX,
                                              csrColIndX, csrRowPtrX,
                                              total_nnz,  nnzPerRowY,
                                              csrValY, csrColIndY,
                                              csrRowPtrY, tol));

    int hNnzPerRowY[m];
    hipComplex hCsrValY[testNNZTotal];
    int hCsrRowPtrY[m + 1];
    int hCsrColIndY[testNNZTotal];

    CHECK_CUDA( hipMemcpy(hNnzPerRowY, nnzPerRowY, m * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrValY, csrValY, testNNZTotal * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrY, csrRowPtrY, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndY, csrColIndY, testNNZTotal * sizeof(int), hipMemcpyDeviceToHost) );

    int hNnzPerRowY_result[] = {0, 2, 2, 2, 1, 2};
    int hCsrRowPtrY_result[] = {0, 0, 2, 4, 6, 7, 9};

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hNnzPerRowY[i] - hNnzPerRowY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrY[i] - hCsrRowPtrY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2csr_compressed test PASSED\n");
    else
        printf("csr2csr_compressed test FAILED: wrong result\n");

    hipsparseDestroy(handle);
    hipFree(csrValX);
    hipFree(csrRowPtrX);
    hipFree(csrColIndX);
    hipFree(csrValY);
    hipFree(nnzPerRowY);
    hipFree(csrRowPtrY);
    hipFree(csrColIndY);
    return 0;
}

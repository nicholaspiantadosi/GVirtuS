#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "utilities.h"

int main(){
    int m = 6, n = 5;
    hipsparseHandle_t  handle;
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    hipsparseMatDescr_t descrX;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrX));
    int total_nnz = 13;

    hipDoubleComplex *csrValX;
    int *csrRowPtrX;
    int *csrColIndX;
    CHECK_CUDA( hipMalloc((void**) &csrValX, sizeof(hipDoubleComplex) * total_nnz) );
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrX, sizeof(int) * (m+1))) ;
    CHECK_CUDA( hipMalloc((void**) &csrColIndX, sizeof(int) * total_nnz)) ;

    hipDoubleComplex hCsrVal[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(-4, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(3.5, 0), make_hipDoubleComplex(5.5, 0), make_hipDoubleComplex(6.5, 0), make_hipDoubleComplex(-9.9, 0)};
    int hCsrRowPtrX[] = {0, 2, 4, 7, 9, 11, 13};
    int hCsrColIndX[] = {0, 1, 1, 2, 0, 3, 4, 2, 4, 3, 4, 0, 2};

    CHECK_CUDA( hipMemcpy(csrValX, hCsrVal, total_nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrRowPtrX, hCsrRowPtrX, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrColIndX, hCsrColIndX, total_nnz * sizeof(int), hipMemcpyHostToDevice) );

    hipDoubleComplex tol = make_hipDoubleComplex(3.5, 0);
    int *nnzPerRowY;
    int testNNZTotal;
    CHECK_CUDA( hipMalloc((void**) &nnzPerRowY,  m * sizeof(int)));
    CHECK_CUSPARSE( hipsparseZnnz_compress(handle, m, descrX, csrValX,
                                         csrRowPtrX, nnzPerRowY,
                                         &testNNZTotal, tol));
    hipDoubleComplex *csrValY;
    int *csrRowPtrY;
    int *csrColIndY;
    CHECK_CUDA( hipMalloc((void**) &csrValY, sizeof(hipDoubleComplex) * testNNZTotal));
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrY, sizeof(int) * (m+1)));
    CHECK_CUDA( hipMalloc((void**) &csrColIndY, sizeof(int) * testNNZTotal));

    CHECK_CUSPARSE( hipsparseZcsr2csr_compress( handle, m, n, descrX, csrValX,
                                              csrColIndX, csrRowPtrX,
                                              total_nnz,  nnzPerRowY,
                                              csrValY, csrColIndY,
                                              csrRowPtrY, tol));

    int hNnzPerRowY[m];
    hipDoubleComplex hCsrValY[testNNZTotal];
    int hCsrRowPtrY[m + 1];
    int hCsrColIndY[testNNZTotal];

    CHECK_CUDA( hipMemcpy(hNnzPerRowY, nnzPerRowY, m * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrValY, csrValY, testNNZTotal * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrY, csrRowPtrY, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndY, csrColIndY, testNNZTotal * sizeof(int), hipMemcpyDeviceToHost) );

    int hNnzPerRowY_result[] = {0, 2, 2, 2, 1, 2};
    int hCsrRowPtrY_result[] = {0, 0, 2, 4, 6, 7, 9};

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hNnzPerRowY[i] - hNnzPerRowY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrY[i] - hCsrRowPtrY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2csr_compressed test PASSED\n");
    else
        printf("csr2csr_compressed test FAILED: wrong result\n");

    hipsparseDestroy(handle);
    hipFree(csrValX);
    hipFree(csrRowPtrX);
    hipFree(csrColIndX);
    hipFree(csrValY);
    hipFree(nnzPerRowY);
    hipFree(csrRowPtrY);
    hipFree(csrColIndY);
    return 0;
}

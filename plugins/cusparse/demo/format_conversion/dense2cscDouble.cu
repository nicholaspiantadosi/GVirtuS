#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnz = 9;
    int lda = m;

    double hA [] = {1, 0, 5, 0,
                  4, 2, 0, 0,
                  0, 3, 0, 9,
                  0, 0, 7, 0,
                  0, 0, 8, 6};

    int hNnzPerCol[] = {2, 2, 2, 1, 2};

    double hCscValA[nnz];
    int hCscRowIndA[nnz];
    int hCscColPtrA[n + 1];

    double hCscValA_result[] = {1, 5, 4, 2, 3, 9, 7, 8, 6};
    int hCscRowIndA_result[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    int hCscColPtrA_result[] = {0, 2, 4, 6, 7, 9};

    // Device memory management
    double *dCscValA, *dA;
    int *dCscRowIndA, *dCscColPtrA, *dNnzPerCol;

    CHECK_CUDA( hipMalloc((void**) &dCscValA,  nnz * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrA, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dNnzPerCol, n * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(double)));

    CHECK_CUDA( hipMemcpy(dA, hA, m * n * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dNnzPerCol, hNnzPerCol, n * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseDdense2csc(handle, m, n, descrA, dA, lda, dNnzPerCol, dCscValA, dCscRowIndA, dCscColPtrA);

    // device result check
    CHECK_CUDA( hipMemcpy(hCscValA, dCscValA, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCscRowIndA, dCscRowIndA, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCscColPtrA, dCscColPtrA, (n + 1) * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCscValA[i] - hCscValA_result[i]) > 0.000001) || (fabs(hCscRowIndA[i] - hCscRowIndA_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (n + 1); i++) {
        if((fabs(hCscColPtrA[i] - hCscColPtrA_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("dense2csc test PASSED\n");
    else
        printf("dense2csc test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCscValA) );
    CHECK_CUDA(hipFree(dCscRowIndA) );
    CHECK_CUDA(hipFree(dCscColPtrA) );
    CHECK_CUDA(hipFree(dA) );
    CHECK_CUDA(hipFree(dNnzPerCol) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
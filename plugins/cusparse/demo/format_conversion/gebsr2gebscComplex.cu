#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int rowBlockDim = 2;
    const int colBlockDim = 2;
    const int mb = 2;
    const int nb = 2;
    const int nnzb = 4;

    hipComplex hBsrValA[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                            make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipComplex hBscVal[] = {make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                           make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                           make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                           make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(0, 0)};
    int hBscRowInd[] = {0, 0, 0, 0};
    int hBscColPtr[] = {0, 0, 0};

    hipComplex hBscVal_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                                  make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0),
                                  make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                                  make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};
    int hBscRowInd_result[] = {0, 1, 0, 1};
    int hBscColPtr_result[] = {0, 2, 4};

    // Device memory management
    hipComplex *dBsrValA, *dBscVal;
    int *dBsrRowPtrA, *dBsrColIndA, *dBscRowInd, *dBscColPtr;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (rowBlockDim * colBlockDim) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBscVal,  nnzb * (rowBlockDim * colBlockDim) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBscRowInd, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBscColPtr, (nb + 1) * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (rowBlockDim * colBlockDim) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscVal, hBscVal, nnzb * (rowBlockDim * colBlockDim) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscRowInd, hBscRowInd, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscColPtr, hBscColPtr, (nb + 1) * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    int pBufferSize;
    void *pBuffer = 0;

    hipsparseCgebsr2gebsc_bufferSize(handle, mb, nb, nnzb, dBsrValA, dBsrRowPtrA, dBsrColIndA, rowBlockDim, colBlockDim, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseCgebsr2gebsc(handle, mb, nb, nnzb, dBsrValA, dBsrRowPtrA, dBsrColIndA, rowBlockDim,
                         colBlockDim, dBscVal, dBscRowInd, dBscColPtr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hBscVal, dBscVal, nnzb * (rowBlockDim * colBlockDim) * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBscRowInd, dBscRowInd, nnzb * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBscColPtr, dBscColPtr, (nb + 1) * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnzb * (rowBlockDim * colBlockDim); i++) {
        if((fabs(hBscVal[i].x - hBscVal_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzb; i++) {
        if((fabs(hBscRowInd[i] - hBscRowInd_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (nb + 1); i++) {
        if((fabs(hBscColPtr[i] - hBscColPtr_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gebsr2gebsc test PASSED\n");
    else
        printf("gebsr2gebsc test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );
    CHECK_CUDA(hipFree(dBscVal) );
    CHECK_CUDA(hipFree(dBscRowInd) );
    CHECK_CUDA(hipFree(dBscColPtr) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
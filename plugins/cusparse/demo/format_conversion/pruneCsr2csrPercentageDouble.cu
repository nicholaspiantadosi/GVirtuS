#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnzA = 9;

    double hCsrValA[] = {1, 4, 2, 3, 5, 7, 8, 9, 6};
    int hCsrRowPtrA[] = {0, 2, 4, 7, 9};
    int hCsrColIndA[] = {0, 1, 1, 2, 0, 3, 4, 2, 4};

    float percentage = 40;

    double hCsrValC_result[] = {5, 7, 8, 9, 6};
    int hCsrRowPtrC_result[] = {0, 0, 0, 3, 5};
    int hCsrColIndC_result[] = {0, 3, 4, 2, 4};

    // Device memory management
    double *dCsrValA, *dCsrValC;
    int *dCsrRowPtrA, *dCsrColIndA, *dCsrRowPtrC, *dCsrColIndC;

    CHECK_CUDA(hipMalloc((void**) &dCsrValA,  nnzA * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**) &dCsrRowPtrA,  (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**) &dCsrColIndA,  nnzA * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dCsrRowPtrC, sizeof(int) * (m + 1)));

    CHECK_CUDA(hipMemcpy(dCsrValA, hCsrValA, nnzA * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dCsrColIndA, hCsrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL ));

    hipsparseMatDescr_t descrC = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL ));

    pruneInfo_t info;
    CHECK_CUSPARSE(hipsparseCreatePruneInfo(&info));

    size_t pBufferSize;
    void *pBuffer = 0;

    CHECK_CUSPARSE(hipsparseDpruneCsr2csrByPercentage_bufferSizeExt(handle, m, n, nnzA, descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA, percentage, descrC, NULL, dCsrRowPtrC, NULL, info, &pBufferSize));

    pBufferSize = 512;

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSize));

    int nnzc;
    int *nnzTotalDevHostPtr = &nnzc;

    CHECK_CUSPARSE(hipsparseDpruneCsr2csrNnzByPercentage(handle, m, n, nnzA, descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA, percentage, descrC, dCsrRowPtrC, nnzTotalDevHostPtr, info, pBuffer));

    nnzc = *nnzTotalDevHostPtr;

    CHECK_CUDA(hipMalloc((void**)&dCsrValC, sizeof(double) * nnzc));
    CHECK_CUDA(hipMalloc((void**)&dCsrColIndC, sizeof(int) * nnzc));

    CHECK_CUSPARSE(hipsparseDpruneCsr2csrByPercentage(handle, m, n, nnzA, descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA, percentage, descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC, info, pBuffer));

    // device result check

    double hCsrValC[nnzc];
    int hCsrRowPtrC[m + 1];
    int hCsrColIndC[nnzc];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzc * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzc * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    if (nnzc != 5) {
        correct = 0;
    }
    for (int i = 0; i < nnzc; i++) {
        if((fabs(hCsrValC[i] - hCsrValC_result[i]) > 0.000001) || (fabs(hCsrColIndC[i] - hCsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrC[i] - hCsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("pruneCsr2csrPercentage test PASSED\n");
    else
        printf("pruneCsr2csrPercentage test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValC) );
    CHECK_CUDA(hipFree(dCsrRowPtrC) );
    CHECK_CUDA(hipFree(dCsrColIndC) );
    CHECK_CUDA(hipFree(dCsrValA) );
    CHECK_CUDA(hipFree(dCsrRowPtrA) );
    CHECK_CUDA(hipFree(dCsrColIndA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
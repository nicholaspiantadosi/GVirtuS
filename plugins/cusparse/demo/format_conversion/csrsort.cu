#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 3;
    int n = 3;
    int nnz = 9;

    float hCsrVal[] = {3, 2, 1, 4, 6, 5, 8, 9, 7};
    int hCsrRowPtr[] = {0, 3, 6, 9};
    int hCsrColInd[] = {2, 1, 0, 0, 2, 1, 1, 2, 0};

    int hCsrColInd_result[] = {0, 1, 2, 0, 1, 2, 0, 1, 2};

    // Device memory management
    float *dCsrVal;
    int *dCsrRowPtr, *dCsrColInd, *dp;

    //CHECK_CUDA( hipMalloc((void**) &dCooValA,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrVal, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dp, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    size_t pBufferSizeInBytes = 0;
    void *pBuffer = NULL;

    CHECK_CUSPARSE(hipsparseXcsrsort_bufferSizeExt(handle, m, n, nnz, dCsrRowPtr, dCsrColInd, &pBufferSizeInBytes));

    if (pBufferSizeInBytes == 0) {
        pBufferSizeInBytes = 1280;
    }

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSizeInBytes * sizeof(char)));

    CHECK_CUSPARSE(hipsparseCreateIdentityPermutation(handle, nnz, dp));

    CHECK_CUSPARSE(hipsparseXcsrsort(handle, m, n, nnz, descrA, dCsrRowPtr, dCsrColInd, dp, pBuffer));

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrColInd, dCsrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCsrColInd[i] - hCsrColInd_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csrsort test PASSED\n");
    else
        printf("csrsort test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dCsrColInd) );
    CHECK_CUDA(hipFree(dCsrRowPtr) );
    CHECK_CUDA(hipFree(dCsrVal) );
    CHECK_CUDA(hipFree(dp) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
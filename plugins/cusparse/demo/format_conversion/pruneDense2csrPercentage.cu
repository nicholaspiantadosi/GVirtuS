#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int lda = m;

    float hA [] = {1, 0, 5, 0,
                    4, 2, 0, 0,
                    0, 3, 0, 9,
                    0, 0, 7, 0,
                    0, 0, 8, 6};

    float percentage = 90;

    float hCsrValC_result[] = {8, 9};
    int hCsrRowPtrC_result[] = {0, 0, 0, 1, 2};
    int hCsrColIndC_result[] = {4, 2};

    // Device memory management
    float *dA;
    float *dCsrValC;
    int *dCsrRowPtrC, *dCsrColIndC;

    CHECK_CUDA(hipMalloc((void**) &dA,  m * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dCsrRowPtrC, sizeof(int) * (m + 1)));

    CHECK_CUDA(hipMemcpy(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrC = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL ));

    pruneInfo_t info;
    CHECK_CUSPARSE(hipsparseCreatePruneInfo(&info));

    size_t pBufferSize;
    void *pBuffer = 0;

    CHECK_CUSPARSE(hipsparseSpruneDense2csrByPercentage_bufferSizeExt(handle, m, n, dA, lda, percentage, descrC, NULL, dCsrRowPtrC, NULL, info, &pBufferSize));

    if(pBufferSize == 0) {
        pBufferSize = 512;
    }

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSize));

    int nnzc;
    int *nnzTotalDevHostPtr = &nnzc;

    CHECK_CUSPARSE(hipsparseSpruneDense2csrNnzByPercentage(handle, m, n, dA, lda, percentage, descrC, dCsrRowPtrC, nnzTotalDevHostPtr, info, pBuffer));

    nnzc = *nnzTotalDevHostPtr;

    CHECK_CUDA(hipMalloc((void**)&dCsrValC, sizeof(float) * nnzc));
    CHECK_CUDA(hipMalloc((void**)&dCsrColIndC, sizeof(int) * nnzc));

    CHECK_CUSPARSE(hipsparseSpruneDense2csrByPercentage(handle, m, n, dA, lda, percentage, descrC, dCsrValC, dCsrRowPtrC, dCsrColIndC, info, pBuffer));

    // device result check

    float hCsrValC[nnzc];
    int hCsrRowPtrC[m + 1];
    int hCsrColIndC[nnzc];

    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnzc * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnzc * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    if (nnzc != 2) {
        correct = 0;
    }
    for (int i = 0; i < nnzc; i++) {
        if((fabs(hCsrValC[i] - hCsrValC_result[i]) > 0.000001) || (fabs(hCsrColIndC[i] - hCsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrC[i] - hCsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("pruneDense2csrPercentage test PASSED\n");
    else
        printf("pruneDense2csrPercentage test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValC) );
    CHECK_CUDA(hipFree(dCsrRowPtrC) );
    CHECK_CUDA(hipFree(dCsrColIndC) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
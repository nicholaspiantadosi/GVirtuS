#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(void)
{
    // Host problem definition
    int m = 3;
    int n = 3;
    int nnz = 9;

    double hCsrVal[] = {3, 2, 1, 4, 6, 5, 8, 9, 7};
    int hCsrRowPtr[] = {0, 3, 6, 9};
    int hCsrColInd[] = {2, 1, 0, 0, 2, 1, 1, 2, 0};

    double hCsrVal_sorted[nnz];
    int hCsrRowPtr_sorted[m + 1];
    int hCsrColInd_sorted[nnz];

    double hCsrVal_sorted_result[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtr_sorted_result[] = {0, 3, 6, 9};
    int hCsrColInd_sorted_result[] = {0, 1, 2, 0, 1, 2, 0, 1 , 2};

    double hCsrVal_unsorted[nnz];
    int hCsrRowPtr_unsorted[m + 1];
    int hCsrColInd_unsorted[nnz];

    // Device memory management
    double *dCsrVal;
    int *dCsrRowPtr, *dCsrColInd;

    CHECK_CUDA( hipMalloc((void**) &dCsrVal, nnz * sizeof(double)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    csru2csrInfo_t info;
    CHECK_CUSPARSE(hipsparseCreateCsru2csrInfo(&info));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    size_t pBufferSize = 0;
    void *pBuffer = NULL;

    CHECK_CUSPARSE(hipsparseDcsru2csr_bufferSizeExt(handle, m, n, nnz, dCsrVal, dCsrRowPtr, dCsrColInd, info, &pBufferSize));

    if (pBufferSize == 0) {
        pBufferSize = 1408;
    }

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSize * sizeof(char)));

    CHECK_CUSPARSE(hipsparseDcsru2csr(handle, m, n, nnz, descrA, dCsrVal, dCsrRowPtr, dCsrColInd, info, pBuffer));

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrVal_sorted, dCsrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtr_sorted, dCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColInd_sorted, dCsrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCsrVal_sorted[i] - hCsrVal_sorted_result[i]) > 0.000001)
            || (fabs(hCsrColInd_sorted[i] - hCsrColInd_sorted_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtr_sorted[i] - hCsrRowPtr_sorted_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csru2csr test PASSED\n");
    else
        printf("csru2csr test FAILED: wrong result\n");

    if (correct) {
        CHECK_CUSPARSE(hipsparseDcsr2csru(handle, m, n, nnz, descrA, dCsrVal, dCsrRowPtr, dCsrColInd, info, pBuffer));

        // device result check
        CHECK_CUDA( hipMemcpy(hCsrVal_unsorted, dCsrVal, nnz * sizeof(double), hipMemcpyDeviceToHost) );
        CHECK_CUDA( hipMemcpy(hCsrRowPtr_unsorted, dCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
        CHECK_CUDA( hipMemcpy(hCsrColInd_unsorted, dCsrColInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

        correct = 1;
        for (int i = 0; i < nnz; i++) {
            if((fabs(hCsrVal_unsorted[i] - hCsrVal[i]) > 0.000001)
               || (fabs(hCsrColInd_unsorted[i] - hCsrColInd[i]) > 0.000001)) {
                correct = 0;
                break;
            }
        }
        for (int i = 0; i < (m + 1); i++) {
            if((fabs(hCsrRowPtr_unsorted[i] - hCsrRowPtr_sorted[i]) > 0.000001)) {
                correct = 0;
                break;
            }
        }

        if (correct)
            printf("csr2csru test PASSED\n");
        else
            printf("csr2csru test FAILED: wrong result\n");
    }

    // device memory deallocation
    CHECK_CUDA(hipFree(dCsrColInd) );
    CHECK_CUDA(hipFree(dCsrRowPtr) );
    CHECK_CUDA(hipFree(dCsrVal) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUSPARSE(hipsparseDestroyCsru2csrInfo(info));

    return EXIT_SUCCESS;
}

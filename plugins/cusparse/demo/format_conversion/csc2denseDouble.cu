#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnz = 9;
    int lda = m;

    double hCscValA[] = {1, 5, 4, 2, 3, 9, 7, 8, 6};
    int hCscRowIndA[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    int hCscColPtrA[] = {0, 2, 4, 6, 7, 9};

    double hA[m * n];

    double hA_result[] = {1, 0, 5, 0,
                         4, 2, 0, 0,
                         0, 3, 0, 9,
                         0, 0, 7, 0,
                         0, 0, 8, 6};

    // Device memory management
    double *dCscValA, *dA;
    int *dCscRowIndA, *dCscColPtrA;

    CHECK_CUDA( hipMalloc((void**) &dCscValA,  nnz * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrA, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(double)));

    CHECK_CUDA( hipMemcpy(dCscValA, hCscValA, nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndA, hCscRowIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrA, hCscColPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseDcsc2dense(handle, m, n, descrA, dCscValA, dCscRowIndA, dCscColPtrA, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(double), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i] - hA_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csc2dense test PASSED\n");
    else
        printf("csc2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCscValA) );
    CHECK_CUDA(hipFree(dCscRowIndA) );
    CHECK_CUDA(hipFree(dCscColPtrA) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
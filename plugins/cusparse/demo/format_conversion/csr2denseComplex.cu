#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 4;
    int nnz = 9;
    int lda = m;

    hipComplex hCsrVal[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0), make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(9, 0)};
    int hCsrRowPtr[] = {0, 3, 4, 7, 9};
    int hCsrColInd[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    hipComplex hA[m * n];

    hipComplex hA_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(0, 0),
                             make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(0, 0), make_hipComplex(8, 0),
                             make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(6, 0), make_hipComplex(0, 0),
                             make_hipComplex(3, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};

    // Device memory management
    hipComplex *dCsrVal, *dA;
    int *dCsrRowPtr, *dCsrColInd;

    CHECK_CUDA( hipMalloc((void**) &dCsrVal,  nnz * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseCcsr2dense(handle, m, n, descrA, dCsrVal, dCsrRowPtr, dCsrColInd, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i].x - hA_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2dense test PASSED\n");
    else
        printf("csr2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCsrVal) );
    CHECK_CUDA(hipFree(dCsrRowPtr) );
    CHECK_CUDA(hipFree(dCsrColInd) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
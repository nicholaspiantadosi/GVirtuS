#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 4;
    int nnz = 9;

    const int rowBlockDim = 2;
    const int colBlockDim = 2;

    int mb = (m + rowBlockDim-1)/rowBlockDim;

    hipComplex hCsrValA[] = {make_hipComplex(1, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(4, 0), make_hipComplex(5, 0),
                            make_hipComplex(6, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(9, 0)};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    hipComplex hBsrValC_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0),
                                   make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                                   make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0),
                                   make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};
    int hBsrRowPtrC_result[] = {0, 2, 4};
    int hBsrColIndC_result[] = {0, 1, 0, 1};

    // Device memory management
    hipComplex *dCsrValA, *dBsrValC;
    int *dCsrRowPtrA, *dCsrColIndA, *dBsrRowPtrC, *dBsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    // Given CSR format (csrRowPtrA, csrColIndA, csrValA) and
    // blocks of BSR format are stored in column-major order.
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
    int base, nnzb;
    int bufferSize;
    void *pBuffer;
    hipsparseCcsr2gebsr_bufferSize(handle, dir, m, n,
                                  descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA,
                                  rowBlockDim, colBlockDim,
                                  &bufferSize);

    if (bufferSize == 0) {
        bufferSize = 528;
    }

    hipMalloc((void**)&pBuffer, bufferSize);
    hipMalloc((void**)&dBsrRowPtrC, sizeof(int) *(mb+1));
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzb;
    hipsparseXcsr2gebsrNnz(handle, dir, m, n,
                          descrA, dCsrRowPtrA, dCsrColIndA,
                          descrC, dBsrRowPtrC, rowBlockDim, colBlockDim,
                          nnzTotalDevHostPtr,
                          pBuffer);
    if (NULL != nnzTotalDevHostPtr){
        nnzb = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzb, dBsrRowPtrC+mb, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, dBsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzb -= base;
    }
    hipMalloc((void**)&dBsrColIndC, sizeof(int)*nnzb);
    hipMalloc((void**)&dBsrValC, sizeof(hipComplex)*(rowBlockDim*colBlockDim)*nnzb);
    hipsparseCcsr2gebsr(handle, dir, m, n,
                       descrA,
                       dCsrValA, dCsrRowPtrA, dCsrColIndA,
                       descrC,
                       dBsrValC, dBsrRowPtrC, dBsrColIndC,
                       rowBlockDim, colBlockDim,
                       pBuffer);

    hipComplex hBsrValC[nnzb * rowBlockDim * colBlockDim];
    int hBsrRowPtrC[mb + 1];
    int hBsrColIndC[nnzb];

    // device result check
    CHECK_CUDA( hipMemcpy(hBsrValC, dBsrValC, nnzb * rowBlockDim * colBlockDim * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrRowPtrC, dBsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrColIndC, dBsrColIndC, nnzb * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnzb * rowBlockDim * colBlockDim; i++) {
        if((fabs(hBsrValC[i].x - hBsrValC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (mb + 1); i++) {
        if((fabs(hBsrRowPtrC[i] - hBsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzb; i++) {
        if((fabs(hBsrColIndC[i] - hBsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2gebsr test PASSED\n");
    else
        printf("csr2gebsr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValA) );
    CHECK_CUDA(hipFree(dCsrRowPtrA) );
    CHECK_CUDA(hipFree(dCsrColIndA) );
    CHECK_CUDA(hipFree(dBsrValC) );
    CHECK_CUDA(hipFree(dBsrRowPtrC) );
    CHECK_CUDA(hipFree(dBsrColIndC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnz = 9;
    int lda = m;

    hipDoubleComplex hCscValA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(9, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(6, 0)};
    int hCscRowIndA[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    int hCscColPtrA[] = {0, 2, 4, 6, 7, 9};

    hipDoubleComplex hA[m * n];

    hipDoubleComplex hA_result[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0),
                             make_hipDoubleComplex(4, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0),
                             make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(9, 0),
                             make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(0, 0),
                             make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(6, 0)};

    // Device memory management
    hipDoubleComplex *dCscValA, *dA;
    int *dCscRowIndA, *dCscColPtrA;

    CHECK_CUDA( hipMalloc((void**) &dCscValA,  nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrA, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dCscValA, hCscValA, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndA, hCscRowIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrA, hCscColPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseZcsc2dense(handle, m, n, descrA, dCscValA, dCscRowIndA, dCscColPtrA, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i].x - hA_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csc2dense test PASSED\n");
    else
        printf("csc2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCscValA) );
    CHECK_CUDA(hipFree(dCscRowIndA) );
    CHECK_CUDA(hipFree(dCscColPtrA) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int rowBlockDim = 2;
    const int colBlockDim = 2;
    const int mb = 2;
    const int nb = 2;
    const int nnzb = 4;
    int m = mb * rowBlockDim;
    int nnz  = nnzb * rowBlockDim * colBlockDim; // number of elements

    hipDoubleComplex hBsrValA[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0),
                            make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0),
                            make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0),
                            make_hipDoubleComplex(6, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(9, 0)};;
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipDoubleComplex hCsrValC[nnz];
    int hCsrRowPtrC[m + 1];
    int hCsrColIndC[nnz];

    hipDoubleComplex hCsrValC_result[] = {make_hipDoubleComplex(1.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(2.000000, 0), make_hipDoubleComplex(3.000000, 0),
                                   make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(4.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(0.000000, 0),
                                   make_hipDoubleComplex(5.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(6.000000, 0), make_hipDoubleComplex(7.000000, 0),
                                   make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(8.000000, 0), make_hipDoubleComplex(0.000000, 0), make_hipDoubleComplex(9.000000, 0)};
    int hCsrRowPtrC_result[] = {0, 4, 8, 12, 16};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3};

    // Device memory management
    hipDoubleComplex *dBsrValA, *dCsrValC;
    int *dBsrRowPtrA, *dBsrColIndA, *dCsrRowPtrC, *dCsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (rowBlockDim * colBlockDim) * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrValC,  nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrC, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndC, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (rowBlockDim * colBlockDim) * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    // Given general BSR format (bsrRowPtrA, bsrColIndA, bsrValA) and
    // blocks of BSR format are stored in column-major order.
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    hipsparseZgebsr2csr(handle, dir, mb, nb,
                       descrA,
                       dBsrValA, dBsrRowPtrA, dBsrColIndA,
                       rowBlockDim, colBlockDim,
                       descrC,
                       dCsrValC, dCsrRowPtrC, dCsrColIndC);

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCsrValC[i].x - hCsrValC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
        if((fabs(hCsrColIndC[i] - hCsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrC[i] - hCsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gebsr2csr test PASSED\n");
    else
        printf("gebsr2csr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );
    CHECK_CUDA(hipFree(dCsrValC) );
    CHECK_CUDA(hipFree(dCsrRowPtrC) );
    CHECK_CUDA(hipFree(dCsrColIndC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
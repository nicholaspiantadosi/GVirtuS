#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int rowBlockDim = 2;
    const int colBlockDim = 2;
    const int mb = 2;
    const int nb = 2;
    const int nnzb = 4;
    int m = mb * rowBlockDim;
    int nnz  = nnzb * rowBlockDim * colBlockDim; // number of elements

    float hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 9};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    float hCsrValC[nnz];
    int hCsrRowPtrC[m + 1];
    int hCsrColIndC[nnz];

    float hCsrValC_result[] = {1, 0, 2, 3, 0, 4, 0, 0, 5, 0, 6, 7, 0, 8, 0, 9};
    int hCsrRowPtrC_result[] = {0, 4, 8, 12, 16};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3};

    // Device memory management
    float *dBsrValA, *dCsrValC;
    int *dBsrRowPtrA, *dBsrColIndA, *dCsrRowPtrC, *dCsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (rowBlockDim * colBlockDim) * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrValC,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrC, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndC, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (rowBlockDim * colBlockDim) * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    // Given general BSR format (bsrRowPtrA, bsrColIndA, bsrValA) and
    // blocks of BSR format are stored in column-major order.
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    hipsparseSgebsr2csr(handle, dir, mb, nb,
                       descrA,
                       dBsrValA, dBsrRowPtrA, dBsrColIndA,
                       rowBlockDim, colBlockDim,
                       descrC,
                       dCsrValC, dCsrRowPtrC, dCsrColIndC);

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnz * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCsrValC[i] - hCsrValC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
        if((fabs(hCsrColIndC[i] - hCsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrC[i] - hCsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gebsr2csr test PASSED\n");
    else
        printf("gebsr2csr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );
    CHECK_CUDA(hipFree(dCsrValC) );
    CHECK_CUDA(hipFree(dCsrRowPtrC) );
    CHECK_CUDA(hipFree(dCsrColIndC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
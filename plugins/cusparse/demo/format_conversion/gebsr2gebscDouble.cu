#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int rowBlockDim = 2;
    const int colBlockDim = 2;
    const int mb = 2;
    const int nb = 2;
    const int nnzb = 4;

    double hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 4};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    double hBscVal[] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    int hBscRowInd[] = {0, 0, 0, 0};
    int hBscColPtr[] = {0, 0, 0};

    double hBscVal_result[] = {1, 0, 0, 4, 5, 0, 0, 8, 2, 0, 3, 0, 6, 0, 7, 4};
    int hBscRowInd_result[] = {0, 1, 0, 1};
    int hBscColPtr_result[] = {0, 2, 4};

    // Device memory management
    double *dBsrValA, *dBscVal;
    int *dBsrRowPtrA, *dBsrColIndA, *dBscRowInd, *dBscColPtr;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (rowBlockDim * colBlockDim) * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBscVal,  nnzb * (rowBlockDim * colBlockDim) * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dBscRowInd, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBscColPtr, (nb + 1) * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (rowBlockDim * colBlockDim) * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscVal, hBscVal, nnzb * (rowBlockDim * colBlockDim) * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscRowInd, hBscRowInd, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBscColPtr, hBscColPtr, (nb + 1) * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    int pBufferSize;
    void *pBuffer = 0;

    hipsparseDgebsr2gebsc_bufferSize(handle, mb, nb, nnzb, dBsrValA, dBsrRowPtrA, dBsrColIndA, rowBlockDim, colBlockDim, &pBufferSize);

    hipMalloc((void**)&pBuffer, pBufferSize);

    hipsparseDgebsr2gebsc(handle, mb, nb, nnzb, dBsrValA, dBsrRowPtrA, dBsrColIndA, rowBlockDim,
                         colBlockDim, dBscVal, dBscRowInd, dBscColPtr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, pBuffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hBscVal, dBscVal, nnzb * (rowBlockDim * colBlockDim) * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBscRowInd, dBscRowInd, nnzb * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBscColPtr, dBscColPtr, (nb + 1) * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnzb * (rowBlockDim * colBlockDim); i++) {
        if((fabs(hBscVal[i] - hBscVal_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzb; i++) {
        if((fabs(hBscRowInd[i] - hBscRowInd_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (nb + 1); i++) {
        if((fabs(hBscColPtr[i] - hBscColPtr_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gebsr2gebsc test PASSED\n");
    else
        printf("gebsr2gebsc test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );
    CHECK_CUDA(hipFree(dBscVal) );
    CHECK_CUDA(hipFree(dBscRowInd) );
    CHECK_CUDA(hipFree(dBscColPtr) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    int m = 4;
    int n = 4;
    int nnz = 9;

    float hCsrVal[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtr[] = {0, 3, 4, 7, 9};
    int hCsrColInd[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    float hCscVal[nnz];
    int hCscColPtr[n + 1];
    int hCscRowInd[nnz];

    float hCscVal_result[] = {1, 5, 4, 8, 2, 6, 3, 7, 9};
    int hCscColPtr_result[] = {0, 2, 4, 6, 9};
    int hCscRowInd_result[] = {0, 2, 1, 3, 0, 2, 0, 2, 3};

    // Device memory management

    float *dCsrVal, *dCscVal;
    int *dCsrRowPtr, *dCsrColInd, *dCscColPtr, *dCscRowInd;

    CHECK_CUDA( hipMalloc((void**) &dCsrVal, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscVal, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtr, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscRowInd, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t bufferSize;
    void *buffer = 0;

    hipsparseStatus_t cs = hipsparseCsr2cscEx2_bufferSize(handle, m, n, nnz, dCsrVal, dCsrRowPtr, dCsrColInd, dCscVal, dCscColPtr, dCscRowInd, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize);

    hipMalloc((void**)&buffer, bufferSize);

    cs = hipsparseCsr2cscEx2(handle, m, n, nnz, dCsrVal, dCsrRowPtr, dCsrColInd, dCscVal, dCscColPtr, dCscRowInd, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);

    // device result check
    CHECK_CUDA( hipMemcpy(hCscVal, dCscVal, nnz * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCscColPtr, dCscColPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCscRowInd, dCscRowInd, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCscVal[i] - hCscVal_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
        if((fabs(hCscRowInd[i] - hCscRowInd_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (n + 1); i++) {
        if((fabs(hCscColPtr[i] - hCscColPtr_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("csr2cscex2 test PASSED\n");
    else
        printf("csr2cscex2 test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dCsrVal));
    CHECK_CUDA(hipFree(dCsrRowPtr));
    CHECK_CUDA(hipFree(dCsrColInd));
    CHECK_CUDA(hipFree(dCscVal));
    CHECK_CUDA(hipFree(dCscColPtr));
    CHECK_CUDA(hipFree(dCscRowInd));
    CHECK_CUDA(hipFree(buffer));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 4;
    int nnz = 9;
    int lda = m;

    float hCsrVal[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtr[] = {0, 3, 4, 7, 9};
    int hCsrColInd[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    float hA[m * n];

    float hA_result[] = {1, 0, 5, 0,
                         0, 4, 0, 8,
                         2, 0, 6, 0,
                         3, 0, 7, 9};

    // Device memory management
    float *dCsrVal, *dA;
    int *dCsrRowPtr, *dCsrColInd;

    CHECK_CUDA( hipMalloc((void**) &dCsrVal,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(float)));

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseScsr2dense(handle, m, n, descrA, dCsrVal, dCsrRowPtr, dCsrColInd, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i] - hA_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2dense test PASSED\n");
    else
        printf("csr2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCsrVal) );
    CHECK_CUDA(hipFree(dCsrRowPtr) );
    CHECK_CUDA(hipFree(dCsrColInd) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
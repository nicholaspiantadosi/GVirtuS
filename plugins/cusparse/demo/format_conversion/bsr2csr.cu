#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    const int blockDim = 2;
    const int mb = 2;

    float hBsrValA[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 9};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    int m = mb * blockDim;
    int nnzb = 4; // number of blocks
    int nnz  = nnzb * blockDim * blockDim; // number of elements

    float hCsrValC[nnz];
    int hCsrRowPtrC[m+1];
    int hCsrColIndC[nnz];

    float hCsrValC_result[] = {1.000000, 0.000000, 2.000000, 3.000000, 0.000000, 4.000000, 0.000000, 0.000000, 5.000000, 0.000000, 6.000000, 7.000000, 0.000000, 8.000000, 0.000000, 9.000000};
    int hCsrRowPtrC_result[] = {0, 4, 8, 12, 16};
    int hCsrColIndC_result[] = {0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3};

    // Device memory management

    float *dBsrValA, *dCsrValC;
    int *dBsrRowPtrA, *dBsrColIndA, *dCsrRowPtrC, *dCsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (blockDim * blockDim) * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrValC, nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrC, (m + 1) * sizeof(int)));
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndC, nnz * sizeof(int)));

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (blockDim * blockDim) * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrValC, hCsrValC, nnz * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrC, hCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndC, hCsrColIndC, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    // Given BSR format (bsrRowPtrA, bsrcolIndA, bsrValA) and
    // blocks of BSR format are stored in column-major order.
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    hipsparseSbsr2csr(handle, dir, mb, mb,
                     descrA,
                     dBsrValA, dBsrRowPtrA, dBsrColIndA,
                     blockDim,
                     descrC,
                     dCsrValC, dCsrRowPtrC, dCsrColIndC);

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrValC, dCsrValC, nnz * sizeof(float), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrC, dCsrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndC, dCsrColIndC, nnz * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCsrValC[i] - hCsrValC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
        if(hCsrColIndC[i] - hCsrColIndC_result[i] > 0) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((hCsrRowPtrC[i] - hCsrRowPtrC_result[i] > 0)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("bsr2csr test PASSED\n");
    else
        printf("bsr2csr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValC));
    CHECK_CUDA(hipFree(dCsrRowPtrC));
    CHECK_CUDA(hipFree(dCsrColIndC));
    CHECK_CUDA(hipFree(dBsrValA));
    CHECK_CUDA(hipFree(dBsrRowPtrA));
    CHECK_CUDA(hipFree(dBsrColIndA));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
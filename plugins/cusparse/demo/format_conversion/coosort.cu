#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnz = 9;

    /*
    float hA [] = {1, 0, 5, 0,
                  4, 2, 0, 0,
                  0, 3, 0, 9,
                  0, 0, 7, 0,
                  0, 0, 8, 6};
    */

    float hCooValA[] = {1, 4, 2, 3, 5, 7, 8, 9, 6};
    int hCooRowIndA[] = {0, 0, 1, 1, 2, 2, 2, 3, 3};
    int hCooColIndA[] = {0, 1, 1, 2, 0, 3, 4, 2, 4};

    int hp[nnz];
    float hCooValASorted[nnz];

    int hCooRowIndA_resultByRow[] = {0, 0, 1, 1, 2, 2, 2, 3, 3};
    int hCooColIndA_resultByRow[] = {0, 1, 1, 2, 0, 3, 4, 2, 4};
    int hp_resultByRow[] = {0, 1, 2, 3, 4, 5, 6, 7, 8};
    float hCooValASorted_resultByRow[] = {1, 4, 2, 3, 5, 7, 8, 9, 6};

    int hCooRowIndA_resultByColumn[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    int hCooColIndA_resultByColumn[] = {0, 0, 1, 1, 2, 2, 3, 4, 4};
    int hp_resultByColumn[] = {0, 4, 1, 2, 3, 7, 5, 6, 8};
    float hCooValASorted_resultByColumn[] = {1, 5, 4, 2, 3, 9, 7, 8, 6};

    // Device memory management
    float *dCooValA, *dCooValASorted;
    int *dCooRowIndA, *dCooColIndA, *dp;

    //CHECK_CUDA( hipMalloc((void**) &dCooValA,  nnz * sizeof(float)));
    CHECK_CUDA( hipMalloc((void**) &dCooRowIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCooColIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCooValA, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dCooValASorted, nnz * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**) &dp, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCooRowIndA, hCooRowIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCooColIndA, hCooColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCooValA, hCooValA, nnz * sizeof(float), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    size_t pBufferSizeInBytes;
    void *pBuffer = 0;

    CHECK_CUSPARSE(hipsparseXcoosort_bufferSizeExt(handle, m, n, nnz, dCooRowIndA, dCooColIndA, &pBufferSizeInBytes));

    pBufferSizeInBytes = 1152;

    CHECK_CUDA(hipMalloc((void**)&pBuffer, pBufferSizeInBytes));

    CHECK_CUSPARSE(hipsparseCreateIdentityPermutation(handle, nnz, dp));

    CHECK_CUSPARSE(hipsparseXcoosortByRow(handle, m, n, nnz, dCooRowIndA, dCooColIndA, dp, pBuffer));

    CHECK_CUSPARSE(hipsparseSgthr(handle, nnz, dCooValA, dCooValASorted, dp, HIPSPARSE_INDEX_BASE_ZERO));

    // device result check
    CHECK_CUDA( hipMemcpy(hCooRowIndA, dCooRowIndA, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCooColIndA, dCooColIndA, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hp, dp, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCooValASorted, dCooValASorted, nnz * sizeof(float), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnz; i++) {
        if((fabs(hCooRowIndA[i] - hCooRowIndA_resultByRow[i]) > 0.000001)
        || (fabs(hCooColIndA[i] - hCooColIndA_resultByRow[i]) > 0.000001)
        || (fabs(hp[i] - hp_resultByRow[i]) > 0.000001)
        || (fabs(hCooValASorted[i] - hCooValASorted_resultByRow[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    CHECK_CUSPARSE(hipsparseXcoosortByColumn(handle, m, n, nnz, dCooRowIndA, dCooColIndA, dp, pBuffer));

    CHECK_CUSPARSE(hipsparseSgthr(handle, nnz, dCooValA, dCooValASorted, dp, HIPSPARSE_INDEX_BASE_ZERO));

    // device result check
    CHECK_CUDA( hipMemcpy(hCooRowIndA, dCooRowIndA, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCooColIndA, dCooColIndA, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hp, dp, nnz * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCooValASorted, dCooValASorted, nnz * sizeof(float), hipMemcpyDeviceToHost) );

    for (int i = 0; i < nnz; i++) {
        if((fabs(hCooRowIndA[i] - hCooRowIndA_resultByColumn[i]) > 0.000001)
           || (fabs(hCooColIndA[i] - hCooColIndA_resultByColumn[i]) > 0.000001)
           || (fabs(hp[i] - hp_resultByColumn[i]) > 0.000001)
           || (fabs(hCooValASorted[i] - hCooValASorted_resultByColumn[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("coosort test PASSED\n");
    else
        printf("coosort test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dCooRowIndA) );
    CHECK_CUDA(hipFree(dCooColIndA) );
    CHECK_CUDA(hipFree(dp) );
    CHECK_CUDA(hipFree(dCooValA) );
    CHECK_CUDA(hipFree(dCooValASorted) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
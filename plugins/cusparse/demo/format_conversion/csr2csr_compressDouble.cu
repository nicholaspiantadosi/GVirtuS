#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include "utilities.h"

int main(){
    int m = 6, n = 5;
    hipsparseHandle_t  handle;
    CHECK_CUSPARSE( hipsparseCreate(&handle) );
    hipsparseMatDescr_t descrX;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrX));
    int total_nnz = 13;

    double *csrValX;
    int *csrRowPtrX;
    int *csrColIndX;
    CHECK_CUDA( hipMalloc((void**) &csrValX, sizeof(double) * total_nnz) );
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrX, sizeof(int) * (m+1))) ;
    CHECK_CUDA( hipMalloc((void**) &csrColIndX, sizeof(int) * total_nnz)) ;

    double hCsrVal[] = {1, 3, -4, 5, 2, 7, 8, 6, 9, 3.5, 5.5, 6.5, -9.9};
    int hCsrRowPtrX[] = {0, 2, 4, 7, 9, 11, 13};
    int hCsrColIndX[] = {0, 1, 1, 2, 0, 3, 4, 2, 4, 3, 4, 0, 2};

    CHECK_CUDA( hipMemcpy(csrValX, hCsrVal, total_nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrRowPtrX, hCsrRowPtrX, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(csrColIndX, hCsrColIndX, total_nnz * sizeof(int), hipMemcpyHostToDevice) );

    double tol = 3.5;
    int *nnzPerRowY;
    int testNNZTotal;
    CHECK_CUDA( hipMalloc((void**) &nnzPerRowY,  m * sizeof(int)));
    CHECK_CUSPARSE( hipsparseDnnz_compress(handle, m, descrX, csrValX,
                                         csrRowPtrX, nnzPerRowY,
                                         &testNNZTotal, tol));
    double *csrValY;
    int *csrRowPtrY;
    int *csrColIndY;
    CHECK_CUDA( hipMalloc((void**) &csrValY, sizeof(double) * testNNZTotal));
    CHECK_CUDA( hipMalloc((void**) &csrRowPtrY, sizeof(int) * (m+1)));
    CHECK_CUDA( hipMalloc((void**) &csrColIndY, sizeof(int) * testNNZTotal));

    CHECK_CUSPARSE( hipsparseDcsr2csr_compress( handle, m, n, descrX, csrValX,
                                              csrColIndX, csrRowPtrX,
                                              total_nnz,  nnzPerRowY,
                                              csrValY, csrColIndY,
                                              csrRowPtrY, tol));

    int hNnzPerRowY[m];
    double hCsrValY[testNNZTotal];
    int hCsrRowPtrY[m + 1];
    int hCsrColIndY[testNNZTotal];

    CHECK_CUDA( hipMemcpy(hNnzPerRowY, nnzPerRowY, m * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrValY, csrValY, testNNZTotal * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrRowPtrY, csrRowPtrY, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hCsrColIndY, csrColIndY, testNNZTotal * sizeof(int), hipMemcpyDeviceToHost) );

    int hNnzPerRowY_result[] = {0, 2, 2, 2, 1, 2};
    int hCsrRowPtrY_result[] = {0, 0, 2, 4, 6, 7, 9};

    int correct = 1;
    for (int i = 0; i < m; i++) {
        if((fabs(hNnzPerRowY[i] - hNnzPerRowY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtrY[i] - hCsrRowPtrY_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2csr_compressed test PASSED\n");
    else
        printf("csr2csr_compressed test FAILED: wrong result\n");

    hipsparseDestroy(handle);
    hipFree(csrValX);
    hipFree(csrRowPtrX);
    hipFree(csrColIndX);
    hipFree(csrValY);
    hipFree(nnzPerRowY);
    hipFree(csrRowPtrY);
    hipFree(csrColIndY);
    return 0;
}

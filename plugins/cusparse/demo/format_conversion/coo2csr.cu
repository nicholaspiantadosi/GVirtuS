#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition

    int hCooRowInd[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    const int nnz = 9;
    const int m = 4;

    int hCsrRowPtr[m + 1];

    int hCsrRowPtr_result[] = {0, 0, 1, 2, 3};

    // Device memory management

    int *dCooRowInd, *dCsrRowPtr;

    CHECK_CUDA( hipMalloc((void**) &dCooRowInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCooRowInd, hCooRowInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseStatus_t cs = hipsparseXcoo2csr(handle, dCooRowInd, nnz, m, dCsrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);

    // device result check
    CHECK_CUDA( hipMemcpy(hCsrRowPtr, dCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < (m + 1); i++) {
        if((fabs(hCsrRowPtr[i] - hCsrRowPtr_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }

    if (correct)
        printf("coo2csr test PASSED\n");
    else
        printf("coo2csr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUDA(hipFree(dCsrRowPtr));
    CHECK_CUDA(hipFree(dCooRowInd));

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
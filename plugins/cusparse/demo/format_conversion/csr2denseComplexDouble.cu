#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 4;
    int nnz = 9;
    int lda = m;

    hipDoubleComplex hCsrVal[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(2, 0), make_hipDoubleComplex(3, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(8, 0), make_hipDoubleComplex(9, 0)};
    int hCsrRowPtr[] = {0, 3, 4, 7, 9};
    int hCsrColInd[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    hipDoubleComplex hA[m * n];

    hipDoubleComplex hA_result[] = {make_hipDoubleComplex(1, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(5, 0), make_hipDoubleComplex(0, 0),
                             make_hipDoubleComplex(0, 0), make_hipDoubleComplex(4, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(8, 0),
                             make_hipDoubleComplex(2, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(6, 0), make_hipDoubleComplex(0, 0),
                             make_hipDoubleComplex(3, 0), make_hipDoubleComplex(0, 0), make_hipDoubleComplex(7, 0), make_hipDoubleComplex(9, 0)};

    // Device memory management
    hipDoubleComplex *dCsrVal, *dA;
    int *dCsrRowPtr, *dCsrColInd;

    CHECK_CUDA( hipMalloc((void**) &dCsrVal,  nnz * sizeof(hipDoubleComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtr, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColInd, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(hipDoubleComplex)));

    CHECK_CUDA( hipMemcpy(dCsrVal, hCsrVal, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtr, hCsrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColInd, hCsrColInd, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseZcsr2dense(handle, m, n, descrA, dCsrVal, dCsrRowPtr, dCsrColInd, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i].x - hA_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2dense test PASSED\n");
    else
        printf("csr2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCsrVal) );
    CHECK_CUDA(hipFree(dCsrRowPtr) );
    CHECK_CUDA(hipFree(dCsrColInd) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
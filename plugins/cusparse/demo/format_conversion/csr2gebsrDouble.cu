#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 4;
    int nnz = 9;

    const int rowBlockDim = 2;
    const int colBlockDim = 2;

    int mb = (m + rowBlockDim-1)/rowBlockDim;

    double hCsrValA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int hCsrRowPtrA[] = {0, 3, 4, 7, 9};
    int hCsrColIndA[] = {0, 2, 3, 1, 0, 2, 3, 1, 3};

    double hBsrValC_result[] = {1, 0, 0, 4, 2, 0, 3, 0, 5, 0, 0, 8, 6, 0, 7, 9};
    int hBsrRowPtrC_result[] = {0, 2, 4};
    int hBsrColIndC_result[] = {0, 1, 0, 1};

    // Device memory management
    double *dCsrValA, *dBsrValC;
    int *dCsrRowPtrA, *dCsrColIndA, *dBsrRowPtrC, *dBsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dCsrValA,  nnz * sizeof(double)));
    CHECK_CUDA( hipMalloc((void**) &dCsrRowPtrA, (m + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCsrColIndA, nnz * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dCsrValA, hCsrValA, nnz * sizeof(double), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrRowPtrA, hCsrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCsrColIndA, hCsrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    // Given CSR format (csrRowPtrA, csrColIndA, csrValA) and
    // blocks of BSR format are stored in column-major order.
    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;
    int base, nnzb;
    int bufferSize;
    void *pBuffer;
    hipsparseDcsr2gebsr_bufferSize(handle, dir, m, n,
                                  descrA, dCsrValA, dCsrRowPtrA, dCsrColIndA,
                                  rowBlockDim, colBlockDim,
                                  &bufferSize);

    if (bufferSize == 0) {
        bufferSize = 528;
    }

    hipMalloc((void**)&pBuffer, bufferSize);
    hipMalloc((void**)&dBsrRowPtrC, sizeof(int) *(mb+1));
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzb;
    hipsparseXcsr2gebsrNnz(handle, dir, m, n,
                          descrA, dCsrRowPtrA, dCsrColIndA,
                          descrC, dBsrRowPtrC, rowBlockDim, colBlockDim,
                          nnzTotalDevHostPtr,
                          pBuffer);
    if (NULL != nnzTotalDevHostPtr){
        nnzb = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzb, dBsrRowPtrC+mb, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, dBsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzb -= base;
    }
    hipMalloc((void**)&dBsrColIndC, sizeof(int)*nnzb);
    hipMalloc((void**)&dBsrValC, sizeof(double)*(rowBlockDim*colBlockDim)*nnzb);
    hipsparseDcsr2gebsr(handle, dir, m, n,
                       descrA,
                       dCsrValA, dCsrRowPtrA, dCsrColIndA,
                       descrC,
                       dBsrValC, dBsrRowPtrC, dBsrColIndC,
                       rowBlockDim, colBlockDim,
                       pBuffer);

    double hBsrValC[nnzb * rowBlockDim * colBlockDim];
    int hBsrRowPtrC[mb + 1];
    int hBsrColIndC[nnzb];

    // device result check
    CHECK_CUDA( hipMemcpy(hBsrValC, dBsrValC, nnzb * rowBlockDim * colBlockDim * sizeof(double), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrRowPtrC, dBsrRowPtrC, (mb + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrColIndC, dBsrColIndC, nnzb * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnzb * rowBlockDim * colBlockDim; i++) {
        if((fabs(hBsrValC[i] - hBsrValC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (mb + 1); i++) {
        if((fabs(hBsrRowPtrC[i] - hBsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzb; i++) {
        if((fabs(hBsrColIndC[i] - hBsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csr2gebsr test PASSED\n");
    else
        printf("csr2gebsr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(dCsrValA) );
    CHECK_CUDA(hipFree(dCsrRowPtrA) );
    CHECK_CUDA(hipFree(dCsrColIndA) );
    CHECK_CUDA(hipFree(dBsrValC) );
    CHECK_CUDA(hipFree(dBsrRowPtrC) );
    CHECK_CUDA(hipFree(dBsrColIndC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
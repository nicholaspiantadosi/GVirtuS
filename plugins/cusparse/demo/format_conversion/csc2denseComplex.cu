#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    int m = 4;
    int n = 5;
    int nnz = 9;
    int lda = m;

    hipComplex hCscValA[] = {make_hipComplex(1, 0), make_hipComplex(5, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(3, 0), make_hipComplex(9, 0), make_hipComplex(7, 0), make_hipComplex(8, 0), make_hipComplex(6, 0)};
    int hCscRowIndA[] = {0, 2, 0, 1, 1, 3, 2, 2, 3};
    int hCscColPtrA[] = {0, 2, 4, 6, 7, 9};

    hipComplex hA[m * n];

    hipComplex hA_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(5, 0), make_hipComplex(0, 0),
                             make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(0, 0),
                             make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0), make_hipComplex(9, 0),
                             make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(0, 0),
                             make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(6, 0)};

    // Device memory management
    hipComplex *dCscValA, *dA;
    int *dCscRowIndA, *dCscColPtrA;

    CHECK_CUDA( hipMalloc((void**) &dCscValA,  nnz * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dCscRowIndA, nnz * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dCscColPtrA, (n + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dA,  lda * n * sizeof(hipComplex)));

    CHECK_CUDA( hipMemcpy(dCscValA, hCscValA, nnz * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscRowIndA, hCscRowIndA, nnz * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dCscColPtrA, hCscColPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseCcsc2dense(handle, m, n, descrA, dCscValA, dCscRowIndA, dCscColPtrA, dA, lda);

    // device result check
    CHECK_CUDA( hipMemcpy(hA, dA, lda * n * sizeof(hipComplex), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < lda * n; i++) {
        if((fabs(hA[i].x - hA_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("csc2dense test PASSED\n");
    else
        printf("csc2dense test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUDA(hipFree(dCscValA) );
    CHECK_CUDA(hipFree(dCscRowIndA) );
    CHECK_CUDA(hipFree(dCscColPtrA) );
    CHECK_CUDA(hipFree(dA) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}
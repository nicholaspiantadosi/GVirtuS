#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hipsparse.h>
#include <time.h>

#include "utilities.h"
#include <hip/hip_runtime_api.h>

#include <limits>

int main(int argn, char *argv[])
{

    // Host problem definition
    const int rowBlockDimA = 2;
    const int colBlockDimA = 2;
    const int mb = 2;
    const int nb = 2;
    const int nnzb = 4;
    const int rowBlockDimC = 2;
    const int colBlockDimC = 2;
    int m = mb * rowBlockDimA;
    int mc = (m + rowBlockDimC - 1)/rowBlockDimC;

    hipComplex hBsrValA[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                            make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};
    int hBsrRowPtrA[] = {0, 2, 4};
    int hBsrColIndA[] = {0, 1, 0, 1};

    hipComplex hBsrValC_result[] = {make_hipComplex(1, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(4, 0), make_hipComplex(2, 0), make_hipComplex(0, 0), make_hipComplex(3, 0), make_hipComplex(0, 0),
                                   make_hipComplex(5, 0), make_hipComplex(0, 0), make_hipComplex(0, 0), make_hipComplex(8, 0), make_hipComplex(6, 0), make_hipComplex(0, 0), make_hipComplex(7, 0), make_hipComplex(9, 0)};
    int hBsrRowPtrC_result[] = {0, 2, 4};
    int hBsrColIndC_result[] = {0, 1, 0, 1};

    // Device memory management
    hipComplex *dBsrValA, *dBsrValC;
    int *dBsrRowPtrA, *dBsrColIndA, *dBsrRowPtrC, *dBsrColIndC;

    CHECK_CUDA( hipMalloc((void**) &dBsrValA,  nnzb * (rowBlockDimA * colBlockDimA) * sizeof(hipComplex)));
    CHECK_CUDA( hipMalloc((void**) &dBsrRowPtrA, (mb + 1) * sizeof(int)) );
    CHECK_CUDA( hipMalloc((void**) &dBsrColIndA, nnzb * sizeof(int)) );

    CHECK_CUDA( hipMemcpy(dBsrValA, hBsrValA, nnzb * (rowBlockDimA * colBlockDimA) * sizeof(hipComplex), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrRowPtrA, hBsrRowPtrA, (mb + 1) * sizeof(int), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dBsrColIndA, hBsrColIndA, nnzb * sizeof(int), hipMemcpyHostToDevice) );

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

    int base, nnzc;
    int bufferSize;
    void *pBuffer = 0;

    hipsparseMatDescr_t descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseMatDescr_t descrC = 0;
    hipsparseCreateMatDescr(&descrC);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL );

    hipsparseCgebsr2gebsr_bufferSize(handle, dir, mb, nb, nnzb,
                                    descrA, hBsrValA, hBsrRowPtrA, hBsrColIndA,
                                    rowBlockDimA, colBlockDimA,
                                    rowBlockDimC, colBlockDimC,
                                    &bufferSize);
    if(bufferSize == 0) {
        bufferSize = 912;
    }
    hipMalloc((void**)&pBuffer, bufferSize);
    hipMalloc((void**)&dBsrRowPtrC, sizeof(int)*(mc+1));
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzc;
    hipsparseXgebsr2gebsrNnz(handle, dir, mb, nb, nnzb,
                            descrA, dBsrRowPtrA, dBsrColIndA,
                            rowBlockDimA, colBlockDimA,
                            descrC, dBsrRowPtrC,
                            rowBlockDimC, colBlockDimC,
                            nnzTotalDevHostPtr,
                            pBuffer);
    if (NULL != nnzTotalDevHostPtr){
        nnzc = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzc, dBsrRowPtrC + mc, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, dBsrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzc -= base;
    }
    hipMalloc((void**)&dBsrColIndC, sizeof(int)*nnzc);
    hipMalloc((void**)&dBsrValC, sizeof(hipComplex)*(rowBlockDimC*colBlockDimC)*nnzc);
    hipsparseCgebsr2gebsr(handle, dir, mb, nb, nnzb,
                         descrA, dBsrValA, dBsrRowPtrA, dBsrColIndA,
                         rowBlockDimA, colBlockDimA,
                         descrC, dBsrValC, dBsrRowPtrC, dBsrColIndC,
                         rowBlockDimC, colBlockDimC,
                         pBuffer);

    // device result check
    hipComplex hBsrValC[nnzc * rowBlockDimC * colBlockDimC];
    int hBsrRowPtrC[mc + 1];
    int hBsrColIndC[nnzc];

    CHECK_CUDA( hipMemcpy(hBsrValC, dBsrValC, nnzc * (rowBlockDimC * colBlockDimC) * sizeof(hipComplex), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrRowPtrC, dBsrRowPtrC, (mc + 1) * sizeof(int), hipMemcpyDeviceToHost) );
    CHECK_CUDA( hipMemcpy(hBsrColIndC, dBsrColIndC, nnzc * sizeof(int), hipMemcpyDeviceToHost) );

    int correct = 1;
    for (int i = 0; i < nnzc * (rowBlockDimC * colBlockDimC); i++) {
        if((fabs(hBsrValC[i].x - hBsrValC_result[i].x) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < (mc + 1); i++) {
        if((fabs(hBsrRowPtrC[i] - hBsrRowPtrC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    for (int i = 0; i < nnzc; i++) {
        if((fabs(hBsrColIndC[i] - hBsrColIndC_result[i]) > 0.000001)) {
            correct = 0;
            break;
        }
    }
    if (correct)
        printf("gebsr2gebsr test PASSED\n");
    else
        printf("gebsr2gebsr test FAILED: wrong result\n");

    // step 6: free resources

    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
    CHECK_CUDA(hipFree(pBuffer));
    CHECK_CUDA(hipFree(dBsrValA) );
    CHECK_CUDA(hipFree(dBsrRowPtrA) );
    CHECK_CUDA(hipFree(dBsrColIndA) );
    CHECK_CUDA(hipFree(dBsrValC) );
    CHECK_CUDA(hipFree(dBsrRowPtrC) );
    CHECK_CUDA(hipFree(dBsrColIndC) );

    // destroy
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return EXIT_SUCCESS;
}